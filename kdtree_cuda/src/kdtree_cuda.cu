#include "hip/hip_runtime.h"


#include <string.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kdtree_cuda/kdtree_cuda.hpp"
#include <vector>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

int sort_axis=0;

typedef struct
{
	int	id;
	float pos[3];
} point_with_id;

typedef struct
{
	int	parent_id;
	int left_id;
	int right_id;
	int axis;
	bool node_is_right;
} node;

bool first=true;

//	年齢(昇順)
int AxisSort(const void * n1, const void * n2)
{
	if (((point_with_id *)n1)->pos[sort_axis] > ((point_with_id *)n2)->pos[sort_axis])
	{
		return 1;
	}
	else if (((point_with_id *)n1)->pos[sort_axis] < ((point_with_id *)n2)->pos[sort_axis])
	{
		return -1;
	}
	else
	{
		return 0;
	}
}

int CreateTree(int* root_id,std::vector <node>& nodes, std::vector<std::vector<float>> points,std::vector<int> group_indices,int parent_id,bool node_is_right)
{
	//入力データ初期化
	int group_size = group_indices.size();
	// std::cout<<"group_size"<<group_size<<std::endl;
	point_with_id point_with_ids[group_size];
	point_with_id axis_point_with_ids[3][group_size];
	for(sort_axis=0; sort_axis<3; sort_axis++){
		for(int i=0;i<group_size;i++){
			point_with_ids[i].id=group_indices[i];
			point_with_ids[i].pos[0]=points[group_indices[i]][0];
			point_with_ids[i].pos[1]=points[group_indices[i]][1];
			point_with_ids[i].pos[2]=points[group_indices[i]][2];
		}
	}

	//ソート
	float max[3],min[3],median[3],length[3];
	int axis_median_id[3];
	int median_id;
	for(sort_axis=0; sort_axis<3; sort_axis++){//x,y,zそれぞれにソート
		// std::cout<<"sort_axis = "<<sort_axis<<std::endl;

		qsort(point_with_ids, group_size, sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < group_size ; i++)
		{
			axis_point_with_ids[sort_axis][i].id = point_with_ids[i].id;
			axis_point_with_ids[sort_axis][i].pos[0] = point_with_ids[i].pos[0];
			axis_point_with_ids[sort_axis][i].pos[1] = point_with_ids[i].pos[1];
			axis_point_with_ids[sort_axis][i].pos[2] = point_with_ids[i].pos[2];
			// printf("%d, %f, %f, %f \n", point_with_ids[i].id, point_with_ids[i].pos[0], point_with_ids[i].pos[1], point_with_ids[i].pos[2]);
		}
		// std::cout<<std::endl;
		//max,min,median,axis_median_id取得
		max[sort_axis]=point_with_ids[group_size-1].pos[sort_axis];//minとmaxいらんかも
		min[sort_axis]=point_with_ids[0].pos[sort_axis];
		length[sort_axis]=max[sort_axis]-min[sort_axis];
		median[sort_axis]=point_with_ids[(group_size-1)/2].pos[sort_axis];//偶数なら小さい方(-1)消せば大きい方
		axis_median_id[sort_axis]=point_with_ids[(group_size-1)/2].id;
	}
	// std::cout<<"x_length = "<< length[0] <<", x_median["<<axis_median_id[0]<<"] = "<<median[0]<<std::endl;
	// std::cout<<"y_length = "<< length[1] <<", y_median["<<axis_median_id[1]<<"] = "<<median[1]<<std::endl;
	// std::cout<<"z_length = "<< length[2] <<", z_median["<<axis_median_id[2]<<"] = "<<median[2]<<std::endl;
	// std::cout<<std::endl;
	// for(int i=0;i<group_size;i++){
	// 	std::cout<<"point_id["<<i<<"] = "<<point_with_ids[i].id<<std::endl;
	// }
	

	//中央値id設定、長軸設定
	if(length[0]>=length[1]&&length[0]>=length[2]){
		median_id=axis_median_id[0];
		nodes[median_id].axis=0;
	}
	if(length[1]>=length[0]&&length[1]>=length[2]){
		median_id=axis_median_id[1];
		nodes[median_id].axis=1;
	}
	if(length[2]>=length[0]&&length[2]>=length[1]){
		median_id=axis_median_id[2];
		nodes[median_id].axis=2;
	}

	//node初期化
	nodes[median_id].left_id=-1;
	nodes[median_id].right_id=-1;

	//親設定、親の左右設定
	nodes[median_id].parent_id=parent_id;
	if(parent_id>=0){//親あり
		if(!node_is_right) nodes[parent_id].left_id=median_id;
		if(node_is_right) nodes[parent_id].right_id=median_id;
	}
	else{//親なし
		*root_id=median_id;
	}
	std::vector<int> right_group(group_size);
	std::vector<int> left_group(group_size);
	int right_count=0;
	int left_count=0;
	for(int i=0;i<=((group_size-1)/2)-1;i++){//愚直
		left_group[left_count] = axis_point_with_ids[nodes[median_id].axis][i].id;
		left_count++;
	}
	left_group.resize(left_count);
	for(int i=((group_size-1)/2)+1;i<group_size;i++){
		right_group[right_count] = axis_point_with_ids[nodes[median_id].axis][i].id;
		right_count++;
	}
	right_group.resize(right_count);
	// std::cout<<"median_id"<<median_id<<std::endl;
	// std::cout<<"parent_id"<<parent_id<<std::endl;
	// std::cout<<"left_id"<<nodes[median_id].left_id<<std::endl;
	// std::cout<<"right_id"<<nodes[median_id].right_id<<std::endl;
	// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;

	//right,left group表示
	// std::cout<<"left_group is (";
	// for(int i=0;i<left_group.size();i++){
	// 	std::cout<<left_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<"right_group is (";
	// for(int i=0;i<right_group.size();i++){
	// 	std::cout<<right_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<"--------------------------------------------------------------------------------"<<std::endl;
	bool left=false;
	bool right=false;
	if(group_size>1){//子がいる
		if(left_group.size()>0){//左に子がいる
			left= CreateTree(root_id,nodes,points,left_group,median_id,false);
		}
		else left=true;
		if(right_group.size()>0){//右に子がいる
			right= CreateTree(root_id,nodes,points,right_group,median_id,true);
		}
		else right=true;
		if(right&&left) return 1;
	}
	else return 1;//子がいない
}

void PointRangeCheckAndAdd(std::vector<int>& range_indices,int head_id,std::vector<std::vector<float>> points,std::vector<float> search_point,float range_sq)
{
	float dist_sq = pow(points[head_id][0]-search_point[0],2)+pow(points[head_id][1]-search_point[1],2)+pow(points[head_id][2]-search_point[2],2);
	if(dist_sq<range_sq){
		range_indices.push_back(head_id);
	} 
}

int SearchSubTree(std::vector<int>& range_indices,int root_id,std::vector <node> nodes,std::vector<std::vector<float>> points,std::vector<float> search_point,float range_sq)
{
	int head_id = root_id;
	bool cross,next_is_right;
	//潜り
	while(1){
		if(search_point[nodes[head_id].axis]>points[head_id][nodes[head_id].axis]) next_is_right = true;
		else next_is_right = false;

		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
			if(next_is_right){//right
				head_id = nodes[head_id].right_id;
			}
			else{//left
				head_id = nodes[head_id].left_id;
			}
		}
		else break;
	}
	//rootが底
	if(head_id==root_id) {
		PointRangeCheckAndAdd(range_indices,head_id,points,search_point,range_sq);
		return 1;
	}
	//昇り
	int last_id;
	while(1){
		cross = false;
		PointRangeCheckAndAdd(range_indices,head_id,points,search_point,range_sq);
		//昇る
		last_id = head_id;
		head_id = nodes[head_id].parent_id;
		//交差判定　1軸のみ低効率
		float axis_diff_sq = pow(points[head_id][nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
		if(axis_diff_sq < range_sq) cross = true;
		int sub_tree=0;
		if(cross){
			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = SearchSubTree(range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = SearchSubTree(range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
		}
		if(head_id==root_id){
			PointRangeCheckAndAdd(range_indices,head_id,points,search_point,range_sq);
			break;
		}
	}
	return 1;
}

// __device__ void d_PointRangeCheckAndAdd(std::vector<int>& range_indices,int head_id,std::vector<std::vector<float>> points,std::vector<float> search_point,float range_sq)//vecotrなしpowなしpushbackなしで作り直す
// {
// 	float dist_sq = pow(points[head_id][0]-search_point[0],2)+pow(points[head_id][1]-search_point[1],2)+pow(points[head_id][2]-search_point[2],2);
// 	if(dist_sq<range_sq){
// 		range_indices.push_back(head_id);
// 	} 
// }

// __device__ int d_SearchSubTree(std::vector<int>& range_indices,int root_id,std::vector <node> nodes,std::vector<std::vector<float>> points,std::vector<float> search_point,float range_sq)//vecotrなしpowなしで作り直す
// {
// 	int head_id = root_id;
// 	bool cross,next_is_right;
// 	//潜り
// 	while(1){
// 		if(search_point[nodes[head_id].axis]>points[head_id][nodes[head_id].axis]) next_is_right = true;
// 		else next_is_right = false;

// 		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
// 		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
// 		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
// 			if(next_is_right){//right
// 				head_id = nodes[head_id].right_id;
// 			}
// 			else{//left
// 				head_id = nodes[head_id].left_id;
// 			}
// 		}
// 		else break;
// 	}
// 	//rootが底
// 	if(head_id==root_id) {
// 		d_PointRangeCheckAndAdd(range_indices,head_id,points,search_point,range_sq);
// 		return 1;
// 	}
// 	//昇り
// 	int last_id;
// 	while(1){
// 		cross = false;
// 		d_PointRangeCheckAndAdd(range_indices,head_id,points,search_point,range_sq);
// 		//昇る
// 		last_id = head_id;
// 		head_id = nodes[head_id].parent_id;
// 		//交差判定　1軸のみ低効率
// 		float axis_diff_sq = pow(points[head_id][nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
// 		if(axis_diff_sq < range_sq) cross = true;
// 		int sub_tree=0;
// 		if(cross){
// 			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = d_SearchSubTree(range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
// 			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = d_SearchSubTree(range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
// 		}
// 		if(head_id==root_id){
// 			d_PointRangeCheckAndAdd(range_indices,head_id,points,search_point,range_sq);
// 			break;
// 		}
// 	}
// 	return 1;
// }

__device__ int EigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 

__global__ void NormalsGPU(float* test_points,int* d_parent_ids,int* d_left_ids,int* d_right_ids,int* d_axes,int root_id,int test_points_size,float* points,int point_size,int* neighbor_points_indices,int* neighbor_start_indices,int neighbor_points_count,float* normals,float* curvatures,long long int* covariance_time,long long int* eigen_time)
{
    // printf("normalsGPU");
    //インデックス取得
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
    unsigned int output_id=50;
    // printf("idx = %d ,", idx);
	if(idx==output_id){
		// std::vector <node> nodes(test_points_size);
		// std::vector<float> search_point={11,5,0};
		// std::vector<int> range_indices;
		// float range_sq = 3*3;
		// //探索関数の実行
		// int range_search = d_SearchSubTree(range_indices,root_id,*nodes,*test_points,search_point,range_sq);//nodes,test_points
		// // std::cout<<"range_indices.size()"<<range_indices.size()<<std::endl;
		// if(range_search==1) {
		// 	printf("device range_indices is [");
		// 	for(int i=0;i<range_indices.size();i++){
		// 		printf("%d,",range_indices[i]);
		// 	}
		// 	printf("]\n");
		// }
	}

    if(idx<point_size-1){//対象スレッド内のみ計算
        //デバッグ用
        // if(idx==0||idx==10||idx==20) printf("points(%d) = %f,%f,%f\n",idx,points[idx*3+0],points[idx*3+1],points[idx*3+2]);
        
        // printf("idx<point_size");
        //近傍点終点インデックスの定義
        int end_indices;
        if(idx==(point_size-1)) end_indices = neighbor_points_count;
        else end_indices = neighbor_start_indices[idx+1]-1;

        int neighbor_size=(end_indices-neighbor_start_indices[idx]+1)/3;
        // if(idx==0||idx==10||idx==20) printf("neighbor(%d) start = %d, end = %d,size = %d\n",idx,neighbor_start_indices[idx],end_indices,neighbor_size);
        // printf("neighbor_size = %d\n", neighbor_size);
        if(neighbor_size>=3){//近傍点数3以上
            long long int covariance_start, covariance_stop;
            long long int eigen_start,eigen_stop;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_start));
            // printf("neighbor_size>=3");
            //平均計算
            float x_average=0,y_average=0,z_average=0;
            // if(idx==output_id) printf("neighbor_size = %d\n",neighbor_size);

            // if(idx==output_id) printf("neighbor_points(%d) = {\n",idx);
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                // デバッグ用
                // if((idx==0||idx==10||idx==20)&&(i==(neighbor_start_indices[idx]))) printf("neighbor_points_indices(%d) = %d\n",idx,neighbor_points_indices[i]);
                // if((idx==0||idx==10||idx==20)&&i==neighbor_start_indices[idx]) printf("neighbor_points(%d) = %f,%f,%f\n",idx,points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                // if(idx==output_id) printf("{%f, %f, %f},\n",points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                x_average+=points[neighbor_points_indices[i]*3+0];
                y_average+=points[neighbor_points_indices[i]*3+1];
                z_average+=points[neighbor_points_indices[i]*3+2];
            }
            // if(idx==output_id) printf("};\n");
            x_average/=neighbor_size;
            y_average/=neighbor_size;
            z_average/=neighbor_size;

            // //要素計算
            float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                sxx+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+0]-x_average);
                syy+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                szz+=(points[neighbor_points_indices[i]*3+2]-z_average)*(points[neighbor_points_indices[i]*3+2]-z_average);

                sxy+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                sxz+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
                syz+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
            }

            sxx/=neighbor_size;
            syy/=neighbor_size;
            szz/=neighbor_size;
            sxy/=neighbor_size;
            sxz/=neighbor_size;
            syz/=neighbor_size;

            //共分散行列
            float a[3*3]={
                sxx,sxy,sxz,
                sxy,syy,syz,
                sxz,syz,szz,
            };

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_stop));
            covariance_time[idx]=covariance_stop - covariance_start;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_start));
            
            // __syncthreads();
            // if(idx==output_id){
            //     printf("                          %f ,%f ,%f \ncovariance matrix(%d)=   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,idx,sxy,syy,syz,sxz,syz,szz);
            // }

            //固有値計算
            float eigen_vector[3 * 3];
            EigenJacobiMethod(a, eigen_vector, 3);

            // __syncthreads();
            // if(neighbor_size<3){
            //     printf("               %f ,%f ,%f \neigen_value=   %f ,%f ,%f \n               %f ,%f ,%f \n\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7],a[8]);
            // }

            float eigen_value[3];
            eigen_value[0]=a[0];
            eigen_value[1]=a[4];
            eigen_value[2]=a[8];

            int min_eigen_axis=0;
            float min_eigen_value=eigen_value[0];
            float eigen_sum=0;
            for(int i=1;i<3;i++){//x,y,z
                if(eigen_value[i]<min_eigen_value){
                    min_eigen_value=eigen_value[i];
                    min_eigen_axis=i;
                }
                //正規化用にnorm計算しておく
                eigen_sum += eigen_value[i];
            }

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_stop));
            eigen_time[idx]=eigen_stop - eigen_start;

            normals[idx*3+0]=eigen_vector[min_eigen_axis+0];
            normals[idx*3+1]=eigen_vector[min_eigen_axis+3];
            normals[idx*3+2]=eigen_vector[min_eigen_axis+6];

            curvatures[idx]=min_eigen_value/eigen_sum;

            // if(idx==output_id){
            //     printf("normals(%d) = %f, %f, %f\n\n\n\n",idx,normals[idx*3+0],normals[idx*3+1],normals[idx*3+2]);
            //     printf("curvature(%d) = %f\n",idx,curvatures[idx]);
            // }

            //デバッグ用
            // normals[idx*3+0]=idx*10+0;
            // normals[idx*3+1]=idx*10+1;
            // normals[idx*3+2]=idx*10+2;
            // printf("normal_x = %f ,normal_y = %f ,normal_z = %f \n", normals[idx*3+1],normals[idx*3+2],normals[idx*3+3]);
        }
        else{
            normals[idx*3+0]=0;
            normals[idx*3+1]=0;
            normals[idx*3+2]=0;
            curvatures[idx]=0;
            covariance_time[idx]=0;
            eigen_time[idx]=0;
        }
    }
    
}

extern void ComputeNormals(std::vector<std::vector<float>> points_array,std::vector<std::vector<int>> neighbor_points_indices,std::vector<int> neighbor_start_indices,int neighbor_points_count,std::vector<std::vector<float>>& normals_array,std::vector<float>& curvatures_array,std::vector<long long int>& covariance_compute_time,std::vector<long long int>& eigen_compute_time){
	std::vector<std::vector<float>> points = {
		{1,7,0},
		{2,5,0},
		{5,2,0},
		{9,9,0},
		{11,5,0},
		{16,12,0},
		{17,2,0},
		{17,9,0},
		{19,12,0},
	};
	std::vector<int> root_indices={0,1,2,3,4,5,6,7,8};


	//nodesとroot_id初期化しなくていい？
	std::vector <node> nodes;
	nodes.resize(points.size());
	// std::cout<<"sizeof(root_indices) = "<<sizeof(root_indices)<<std::endl;
	int root_id=-1;
	int create_end = CreateTree(&root_id,nodes,points,root_indices,-1,false);
	// if(first){
	// 	if(create_end==1){
	// 		for(int i=0;i<points.size();i++){
	// 			std::cout<<"node["<<i<<"] axis = "<<nodes[i].axis<<", parent_id = "<<nodes[i].parent_id<<", left_id = "<<nodes[i].left_id<<", right_id = "<<nodes[i].right_id<<std::endl;
	// 		}
	// 	}
	// } 

	// std::vector<float> search_point={8,1,0};
	std::vector<float> search_point={11,5,0};
	std::vector<int> range_indices;
	float range_sq = 3*3;
	//探索関数の実行
	int range_search = SearchSubTree(range_indices,root_id,nodes,points,search_point,range_sq);
	// std::cout<<"range_indices.size()"<<range_indices.size()<<std::endl;
	if(first){
		if(range_search==1) {
			std::cout<<"host range_indices is [";
			for(int i=0;i<range_indices.size();i++){
				std::cout<<range_indices[i]<<",";
			}
			std::cout<<"]"<<std::endl;
		}
	}


	// std::cout<<"3.01"<<std::endl;
    //ホスト1次配列宣言
	//kd
	std::vector<float> h_test_points(points.size() * 3);
	std::vector<int> h_parent_ids(points.size());
	std::vector<int> h_left_ids(points.size());
	std::vector<int> h_right_ids(points.size());
	std::vector<int> h_axes(points.size());

	//normal
    std::vector<float> h_points(points_array.size() * 3);
    std::vector<int> h_neighbor_points_indices(neighbor_points_count);
    std::vector<float> h_normals(points_array.size() * 3);
    std::vector<float> h_curvatures(points_array.size());
    std::vector<long long int> h_covariance_compute_time(points_array.size());
    std::vector<long long int> h_eigen_compute_time(points_array.size());
    // std::cout<<"3.02"<<std::endl;
    //デバイス1次配列宣言
	//kd
	float *d_test_points;
	int *d_parent_ids,*d_left_ids,*d_right_ids,*d_axes;
	//normal
    float *d_points,*d_normals,*d_curvatures;
    int *d_neighbor_points_indices,*d_neighbor_start_indices;
    long long int *d_covariance_compute_time,*d_eigen_compute_time;


    // std::cout<<"3.03"<<std::endl;
    //メモリ確保
	hipMalloc((void **)&d_test_points, points.size() * 3 * sizeof(float));
	hipMalloc((void **)&d_parent_ids, points.size() * sizeof(int));
	hipMalloc((void **)&d_left_ids, points.size() * sizeof(int));
	hipMalloc((void **)&d_right_ids, points.size() * sizeof(int));
	hipMalloc((void **)&d_axes, points.size() * sizeof(int));

    hipMalloc((void **)&d_points, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_neighbor_points_indices, neighbor_points_count * sizeof(int));
    hipMalloc((void **)&d_neighbor_start_indices, points_array.size() * sizeof(int));
    hipMalloc((void **)&d_normals, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_curvatures, points_array.size() * sizeof(float));
    hipMalloc((void **)&d_covariance_compute_time, points_array.size() * sizeof(long long int));
    hipMalloc((void **)&d_eigen_compute_time, points_array.size() * sizeof(long long int));
    // std::cout<<"3.04"<<std::endl;
    //1次配列化
    int k=0,l=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            h_points[k]=points_array[i][j];
            k++;
        }
        for(int j=0;j<neighbor_points_indices[i].size();j++){//近傍
            h_neighbor_points_indices[l]=neighbor_points_indices[i][j];
            l++;
        }
    }
	k=0;
	for(int i=0;i<points.size();i++){
		for(int j=0;j<3;j++){
			h_test_points[k]=points[i][j];
			k++;
		}
		h_parent_ids[i]=nodes[i].parent_id;
		h_left_ids[i]=nodes[i].left_id;
		h_right_ids[i]=nodes[i].right_id;
		h_axes[i]=nodes[i].axis;
	}


    // std::cout<<"3.05"<<std::endl;
    //コピー
	hipMemcpy(d_test_points, &h_test_points[0], points.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_parent_ids, &h_parent_ids[0], points.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_left_ids, &h_left_ids[0], points.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_right_ids, &h_right_ids[0], points.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_axes, &h_axes[0], points.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_points, &h_points[0], points_array.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_points_indices, &h_neighbor_points_indices[0], neighbor_points_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_start_indices, &neighbor_start_indices[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
    // std::cout<<"3.06"<<std::endl;
    //並列スレッド設定
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((points_array.size() + block.x - 1) / block.x, 1);
    // std::cout<<"3.07"<<std::endl;
    // std::cout<<"normalsGPUstart"<<std::endl;
    hipDeviceSetLimit(hipLimitStackSize, 1024*8);
    //実行
    NormalsGPU<<<grid,block>>>(d_test_points,d_parent_ids,d_left_ids,d_right_ids,d_axes,root_id,points.size(),d_points,points_array.size(),d_neighbor_points_indices,d_neighbor_start_indices,neighbor_points_count,d_normals,d_curvatures,d_covariance_compute_time,d_eigen_compute_time);
    // std::cout<<"normalsGPUend"<<std::endl;
    // std::cout<<"3.08"<<std::endl;
    //コピー
    hipMemcpy(&h_normals[0], d_normals, points_array.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_curvatures[0], d_curvatures, points_array.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_covariance_compute_time[0], d_covariance_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_compute_time[0], d_eigen_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    // std::cout<<"3.09"<<std::endl;
    //2次配列化
    k=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            normals_array[i][j]=h_normals[k];
            k++;
        }
        curvatures_array[i]=h_curvatures[i];
        covariance_compute_time[i]=h_covariance_compute_time[i];
        eigen_compute_time[i]=h_eigen_compute_time[i];
    }
    // std::cout<<"cu_normals : "<<normals_array[0][0]<<","<<normals_array[0][1]<<","<<normals_array[0][2]<<std::endl;
    // std::cout<<"3.10"<<std::endl;
    //メモリ解放
	//kd
	hipFree(d_test_points);
	hipFree(d_parent_ids);
	hipFree(d_left_ids);
	hipFree(d_right_ids);
	hipFree(d_axes);
	//normal
    hipFree(d_points);
    hipFree(d_neighbor_points_indices);
    hipFree(d_neighbor_start_indices);
    hipFree(d_normals);
    hipFree(d_curvatures);
    hipFree(d_covariance_compute_time);
    hipFree(d_eigen_compute_time);
}