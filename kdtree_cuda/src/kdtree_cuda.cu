#include "hip/hip_runtime.h"


#include <string.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kdtree_cuda/kdtree_cuda.hpp"
#include <vector>
#include <iterator>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int sort_axis=0;
int frames=0;
typedef struct
{
	int	id;
	float pos[3];
} point_with_id;

typedef struct
{
	int	parent_id;
	int left_id;
	int right_id;
	int axis;
} node;

typedef struct
{
	int i;
	bool ready;
} int_with_ready;

typedef struct
{
	bool ready;
	bool node_is_right;
	int parent_id;
	int left_id;
	int right_id;
	int depth;
	int axis;
	int middle;
	int group_size;
	int *x_sort_ids;
	int *y_sort_ids;
	int *z_sort_ids;
} detailed_node;

bool first=true;

//	年齢(昇順)
__host__ int AxisSort(const void * n1, const void * n2)
{
	if (((point_with_id *)n1)->pos[sort_axis] > ((point_with_id *)n2)->pos[sort_axis])
	{
		return 1;
	}
	else if (((point_with_id *)n1)->pos[sort_axis] < ((point_with_id *)n2)->pos[sort_axis])
	{
		return -1;
	}
	else
	{
		return 0;
	}
}

__host__ int CreateTree(int* root_id,std::vector <node>& nodes, std::vector<std::vector<float>> points,std::vector<int> group_indices,int parent_id,bool node_is_right)
{
	//入力データ初期化
	int group_size = group_indices.size();
	// std::cout<<"group_size"<<group_size<<std::endl;
	point_with_id point_with_ids[group_size];
	std::vector<std::vector<int>> axis_sort_ids(3, std::vector<int>(group_size));
	// std::cout<<"oppai 1 "<<std::endl;
	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////3*points
		point_with_ids[i].id=group_indices[i];
		point_with_ids[i].pos[0]=points[group_indices[i]][0];
		point_with_ids[i].pos[1]=points[group_indices[i]][1];
		point_with_ids[i].pos[2]=points[group_indices[i]][2];
	}
	// std::cout<<"oppai 2 "<<std::endl;

	//ソート
	float max[3],min[3],median[3],length[3];
	int axis_median_id[3];
	int median_id;
	for(sort_axis=0; sort_axis<3; sort_axis++){//x,y,zそれぞれにソート
		// std::cout<<"sort_axis = "<<sort_axis<<std::endl;
		qsort(point_with_ids, group_size, sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < group_size ; i++){///////////////////////////////////////////////////////////////////////////////////////////3*points
			axis_sort_ids[sort_axis][i]=point_with_ids[i].id;
			// printf("%d, %f, %f, %f \n", point_with_ids[i].id, point_with_ids[i].pos[0], point_with_ids[i].pos[1], point_with_ids[i].pos[2]);
		}
		// std::cout<<std::endl;
		//max,min,median,axis_median_id取得
		max[sort_axis]=point_with_ids[group_size-1].pos[sort_axis];//minとmaxいらんかも
		min[sort_axis]=point_with_ids[0].pos[sort_axis];
		length[sort_axis]=max[sort_axis]-min[sort_axis];
		median[sort_axis]=point_with_ids[(group_size-1)/2].pos[sort_axis];//偶数なら小さい方(-1)消せば大きい方
		axis_median_id[sort_axis]=point_with_ids[(group_size-1)/2].id;
	}
	// std::cout<<"oppai 3 "<<std::endl;
	// std::cout<<"x_length = "<< length[0] <<", x_median["<<axis_median_id[0]<<"] = "<<median[0]<<std::endl;
	// std::cout<<"y_length = "<< length[1] <<", y_median["<<axis_median_id[1]<<"] = "<<median[1]<<std::endl;
	// std::cout<<"z_length = "<< length[2] <<", z_median["<<axis_median_id[2]<<"] = "<<median[2]<<std::endl;
	// std::cout<<std::endl;
	// for(int i=0;i<group_size;i++){
	// 	std::cout<<"point_id["<<i<<"] = "<<point_with_ids[i].id<<std::endl;
	// }
	

	//中央値id設定、長軸設定
	if(length[0]>=length[1]&&length[0]>=length[2]){
		median_id=axis_median_id[0];
		nodes[median_id].axis=0;
	}
	if(length[1]>=length[0]&&length[1]>=length[2]){
		median_id=axis_median_id[1];
		nodes[median_id].axis=1;
	}
	if(length[2]>=length[0]&&length[2]>=length[1]){
		median_id=axis_median_id[2];
		nodes[median_id].axis=2;
	}
	// std::cout<<"oppai 4 "<<std::endl;

	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////points
		group_indices[i]=axis_sort_ids[nodes[median_id].axis][i];
	}
	// std::cout<<"oppai 5 "<<std::endl;
	// memcpy(&group_indices[0], axis_sort_ids[nodes[median_id].axis], group_size*sizeof(int));
	// std::vector<int> group_indices2;
	// group_indices.resize(0);
	// copy(axis_sort_ids[nodes[median_id].axis].begin(), axis_sort_ids[nodes[median_id].axis].end(), back_inserter(group_indices) );



	//node初期化
	nodes[median_id].left_id=-1;
	nodes[median_id].right_id=-1;

	//親設定、親の左右設定
	nodes[median_id].parent_id=parent_id;
	if(parent_id>=0){//親あり
		if(!node_is_right) nodes[parent_id].left_id=median_id;
		if(node_is_right) nodes[parent_id].right_id=median_id;
	}
	else{//親なし
		*root_id=median_id;
	}
	// std::cout<<"oppai 6 "<<std::endl;

	// std::vector<int> right_group(group_size);
	// std::vector<int> left_group(group_size);
	// int right_count=0;
	// int left_count=0;

	// for(int i=0;i<=((group_size-1)/2)-1;i++){////////////////////////////////////////////////////////////////////////////////////points
	// 	left_group[left_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	left_count++;
	// }
	// left_group.resize(left_count);///////////////////////////////////////////////////////////////////////////////////////////////points
	// for(int i=((group_size-1)/2)+1;i<group_size;i++){
	// 	right_group[right_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	right_count++;
	// }
	// right_group.resize(right_count);

	// std::vector<int> v0{1,2,3,4};
	// std::vector<int> v9(v0.begin(),v0.end());  // [1,2,3,4]
	// std::vector<int> v10(v0.begin(),v0.end());  

	// group_indices.resize(8);
	// for(int i=0;i<8;i++){
	// 	group_indices[i]=i;
	// }
	// size_t middle = ((8-1)/2);

	size_t middle = ((group_size-1)/2);
	std::vector<int>::iterator middleIter(group_indices.begin());
	std::advance(middleIter, middle);
	// std::cout<<"advance end"<<std::endl;

	std::vector<int> left_group(group_indices.begin(), middleIter);
	++middleIter;
	std::vector<int> right_group(middleIter, group_indices.end());
	// std::cout<<"oppai 7 "<<std::endl;
	// std::cout<<"group end"<<std::endl;
	// std::cout<<"left group is [";
	// for(int i=0;i<left_group.size();i++){
	// 	if(i<10) std::cout<<left_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	// std::cout<<"right group is [";
	// for(int i=0;i<right_group.size();i++){
	// 	if(i<10) std::cout<<right_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	//


	// std::cout<<"median_id"<<median_id<<std::endl;
	// std::cout<<"parent_id"<<parent_id<<std::endl;
	// std::cout<<"left_id"<<nodes[median_id].left_id<<std::endl;
	// std::cout<<"right_id"<<nodes[median_id].right_id<<std::endl;
	// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;

	//right,left group表示
	// std::cout<<"left_group is (";
	// for(int i=0;i<left_group.size();i++){
	// 	std::cout<<left_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<"right_group is (";
	// for(int i=0;i<right_group.size();i++){
	// 	std::cout<<right_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<"--------------------------------------------------------------------------------"<<std::endl;
	bool left=false;
	bool right=false;
	if(group_size>1){//子がいる
		if(left_group.size()>0){//左に子がいる
			left= CreateTree(root_id,nodes,points,left_group,median_id,false);
		}
		else left=true;
		if(right_group.size()>0){//右に子がいる
			right= CreateTree(root_id,nodes,points,right_group,median_id,true);
		}
		else right=true;
		// std::cout<<"oppai 8 "<<std::endl;
		if(right&&left) return 1;
	}
	else return 1;//子がいない
}

__host__ int CreateNode(int* root_id,int point_size,std::vector <node>& nodes, std::vector<std::vector<int>> axis_sort_ids,int depth,int parent_id,bool node_is_right)
{

	// std::cout << "oppai 0" << std::endl;
	// std::cout << std::endl;
	int group_size = axis_sort_ids[0].size();
	int axis = depth % 3;
	size_t middle = ((group_size-1)/2);
	int median_id = axis_sort_ids[axis][middle];
	nodes[median_id].axis = axis;
	nodes[median_id].parent_id = parent_id;
	nodes[median_id].left_id = -1;
	nodes[median_id].right_id = -1;
	if(parent_id >= 0){ // 親あり
		if(!node_is_right) nodes[parent_id].left_id = median_id;
		if(node_is_right) nodes[parent_id].right_id = median_id;
	}
	else{ // 親なし
		*root_id = median_id;
	}
	// std::cout << "oppai 4" << std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout << "axis_sort_ids ="<<std::endl;
	// for(int j = 0; j < 3; j++){
	// 	if(j==0) std::cout << "x =";
	// 	if(j==1) std::cout << "y =";
	// 	if(j==2) std::cout << "z =";
	// 	for(int i = 0; i < group_size; i++){
	// 		std::cout << axis_sort_ids[j][i] << ",";
	// 	}
	// 	std::cout<<std::endl;
	// }

	if(group_size > 1){ // 子あり
		std::vector<int>::iterator middle_iter(axis_sort_ids[axis].begin());
		std::advance(middle_iter,middle);
		std::vector<int> left_group(axis_sort_ids[axis].begin(),middle_iter);
		++middle_iter;
		std::vector<int> right_group(middle_iter,axis_sort_ids[axis].end());

		// std::cout<<"median_id"<<median_id<<std::endl;
		// std::cout<<"middle"<<middle<<std::endl;
		// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;
		// std::cout<<"group is (";
		// for(int i=0;i<group_size;i++){
		// 	std::cout<<axis_sort_ids[axis][i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"left_group is (";
		// for(int i=0;i<left_group.size();i++){
		// 	std::cout<<left_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"right_group is (";
		// for(int i=0;i<right_group.size();i++){
		// 	std::cout<<right_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;

		// std::cout << "oppai 1" << std::endl;

		std::vector<std::vector<int>> left_axis_sort_ids(3,std::vector<int>(left_group.size()));
		std::vector<std::vector<int>> right_axis_sort_ids(3,std::vector<int>(right_group.size()));

		std::vector<int> next_group(point_size,0);/////////////これどうにかしたい
		std::vector<int> left_axis_count(3,0);
		std::vector<int> right_axis_count(3,0);
		// std::cout << "oppai 1.5" << std::endl;
		// std::cout << "next_group.size()" << next_group.size() <<std::endl;
		// std::cout << "left_group.size()" << left_group.size() <<std::endl;
		// std::cout << "right_group.size()" << right_group.size() <<std::endl;
		for(int i = 0; i < left_group.size(); i++){
			// std::cout << "oppai 1.51" << std::endl;
			left_axis_sort_ids[axis][i] = left_group[i];
			// std::cout << "oppai 1.52" << std::endl;
			// std::cout << "left_group[i]" << left_group[i] <<std::endl;
			next_group[left_group[i]] = -1;//これで死んでそう//left_group[i]がnext_groupのレンジを超えている//この式の参照indexおかしい//1段目では正しく作用
		}
		// std::cout << "oppai 1.6" << std::endl;
		for(int i = 0; i < right_group.size(); i++){
			right_axis_sort_ids[axis][i] = right_group[i];
			// std::cout << "right_group[i]" << right_group[i] <<std::endl;
			next_group[right_group[i]] = 1;
		}
		// std::cout << "oppai 2" << std::endl;
		for(int i = 0; i < group_size; i++){
			for(int j = 0; j < 3; j++){
				if(j==axis) continue;
				if(next_group[axis_sort_ids[j][i]] == -1){
					left_axis_sort_ids[j][left_axis_count[j]] = axis_sort_ids[j][i];
					left_axis_count[j]++;
					// std::cout << "left_axis_count["<<j<<"] = "<<left_axis_count[j]<<std::endl;
				}
				else if(next_group[axis_sort_ids[j][i]] == 1){
					right_axis_sort_ids[j][right_axis_count[j]] = axis_sort_ids[j][i];
					right_axis_count[j]++;
					// std::cout << "right_axis_count["<<j<<"] = "<<right_axis_count[j]<<std::endl;
				}
			}
		}

		bool left = false;
		bool right = false;
		if(left_group.size() > 0) left = CreateNode(root_id,point_size,nodes,left_axis_sort_ids,depth+1,median_id,false);
		else left = true;

		if(right_group.size() > 0) right = CreateNode(root_id,point_size,nodes,right_axis_sort_ids,depth+1,median_id,true);
		else right = true;

		if(right&&left) return 1;
	}
	else return 1;
}

__global__ void d_CreateNode(int point_size,int group_size,int depth,int parent_id,bool node_is_right,int *x_sort_ids,int *y_sort_ids,int *z_sort_ids,int *root_id,node* nodes)
{

	// printf("create node open\n");
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
	printf("idx = %d, ",idx);
	// printf("group size = %d\n",group_size);
	int axis = depth % 3;
	size_t middle = ((group_size-1)/2);
	int median_id;
	if(axis==0) median_id = x_sort_ids[middle];
	if(axis==1) median_id = y_sort_ids[middle];
	if(axis==2) median_id = z_sort_ids[middle];

	// if(median_id==1||median_id==13||median_id==14||median_id==15||median_id==19||median_id==36||median_id==45){//ここでは正しい
	// 	printf("1 x_sort_ids[] = ");
	// 	for(int i=0;i<group_size;i++){
	// 		printf("%d,",x_sort_ids[i]);
	// 	}
	// 	printf("\n");
	// }

	int *copy_x_sort_ids,*copy_y_sort_ids,*copy_z_sort_ids;
	copy_x_sort_ids = (int *)malloc(group_size * sizeof(int));
	copy_y_sort_ids = (int *)malloc(group_size * sizeof(int));
	copy_z_sort_ids = (int *)malloc(group_size * sizeof(int));
	memcpy(copy_x_sort_ids, x_sort_ids, group_size * sizeof(int));
	memcpy(copy_y_sort_ids, y_sort_ids, group_size * sizeof(int));
	memcpy(copy_z_sort_ids, z_sort_ids, group_size * sizeof(int));

	nodes[median_id].axis = axis;
	nodes[median_id].parent_id = parent_id;
	nodes[median_id].left_id = -1;
	nodes[median_id].right_id = -1;
	// if(median_id==1||median_id==13||median_id==14||median_id==15||median_id==19||median_id==36||median_id==45){//ここでは正しい
	// 	printf("2 copy_x_sort_ids[] = ");
	// 	for(int i=0;i<group_size;i++){
	// 		printf("%d,",copy_x_sort_ids[i]);
	// 	}
	// 	printf("\n");
	// }

	// printf("1");
	if(parent_id >= 0){ // 親あり
		free(x_sort_ids);
		free(y_sort_ids);
		free(z_sort_ids);
		if(!node_is_right) nodes[parent_id].left_id = median_id;
		if(node_is_right) nodes[parent_id].right_id = median_id;
	}
	else{ // 親なし
		printf("root update\n");
		*root_id = median_id;
	}
	// printf("2");
	if(group_size > 1){ // 子あり
		int left_group_size = 0;
		int right_group_size = 0;
		int *left_x_sort_ids,*left_y_sort_ids,*left_z_sort_ids;
		int *right_x_sort_ids,*right_y_sort_ids,*right_z_sort_ids;
		int *next_group;
		next_group = (int *)malloc(point_size * sizeof(int));
		left_x_sort_ids = (int *)malloc(middle * sizeof(int));
		left_y_sort_ids = (int *)malloc(middle * sizeof(int));
		left_z_sort_ids = (int *)malloc(middle * sizeof(int));
		right_x_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		right_y_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		right_z_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		int left_axis_count[3]={0,0,0};
		int right_axis_count[3]={0,0,0};
		// printf("\n\n\n");
		// printf("median_id = %d\n",median_id);
		// if(!node_is_right) printf("node is left\n");
		// else printf("node is right\n");
		// printf("parent_id = %d\n",parent_id);
		// printf("middle = %d\n",middle);
		// printf("axis = %d\n",nodes[median_id].axis);

		// printf("3");
		// printf("axis = %d",axis);
		// if(median_id==1||median_id==14||median_id==19||median_id==36||median_id==45){//ここでは正しい
		// 	printf("3 copy_x_sort_ids[] = ");
		// 	for(int i=0;i<group_size;i++){
		// 		printf("%d,",copy_x_sort_ids[i]);
		// 	}
		// 	printf("\n");
		// }


		if(axis==0){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_x_sort_ids[left_axis_count[0]] = copy_x_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[0]++;
					// printf("3.02 ");
					// printf("parent_id = %d",parent_id);
					// printf("copy_x_sort_ids[%d] = %d ",i,copy_x_sort_ids[i]);
					next_group[copy_x_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_x_sort_ids[i],next_group[copy_x_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_x_sort_ids[right_axis_count[0]] = copy_x_sort_ids[i];
					right_axis_count[0]++;
					next_group[copy_x_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_x_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}
		}
		else if(axis==1){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_y_sort_ids[left_axis_count[1]] = copy_y_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[1]++;
					// printf("3.02 ");
					// printf("copy_y_sort_ids[%d] = %d ",i,copy_y_sort_ids[i]);
					next_group[copy_y_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_y_sort_ids[i],next_group[copy_y_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_y_sort_ids[right_axis_count[1]] = copy_y_sort_ids[i];
					right_axis_count[1]++;
					next_group[copy_y_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_y_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}

		}
		else if(axis==2){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_z_sort_ids[left_axis_count[2]] = copy_z_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[2]++;
					// printf("3.02 ");
					// printf("copy_z_sort_ids[%d] = %d ",i,copy_z_sort_ids[i]);
					next_group[copy_z_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_z_sort_ids[i],next_group[copy_z_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_z_sort_ids[right_axis_count[2]] = copy_z_sort_ids[i];
					right_axis_count[2]++;
					next_group[copy_z_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_z_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}
		}
		// printf("\n");
		// printf("4");
		left_group_size = left_axis_count[axis];
		right_group_size = right_axis_count[axis];

		// printf("group is (");
		// for(int i=0;i<group_size;i++){
		// 	if(axis==0) printf("%d,",copy_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",copy_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",copy_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("left_group is (");
		// for(int i=0;i<left_group_size;i++){
		// 	if(axis==0) printf("%d,",left_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",left_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",left_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("right_group is (");
		// for(int i=0;i<right_group_size;i++){
		// 	if(axis==0) printf("%d,",right_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",right_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",right_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("\n\n\n");

		for(int i = 0; i < group_size; i++){
			for(int j = 0; j < 3; j++){
				if(j==axis) continue;
				if(j==0){//x実装
					if(next_group[copy_x_sort_ids[i]] == -1){
						left_x_sort_ids[left_axis_count[j]] = copy_x_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_x_sort_ids[i]] == 1){
						right_x_sort_ids[right_axis_count[j]] = copy_x_sort_ids[i];
						right_axis_count[j]++;
					}
				}
				if(j==1){//y実装
					if(next_group[copy_y_sort_ids[i]] == -1){
						left_y_sort_ids[left_axis_count[j]] = copy_y_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_y_sort_ids[i]] == 1){
						right_y_sort_ids[right_axis_count[j]] = copy_y_sort_ids[i];
						right_axis_count[j]++;
					}
				}
				if(j==2){//z実装
					if(next_group[copy_z_sort_ids[i]] == -1){
						left_z_sort_ids[left_axis_count[j]] = copy_z_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_z_sort_ids[i]] == 1){
						right_z_sort_ids[right_axis_count[j]] = copy_z_sort_ids[i];
						right_axis_count[j]++;
					}
				}
			}
		}
		free(copy_x_sort_ids);
		free(copy_y_sort_ids);
		free(copy_z_sort_ids);
		free(next_group);
		// printf("5");
		// if(median_id==18) printf("Hit!!!!!!!!!!!!!!! left_x_sort_ids[16] = %d\n\n\n",left_x_sort_ids[16]);
		// if(median_id==33) printf("Hit!!!!!!!!!!!!!!! right_x_sort_ids[16] = %d\n\n\n",right_x_sort_ids[16]);
		// if(median_id==3145) printf("Hit!!!!!!!!!!!!!!! left_x_sort_ids[156] = %d\n\n\n",left_x_sort_ids[156]);
		// if(median_id==3877) printf("Hit!!!!!!!!!!!!!!! left_y_sort_ids[1] = %d\n\n\n",left_x_sort_ids[1]);
		// if(median_id==3888) printf("Hit!!!!!!!!!!!!!!! left_y_sort_ids[1] = %d\n\n\n",left_y_sort_ids[1]);
		// hipDeviceSynchronize();
		if(left_group_size > 0) d_CreateNode<<<1, 1>>>(point_size,left_group_size,depth+1,median_id,false,left_x_sort_ids,left_y_sort_ids,left_z_sort_ids,root_id,nodes);
		else {
			free(left_x_sort_ids);
			free(left_y_sort_ids);
			free(left_z_sort_ids);
		}
		if(right_group_size > 0) d_CreateNode<<<1, 1>>>(point_size,right_group_size,depth+1,median_id,true,right_x_sort_ids,right_y_sort_ids,right_z_sort_ids,root_id,nodes);
		else {
			free(right_x_sort_ids);
			free(right_y_sort_ids);
			free(right_z_sort_ids);
		}
		// hipDeviceSynchronize();
		// free(left_x_sort_ids);
		// free(left_y_sort_ids);
		// free(left_z_sort_ids);
		// free(right_x_sort_ids);
		// free(right_y_sort_ids);
		// free(right_z_sort_ids);
		
	}
}

__global__ void d_DepthCreateNode(int point_size,detailed_node* nodes,int* end_list)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
	// printf("idx = %d\n",idx);
	if(idx < point_size){//計算領域
		// printf("idx = %d\n",idx);
		if(nodes[idx].ready&&(0 > end_list[idx])){//該当ノード
			// printf("\n\n\n");
			// printf("\nmedian_id = %d\n",idx);
			// if(!nodes[idx].node_is_right) printf("node is left\n");
			// else printf("node is right\n");
			// printf("parent_id = %d\n",nodes[idx].parent_id);
			// printf("middle = %d\n",nodes[idx].middle);
			// printf("axis = %d\n",nodes[idx].axis);
			// if(nodes[idx].node_is_right) printf("device depth = %d\n",nodes[idx].depth);//間引きのため右のみ
			// printf("0");
			nodes[idx].left_id = -1;
			nodes[idx].right_id = -1;
			if(nodes[idx].group_size>1){//子あり
				// printf("\nmedian_id = %d\n",idx);
				int left_group_size = 0;
				int right_group_size = 0;
				int *left_x_sort_ids,*left_y_sort_ids,*left_z_sort_ids;
				int *right_x_sort_ids,*right_y_sort_ids,*right_z_sort_ids;
				int *next_group;
				next_group = (int *)malloc(point_size * sizeof(int));
				left_x_sort_ids = (int *)malloc(nodes[idx].middle * sizeof(int));
				left_y_sort_ids = (int *)malloc(nodes[idx].middle * sizeof(int));
				left_z_sort_ids = (int *)malloc(nodes[idx].middle * sizeof(int));
				right_x_sort_ids = (int *)malloc((nodes[idx].group_size - (nodes[idx].middle + 1)) * sizeof(int));
				right_y_sort_ids = (int *)malloc((nodes[idx].group_size - (nodes[idx].middle + 1)) * sizeof(int));
				right_z_sort_ids = (int *)malloc((nodes[idx].group_size - (nodes[idx].middle + 1)) * sizeof(int));
				int left_axis_count[3]={0,0,0};
				int right_axis_count[3]={0,0,0};

				if(nodes[idx].axis==0){
					for(int i = 0; i < nodes[idx].group_size; i++){
						if(point_size<nodes[idx].x_sort_ids[i]) printf("out of range nodes[idx].x_sort_ids[%d] = %d \n",i,nodes[idx].x_sort_ids[i]);
						if(point_size<nodes[idx].y_sort_ids[i]) printf("out of range nodes[idx].y_sort_ids[%d] = %d \n",i,nodes[idx].y_sort_ids[i]);
						if(point_size<nodes[idx].z_sort_ids[i]) printf("out of range nodes[idx].z_sort_ids[%d] = %d \n",i,nodes[idx].z_sort_ids[i]);
						if(i<nodes[idx].middle){
							left_x_sort_ids[left_axis_count[0]] = nodes[idx].x_sort_ids[i];
							left_axis_count[0]++;
							next_group[nodes[idx].x_sort_ids[i]] = -1;
						}
						else if(i>nodes[idx].middle){
							right_x_sort_ids[right_axis_count[0]] = nodes[idx].x_sort_ids[i];
							right_axis_count[0]++;
							next_group[nodes[idx].x_sort_ids[i]] = 1;
						}
						else{
							next_group[nodes[idx].x_sort_ids[i]] = 0;
						}
					}
				}
				else if(nodes[idx].axis==1){
					for(int i = 0; i < nodes[idx].group_size; i++){
						if(point_size<nodes[idx].x_sort_ids[i]) printf("out of range nodes[idx].x_sort_ids[%d] = %d \n",i,nodes[idx].x_sort_ids[i]);
						if(point_size<nodes[idx].y_sort_ids[i]) printf("out of range nodes[idx].y_sort_ids[%d] = %d \n",i,nodes[idx].y_sort_ids[i]);
						if(point_size<nodes[idx].z_sort_ids[i]) printf("out of range nodes[idx].z_sort_ids[%d] = %d \n",i,nodes[idx].z_sort_ids[i]);
						if(i<nodes[idx].middle){
							left_y_sort_ids[left_axis_count[1]] = nodes[idx].y_sort_ids[i];
							left_axis_count[1]++;
							next_group[nodes[idx].y_sort_ids[i]] = -1;
						}
						else if(i>nodes[idx].middle){
							right_y_sort_ids[right_axis_count[1]] = nodes[idx].y_sort_ids[i];
							right_axis_count[1]++;
							next_group[nodes[idx].y_sort_ids[i]] = 1;
						}
						else{
							next_group[nodes[idx].y_sort_ids[i]] = 0;
						}
					}

				}
				else if(nodes[idx].axis==2){
					for(int i = 0; i < nodes[idx].group_size; i++){
						if(point_size<nodes[idx].x_sort_ids[i]) printf("out of range nodes[idx].x_sort_ids[%d] = %d \n",i,nodes[idx].x_sort_ids[i]);
						if(point_size<nodes[idx].y_sort_ids[i]) printf("out of range nodes[idx].y_sort_ids[%d] = %d \n",i,nodes[idx].y_sort_ids[i]);
						if(point_size<nodes[idx].z_sort_ids[i]) printf("out of range nodes[idx].z_sort_ids[%d] = %d \n",i,nodes[idx].z_sort_ids[i]);
						if(i<nodes[idx].middle){
							left_z_sort_ids[left_axis_count[2]] = nodes[idx].z_sort_ids[i];
							left_axis_count[2]++;
							next_group[nodes[idx].z_sort_ids[i]] = -1;
						}
						else if(i>nodes[idx].middle){
							right_z_sort_ids[right_axis_count[2]] = nodes[idx].z_sort_ids[i];
							right_axis_count[2]++;
							next_group[nodes[idx].z_sort_ids[i]] = 1;
						}
						else{
							next_group[nodes[idx].z_sort_ids[i]] = 0;
						}
					}
				}
				hipDeviceSynchronize();
				// printf("depth = %d\n",nodes[idx].depth);
				// printf("median_id = %d \n",idx);
				hipDeviceSynchronize();
				// printf("1 ");
				left_group_size = left_axis_count[nodes[idx].axis];
				right_group_size = right_axis_count[nodes[idx].axis];

				for(int i = 0; i < nodes[idx].group_size; i++){
					for(int j = 0; j < 3; j++){
						if(j==nodes[idx].axis) continue;
						if(j==0){//x実装
							if(next_group[nodes[idx].x_sort_ids[i]] == -1){
								left_x_sort_ids[left_axis_count[j]] = nodes[idx].x_sort_ids[i];
								left_axis_count[j]++;
							}
							else if(next_group[nodes[idx].x_sort_ids[i]] == 1){
								right_x_sort_ids[right_axis_count[j]] = nodes[idx].x_sort_ids[i];
								right_axis_count[j]++;
							}
						}
						if(j==1){//y実装
							if(next_group[nodes[idx].y_sort_ids[i]] == -1){
								left_y_sort_ids[left_axis_count[j]] = nodes[idx].y_sort_ids[i];
								left_axis_count[j]++;
							}
							else if(next_group[nodes[idx].y_sort_ids[i]] == 1){
								right_y_sort_ids[right_axis_count[j]] = nodes[idx].y_sort_ids[i];
								right_axis_count[j]++;
							}
						}
						if(j==2){//z実装
							if(next_group[nodes[idx].z_sort_ids[i]] == -1){
								left_z_sort_ids[left_axis_count[j]] = nodes[idx].z_sort_ids[i];
								left_axis_count[j]++;
							}
							else if(next_group[nodes[idx].z_sort_ids[i]] == 1){
								right_z_sort_ids[right_axis_count[j]] = nodes[idx].z_sort_ids[i];
								right_axis_count[j]++;
							}
						}
					}
				}
				hipDeviceSynchronize();
				// printf("2 ");
				free(next_group);
				int next_axis = (nodes[idx].depth + 1) % 3;
				if(left_group_size > 0){
					size_t left_middle = ((left_group_size - 1) / 2);
					int left_median_id;
					if(next_axis == 0) left_median_id = left_x_sort_ids[left_middle];
					if(next_axis == 1) left_median_id = left_y_sort_ids[left_middle];
					if(next_axis == 2) left_median_id = left_z_sort_ids[left_middle];

					nodes[idx].left_id = left_median_id;

					nodes[left_median_id].ready = true;
					nodes[left_median_id].node_is_right = false;
					nodes[left_median_id].parent_id = idx;
					nodes[left_median_id].depth = nodes[idx].depth + 1;
					nodes[left_median_id].axis = next_axis;
					nodes[left_median_id].middle = left_middle;
					nodes[left_median_id].group_size = left_group_size;
					nodes[left_median_id].x_sort_ids = left_x_sort_ids;
					nodes[left_median_id].y_sort_ids = left_y_sort_ids;
					nodes[left_median_id].z_sort_ids = left_z_sort_ids;
				}
				if(right_group_size > 0){
					size_t right_middle = ((right_group_size - 1) / 2);
					int right_median_id;
					if(next_axis == 0) right_median_id = right_x_sort_ids[right_middle];
					if(next_axis == 1) right_median_id = right_y_sort_ids[right_middle];
					if(next_axis == 2) right_median_id = right_z_sort_ids[right_middle];

					nodes[idx].right_id = right_median_id;

					nodes[right_median_id].ready = true;
					nodes[right_median_id].node_is_right = true;
					nodes[right_median_id].parent_id = idx;
					nodes[right_median_id].depth = nodes[idx].depth + 1;
					nodes[right_median_id].axis = next_axis;
					nodes[right_median_id].middle = right_middle;
					nodes[right_median_id].group_size = right_group_size;
					nodes[right_median_id].x_sort_ids = right_x_sort_ids;
					nodes[right_median_id].y_sort_ids = right_y_sort_ids;
					nodes[right_median_id].z_sort_ids = right_z_sort_ids;
				}
				hipDeviceSynchronize();
				// printf("3 ");
			}
			if(nodes[idx].parent_id >= 0){//親あり
				free(nodes[idx].x_sort_ids);
				free(nodes[idx].y_sort_ids);
				free(nodes[idx].z_sort_ids);
			}
			hipDeviceSynchronize();
			// printf("4 ");
			end_list[idx] = 1;
			hipDeviceSynchronize();
			// printf("5 ");
		}
	}
}

__device__ void d_PointRangeCheckAndAdd(int *range_indices_size,int *range_indices,int head_id,float* points,float* search_point,float range_sq)
{
	float dist_sq = powf(points[head_id*3+0]-search_point[0],2)+powf(points[head_id*3+1]-search_point[1],2)+powf(points[head_id*3+2]-search_point[2],2);
	if(dist_sq<range_sq){
		// printf("device inside");
		range_indices[*range_indices_size] = head_id;
		*range_indices_size+=1;
	} 
}

__device__ int d_SearchSubTree(int *range_indices_size,int *range_indices,int root_id,node* nodes,float* points,float* search_point,float range_sq)
{
	int head_id = root_id;
	bool cross,next_is_right;
	//潜り
	while(1){
		if(search_point[nodes[head_id].axis]>points[head_id*3+nodes[head_id].axis]) next_is_right = true;
		else next_is_right = false;

		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
			if(next_is_right){//right
				head_id = nodes[head_id].right_id;
			}
			else{//left
				head_id = nodes[head_id].left_id;
			}
		}
		else break;
	}
	//rootが底
	if(head_id==root_id) {
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		return 1;
	}
	//昇り
	int last_id;
	while(1){
		cross = false;
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		//昇る
		last_id = head_id;
		head_id = nodes[head_id].parent_id;
		//交差判定　1軸のみ低効率
		float axis_diff_sq = powf(points[head_id*3+nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
		if(axis_diff_sq < range_sq) cross = true;
		int sub_tree=0;
		if(cross){
			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
		}
		if(head_id==root_id){
			d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
			break;
		}
	}

	return 1;
}

__device__ int d_SearchSubTree2(int *range_indices_size,int *range_indices,int root_id,detailed_node* nodes,float* points,float* search_point,float range_sq)
{
	int head_id = root_id;
	bool cross,next_is_right;
	//潜り
	while(1){
		if(search_point[nodes[head_id].axis]>points[head_id*3+nodes[head_id].axis]) next_is_right = true;
		else next_is_right = false;

		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
			if(next_is_right){//right
				head_id = nodes[head_id].right_id;
			}
			else{//left
				head_id = nodes[head_id].left_id;
			}
		}
		else break;
	}
	//rootが底
	if(head_id==root_id) {
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		return 1;
	}
	//昇り
	int last_id;
	while(1){
		cross = false;
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		//昇る
		last_id = head_id;
		head_id = nodes[head_id].parent_id;
		//交差判定　1軸のみ低効率
		float axis_diff_sq = powf(points[head_id*3+nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
		if(axis_diff_sq < range_sq) cross = true;
		int sub_tree=0;
		if(cross){
			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = d_SearchSubTree2(range_indices_size,range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = d_SearchSubTree2(range_indices_size,range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
		}
		if(head_id==root_id){
			d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
			break;
		}
	}

	return 1;
}

__device__ int EigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 

__global__ void NormalsGPU(/*detailed_node* detailed_nodes,*/long long int* neighbor_time,int *point_neighbor_size,int* point_neighbor,int* d_parent_ids,int* d_left_ids,int* d_right_ids,int* d_axes,int root_id,float* points,int point_size,int* neighbor_points_indices,int* neighbor_start_indices,int neighbor_points_count,float* normals,float* curvatures,long long int* covariance_time,long long int* eigen_time)
{
    // printf("normalsGPU");
    //インデックス取得
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
    unsigned int output_id=50;
    // printf("idx = %d ,", idx);
	if(idx==output_id){
		long long int neighbor_start, neighbor_stop;
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_start));
		////////////////////////////////////////////////////////////////
		node *nodes = (node*)malloc(sizeof(node) * point_size);

		for(int i=0;i<point_size;i++){
			nodes[i].parent_id=d_parent_ids[i];
			nodes[i].left_id=d_left_ids[i];
			nodes[i].right_id=d_right_ids[i];
			nodes[i].axis=d_axes[i];
		}
		float search_point[3];
		search_point[0]=points[idx*3+0];
		search_point[1]=points[idx*3+1];
		search_point[2]=points[idx*3+2];
		
		int *range_indices = (int*)malloc(sizeof(int) * point_size);
		int range_indices_size = 0;

		float range_sq = 0.15*0.15;

		//探索関数の実行
		int range_search = d_SearchSubTree(&range_indices_size,range_indices,root_id,nodes,points,search_point,range_sq);
		// int range_search = d_SearchSubTree2(&range_indices_size,range_indices,root_id,detailed_nodes,points,search_point,range_sq);
		// std::cout<<"range_indices.size()"<<range_indices.size()<<std::endl;
		if(range_search==1) {
			// printf("device range_indices size is =%d",range_indices_size);
			// printf("device range_indices is [");
			for(int i=0;i<range_indices_size;i++){
				// printf("%d,",range_indices[i]);
				point_neighbor[i]=range_indices[i];
			}
			// printf("]\n");
			point_neighbor_size[0]=range_indices_size;
			// printf("device size %d\n",range_indices_size);
		}
		
		free (nodes);
		free (range_indices);
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_stop));
		neighbor_time[idx]=neighbor_stop - neighbor_start;
		////////////////////////////////////////////////////////////////////
	}


    if(idx<point_size){//対象スレッド内のみ計算
        //デバッグ用
        // if(idx==0||idx==10||idx==20) printf("points(%d) = %f,%f,%f\n",idx,points[idx*3+0],points[idx*3+1],points[idx*3+2]);
        
        // printf("idx<point_size");
        //近傍点終点インデックスの定義
        int end_indices;
        if(idx==(point_size-1)) end_indices = neighbor_points_count;
        else end_indices = neighbor_start_indices[idx+1]-1;

        int neighbor_size=(end_indices-neighbor_start_indices[idx]+1)/3;
        // if(idx==0||idx==10||idx==20) printf("neighbor(%d) start = %d, end = %d,size = %d\n",idx,neighbor_start_indices[idx],end_indices,neighbor_size);
        // printf("neighbor_size = %d\n", neighbor_size);
        if(neighbor_size>=3){//近傍点数3以上
            long long int covariance_start, covariance_stop;
            long long int eigen_start,eigen_stop;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_start));
            // printf("neighbor_size>=3");
            //平均計算
            float x_average=0,y_average=0,z_average=0;
            // if(idx==output_id) printf("neighbor_size = %d\n",neighbor_size);

            // if(idx==output_id) printf("neighbor_points(%d) = {\n",idx);
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                // デバッグ用
                // if((idx==0||idx==10||idx==20)&&(i==(neighbor_start_indices[idx]))) printf("neighbor_points_indices(%d) = %d\n",idx,neighbor_points_indices[i]);
                // if((idx==0||idx==10||idx==20)&&i==neighbor_start_indices[idx]) printf("neighbor_points(%d) = %f,%f,%f\n",idx,points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                // if(idx==output_id) printf("{%f, %f, %f},\n",points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                x_average+=points[neighbor_points_indices[i]*3+0];
                y_average+=points[neighbor_points_indices[i]*3+1];
                z_average+=points[neighbor_points_indices[i]*3+2];
            }
            // if(idx==output_id) printf("};\n");
            x_average/=neighbor_size;
            y_average/=neighbor_size;
            z_average/=neighbor_size;

            // //要素計算
            float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                sxx+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+0]-x_average);
                syy+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                szz+=(points[neighbor_points_indices[i]*3+2]-z_average)*(points[neighbor_points_indices[i]*3+2]-z_average);

                sxy+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                sxz+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
                syz+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
            }

            sxx/=neighbor_size;
            syy/=neighbor_size;
            szz/=neighbor_size;
            sxy/=neighbor_size;
            sxz/=neighbor_size;
            syz/=neighbor_size;

            //共分散行列
            float a[3*3]={
                sxx,sxy,sxz,
                sxy,syy,syz,
                sxz,syz,szz,
            };

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_stop));
            covariance_time[idx]=covariance_stop - covariance_start;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_start));
            
            // __syncthreads();
            // if(idx==output_id){
            //     printf("                          %f ,%f ,%f \ncovariance matrix(%d)=   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,idx,sxy,syy,syz,sxz,syz,szz);
            // }

            //固有値計算
            float eigen_vector[3 * 3];
            EigenJacobiMethod(a, eigen_vector, 3);

            // __syncthreads();
            // if(neighbor_size<3){
            //     printf("               %f ,%f ,%f \neigen_value=   %f ,%f ,%f \n               %f ,%f ,%f \n\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7],a[8]);
            // }

            float eigen_value[3];
            eigen_value[0]=a[0];
            eigen_value[1]=a[4];
            eigen_value[2]=a[8];

            int min_eigen_axis=0;
            float min_eigen_value=eigen_value[0];
            float eigen_sum=0;
            for(int i=1;i<3;i++){//x,y,z
                if(eigen_value[i]<min_eigen_value){
                    min_eigen_value=eigen_value[i];
                    min_eigen_axis=i;
                }
                //正規化用にnorm計算しておく
                eigen_sum += eigen_value[i];
            }

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_stop));
            eigen_time[idx]=eigen_stop - eigen_start;

            normals[idx*3+0]=eigen_vector[min_eigen_axis+0];
            normals[idx*3+1]=eigen_vector[min_eigen_axis+3];
            normals[idx*3+2]=eigen_vector[min_eigen_axis+6];

            curvatures[idx]=min_eigen_value/eigen_sum;

            // if(idx==output_id){
            //     printf("normals(%d) = %f, %f, %f\n\n\n\n",idx,normals[idx*3+0],normals[idx*3+1],normals[idx*3+2]);
            //     printf("curvature(%d) = %f\n",idx,curvatures[idx]);
            // }

            //デバッグ用
            // normals[idx*3+0]=idx*10+0;
            // normals[idx*3+1]=idx*10+1;
            // normals[idx*3+2]=idx*10+2;
            // printf("normal_x = %f ,normal_y = %f ,normal_z = %f \n", normals[idx*3+1],normals[idx*3+2],normals[idx*3+3]);
        }
        else{
            normals[idx*3+0]=0;
            normals[idx*3+1]=0;
            normals[idx*3+2]=0;
            curvatures[idx]=0;
            covariance_time[idx]=0;
            eigen_time[idx]=0;
        }
    }
    
}

// __global__ void ChildKernel(void* data)
// {

// 	printf("child : %d, %d\n", blockIdx.x, threadIdx.x);

// }

// __global__ void ParentKernel(void* data)
// {

// 	printf("parent: %d, %d\n", blockIdx.x, threadIdx.x);

// 	ChildKernel<<<1, 2>>>(data);
// 	hipDeviceSynchronize();

// }

// __global__ void KernelFunctionArgumentTypeCheck(node* nodes)
// {
// 	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
//     unsigned int idx = ix;
// 	printf("KernelFunctionArgumentTypeCheck: %d, %d\n", blockIdx.x, threadIdx.x);
// 	for(int i=0;i<3;i++){
// 		nodes[i].parent_id = i; nodes[i].left_id = i; nodes[i].right_id = i; nodes[i].axis = i;
// 	}
// }

// __global__ void MyKernel(float* devPtr, size_t pitch, int width, int height)
// {
//   for (int r = 0; r < height; ++r) {
//     float* row = (float*)((char*)devPtr + r * pitch);
//     for (int c = 0; c < width; ++c) {
//       float element = row[c];
//     }
//   }
// }

// __global__ void d_ParallelRecursionTest(int data_size,int_with_ready* data)
// {
// 	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
//     unsigned int idx = ix;
// 	if(idx<data_size){
// 		if(data[idx].ready){
// 			if(idx!=0) data[idx].i = data[idx-1].i + 1;
// 			data[idx+1].ready = true;
// 		}
// 	}
// }

extern void ComputeNormals(std::vector<long long int>& neighbor_time,std::vector<int>& point_neighbor,std::vector<std::vector<float>> points_array,std::vector<std::vector<int>> neighbor_points_indices,std::vector<int> neighbor_start_indices,int neighbor_points_count,std::vector<std::vector<float>>& normals_array,std::vector<float>& curvatures_array,std::vector<long long int>& covariance_compute_time,std::vector<long long int>& eigen_compute_time)
{
	//minimum recursive function in cuda
	// int nbytes = 1024;
	// int *data_dev = 0;
	// hipMalloc((void**)&data_dev, nbytes);
	// hipMemset(data_dev, 255, nbytes);
	// ParentKernel<<<1, 2>>>(data_dev);

	//Kernel Function Argument Type Check
	// std::vector<node> h_nodes_test(3);
	// node *d_nodes_test;
	// hipMalloc((void **)&d_nodes_test, 3 * sizeof(node));
	// KernelFunctionArgumentTypeCheck<<<1, 1>>>(d_nodes_test);
	// hipMemcpy(&h_nodes_test[0], d_nodes_test, 3 * sizeof(node), hipMemcpyDeviceToHost);
	// hipFree(d_nodes_test);
	// for(int i=0;i<3;i++){
	// 	std::cout<<"nodes = "<<h_nodes_test[i].parent_id<<","<<h_nodes_test[i].left_id<<","<<h_nodes_test[i].right_id<<","<<h_nodes_test[i].axis<<std::endl;
	// }

	//2d確保
	// int width = 64, height = 64;
	// float* devPtr;
	// size_t pitch;
	// hipMallocPitch(&devPtr, &pitch, width * sizeof(float), height);
	// MyKernel<<<100, 512>>>(devPtr, pitch, width, height);

	//Parallel Recursion Test
	// int data_size=10;
	// std::vector<int_with_ready> h_test_data(data_size);
	// for(int i=0;i<data_size;i++){
	// 	h_test_data[i].i = 0;
	// 	h_test_data[i].ready = false;
	// }
	// h_test_data[0].i = 0;
	// h_test_data[0].ready = true;
	// int_with_ready *d_test_data;
	// hipMalloc((void **)&d_test_data, data_size * sizeof(int_with_ready));
	// hipMemcpy(d_test_data, &h_test_data[0], data_size * sizeof(int_with_ready), hipMemcpyHostToDevice);
	// int dimx_test = 32;
    // dim3 block_test(dimx_test, 1);
    // dim3 grid_test((data_size + block_test.x - 1) / block_test.x, 1);
	// //ここ繰り返す
	// for(int i=0;i<data_size;i++){
	// 	d_ParallelRecursionTest<<<grid_test, block_test>>>(data_size,d_test_data);
	// }
	// hipMemcpy(&h_test_data[0], d_test_data, data_size * sizeof(int_with_ready), hipMemcpyDeviceToHost);
	// hipFree(d_test_data);
	// for(int i=0;i<data_size;i++){
	// 	std::cout<<"data["<<i<<"] = "<<h_test_data[i].i<<std::endl;
	// }

	// points_array.clear();
	// points_array.resize(8);
	// points_array = {{6, 0, 0}, 
	// 				{5, 3, 0},
	// 				{3, 4, 0},
	// 				{4, 6, 0},
	// 				{2, 5, 0},
	// 				{1, 2, 0},
	// 				{0, 1, 0},
	// 				{-3.21161e+38,4.57384e-41,-3.21161e+38}};
	// // points_array.resize(7);

	// int test_size = 684;
	// if(points_array.size()>test_size) points_array.resize(test_size);

	clock_t build_start,build_end;
	// build_start = clock();
	// if(frames==43) std::cout<<"dead point is ("<<points_array[77][0]<<","<<points_array[77][0]<<","<<points_array[77][0]<<")"<<std::endl;

	int root_id=-1;
	/////////////////////////////////////////////////////////////////////////////////////////
	std::vector <node> nodes;
	nodes.resize(points_array.size());
	std::vector<std::vector<int>> axis_sort_ids(3,std::vector<int>(points_array.size()));
	point_with_id point_with_ids[points_array.size()];
	for(int i=0;i<points_array.size();i++){
		point_with_ids[i].id = i;
		point_with_ids[i].pos[0] = points_array[i][0];
		point_with_ids[i].pos[1] = points_array[i][1];
		point_with_ids[i].pos[2] = points_array[i][2];
	}
	for(sort_axis=0; sort_axis<3; sort_axis++){
		qsort(point_with_ids, points_array.size(), sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < points_array.size() ; i++){
			axis_sort_ids[sort_axis][i]=point_with_ids[i].id;
		}
	}
	int create_end = CreateNode(&root_id,points_array.size(),nodes,axis_sort_ids,0,-1,false);
	/////////////////////////////////////////////////////////////////////////////////////////

	/////////////////////////////////////////////////////////////////////////////////////////
	// std::vector <node> nodes;
	// nodes.resize(points_array.size());
	// std::vector<int> x_sort_ids(points_array.size());
	// std::vector<int> y_sort_ids(points_array.size());
	// std::vector<int> z_sort_ids(points_array.size());
	// point_with_id point_with_ids[points_array.size()];
	// for(int i=0;i<points_array.size();i++){
	// 	point_with_ids[i].id = i;
	// 	point_with_ids[i].pos[0] = points_array[i][0];
	// 	point_with_ids[i].pos[1] = points_array[i][1];
	// 	point_with_ids[i].pos[2] = points_array[i][2];
	// }
	// for(sort_axis=0; sort_axis<3; sort_axis++){
	// 	qsort(point_with_ids, points_array.size(), sizeof(point_with_id), AxisSort);
	// 	for (int i=0 ; i < points_array.size() ; i++){
	// 		if(sort_axis==0){
	// 			x_sort_ids[i]=point_with_ids[i].id;
	// 		}
	// 		if(sort_axis==1){
	// 			y_sort_ids[i]=point_with_ids[i].id;
	// 		}
	// 		if(sort_axis==2){
	// 			z_sort_ids[i]=point_with_ids[i].id;
	// 		}
	// 	}
	// }
	// int *d_x_sort_ids,*d_y_sort_ids,*d_z_sort_ids,*d_root_id;
	// node *d_nodes;
	// hipMalloc((void **)&d_x_sort_ids, points_array.size() * sizeof(int));
	// hipMalloc((void **)&d_y_sort_ids, points_array.size() * sizeof(int));
	// hipMalloc((void **)&d_z_sort_ids, points_array.size() * sizeof(int));
	// hipMalloc((void **)&d_root_id, sizeof(int));
	// hipMalloc((void **)&d_nodes, points_array.size() * sizeof(node));
	// hipMemcpy(d_x_sort_ids, &x_sort_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	// hipMemcpy(d_y_sort_ids, &y_sort_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	// hipMemcpy(d_z_sort_ids, &z_sort_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	// hipDeviceSetLimit(hipLimitStackSize, 1024*1024);
	// // std::cout << "frames" << frames <<"------------------------------------------------------------------------------------------------------------"<< std::endl;
	// d_CreateNode<<<1, 1>>>(points_array.size(),points_array.size(),0,-1,false,d_x_sort_ids,d_y_sort_ids,d_z_sort_ids,d_root_id,d_nodes);
	// hipMemcpy(&root_id, d_root_id, sizeof(int), hipMemcpyDeviceToHost);
	// hipMemcpy(&nodes[0], d_nodes, points_array.size() * sizeof(node), hipMemcpyDeviceToHost);
	// hipFree(d_x_sort_ids);
	// hipFree(d_y_sort_ids);
	// hipFree(d_z_sort_ids);
	// hipFree(d_root_id);
	// hipFree(d_nodes);
	/////////////////////////////////////////////////////////////////////////////////////////
	build_start = clock();
	/////////////////////////////////////////////////////////////////////////////////////////施工
	if(first){
		int test_size = 684;
		std::vector<std::vector<float>> test_points(test_size);
		test_points   = {{-0.366395,0.518837,0.94393},
						{-0.3442,0.522,0.948},
						{-0.407158,0.513263,0.990368},
						{-0.373634,0.517484,0.975457},
						{-0.326235,0.523437,0.976535},
						{-0.278208,0.529258,0.987033},
						{-0.237484,0.533419,0.996065},
						{-0.453,0.509,1.047},
						{-0.419682,0.510802,1.02677},
						{-0.37531,0.516848,1.02439},
						{-0.325245,0.521911,1.02402},
						{-0.275882,0.527577,1.02387},
						{-0.225303,0.53295,1.02343},
						{-0.176959,0.538552,1.02896},
						{-0.126577,0.544138,1.03702},
						{-0.0879474,0.547658,1.04142},
						{-0.075,0.551167,1.04417},
						{-0.0402,0.5534,1.0482},
						{-0.463693,0.507173,1.08027},
						{-0.425837,0.511141,1.07546},
						{-0.374025,0.515198,1.0739},
						{-0.324798,0.519641,1.0751},
						{-0.275609,0.52514,1.07413},
						{-0.225977,0.5309,1.07382},
						{-0.175565,0.537407,1.07381},
						{-0.125527,0.542495,1.07441},
						{-0.085626,0.547309,1.07574},
						{-0.0625909,0.551927,1.07175},
						{-0.0259132,0.554694,1.07533},
						{0.0238439,0.561208,1.08147},
						{0.0735357,0.565329,1.08422},
						{0.122252,0.570738,1.08775},
						{0.171655,0.57606,1.091},
						{0.224524,0.581587,1.09262},
						{0.271811,0.587622,1.0953},
						{0.335615,0.592154,1.09431},
						{0.377,0.597143,1.09557},
						{0.373,0.6,1.095},
						{0.810972,-0.313278,1.12881},
						{0.798727,-0.255727,1.119},
						{0.805384,-0.277361,1.1277},
						{0.796467,-0.223848,1.12138},
						{0.8021,-0.2327,1.1421},
						{0.792618,-0.17474,1.12536},
						{0.788477,-0.124848,1.1264},
						{0.784994,-0.0752322,1.12714},
						{0.78113,-0.0246827,1.12686},
						{0.776771,0.0251525,1.12547},
						{0.772898,0.0706549,1.12891},
						{0.764793,0.109414,1.14103},
						{-0.505143,0.501929,1.12907},
						{-0.474858,0.50516,1.12398},
						{-0.425741,0.509171,1.12531},
						{-0.375282,0.512077,1.12378},
						{-0.32462,0.51597,1.12428},
						{-0.275349,0.521452,1.12449},
						{-0.225269,0.528537,1.12363},
						{-0.175598,0.535106,1.12393},
						{-0.125695,0.540138,1.12423},
						{-0.0776875,0.545903,1.12401},
						{-0.04524,0.54884,1.13612},
						{-0.0612273,0.550046,1.11395},
						{-0.02315,0.552017,1.12214},
						{0.024327,0.55708,1.12316},
						{0.0744901,0.562247,1.12303},
						{0.125197,0.568099,1.12301},
						{0.174717,0.57432,1.12358},
						{0.223955,0.579372,1.1221},
						{0.273579,0.585704,1.12294},
						{0.32319,0.590918,1.12511},
						{0.37218,0.595685,1.12872},
						{0.4095,0.5974,1.1328},
						{0.371583,0.601,1.10942},
						{0.403333,0.601333,1.11233},
						{0.826267,-0.3588,1.18573},
						{0.817554,-0.321124,1.17598},
						{0.812171,-0.276151,1.17377},
						{0.805374,-0.225041,1.17285},
						{0.798066,-0.164738,1.16721},
						{0.801645,-0.183785,1.17857},
						{0.794746,-0.124403,1.1739},
						{0.800167,-0.144667,1.195},
						{0.78948,-0.0748767,1.17525},
						{0.783503,-0.0248965,1.1758},
						{0.77932,0.024694,1.17807},
						{0.776364,0.0740698,1.1785},
						{0.773065,0.121083,1.18056},
						{0.772833,0.157167,1.19533},
						{-0.5561,0.4893,1.1905},
						{-0.518054,0.493649,1.19222},
						{-0.490667,0.496944,1.19311},
						{-0.504308,0.501385,1.17554},
						{-0.474961,0.502315,1.17324},
						{-0.425328,0.504474,1.17573},
						{-0.374875,0.507424,1.17408},
						{-0.325989,0.511528,1.17452},
						{-0.275243,0.517389,1.17372},
						{-0.226186,0.525893,1.17353},
						{-0.175715,0.533023,1.17438},
						{-0.125583,0.538161,1.17435},
						{-0.0761044,0.544071,1.17433},
						{-0.0343077,0.548808,1.17758},
						{-0.0528571,0.550143,1.15614},
						{-0.0172234,0.550915,1.17165},
						{0.0240292,0.555445,1.17475},
						{0.0739699,0.561253,1.17449},
						{0.124118,0.566409,1.17414},
						{0.174776,0.572511,1.17313},
						{0.224104,0.578422,1.17095},
						{0.271018,0.58445,1.16871},
						{0.323835,0.589722,1.1705},
						{0.376226,0.594538,1.16799},
						{0.40787,0.596652,1.16965},
						{0.4205,0.6,1.1945},
						{0.83,-0.4065,1.24425},
						{0.826382,-0.367553,1.22418},
						{0.813475,-0.323839,1.21742},
						{0.814195,-0.275325,1.22425},
						{0.808896,-0.225645,1.22365},
						{0.804096,-0.174585,1.22486},
						{0.79766,-0.117817,1.22501},
						{0.801,-0.141453,1.22385},
						{0.794089,-0.0753526,1.22598},
						{0.788794,-0.0249533,1.22524},
						{0.784247,0.0250411,1.22649},
						{0.781012,0.0746795,1.22605},
						{0.776785,0.125288,1.22471},
						{0.776345,0.165,1.22697},
						{-0.560455,0.486091,1.20964},
						{-0.515483,0.491067,1.21488},
						{-0.477713,0.496537,1.22591},
						{-0.425786,0.497224,1.23159},
						{-0.391909,0.498409,1.239},
						{-0.4642,0.5,1.20507},
						{-0.422043,0.500575,1.20881},
						{-0.373162,0.502331,1.22094},
						{-0.325213,0.508113,1.22447},
						{-0.275106,0.512754,1.22345},
						{-0.226362,0.521324,1.22395},
						{-0.175503,0.529983,1.2245},
						{-0.124551,0.535005,1.22347},
						{-0.0750595,0.541268,1.22405},
						{-0.0300147,0.546779,1.22517},
						{-0.0109706,0.550294,1.21759},
						{0.0249639,0.553747,1.22478},
						{0.074064,0.558971,1.22454},
						{0.122851,0.564915,1.22287},
						{0.1753,0.570693,1.22293},
						{0.2186,0.576189,1.22424},
						{0.260114,0.581114,1.21446},
						{0.326227,0.588409,1.23209},
						{0.357091,0.592545,1.22809},
						{0.434105,0.597947,1.23211},
						{0.437,0.6,1.245},
						{0.839824,-0.465471,1.29259},
						{0.834353,-0.423235,1.27829},
						{0.82743,-0.381431,1.27497},
						{0.799,-0.336,1.272},
						{0.809821,-0.329615,1.26713},
						{0.813595,-0.273601,1.27463},
						{0.808752,-0.225829,1.27247},
						{0.806273,-0.175813,1.27247},
						{0.798217,-0.114022,1.2657},
						{0.80145,-0.13185,1.2765},
						{0.794799,-0.0754154,1.27154},
						{0.8,-0.088,1.299},
						{0.789886,-0.0252819,1.27299},
						{0.785136,0.0249214,1.2724},
						{0.781251,0.0750477,1.27358},
						{0.777928,0.124686,1.27433},
						{0.776337,0.167805,1.27304},
						{0.775429,0.223714,1.28836},
						{-0.472218,0.492964,1.27237},
						{-0.429344,0.492885,1.26484},
						{-0.375438,0.497212,1.2718},
						{-0.336269,0.4985,1.29004},
						{-0.355474,0.500684,1.26284},
						{-0.32208,0.503174,1.2692},
						{-0.274956,0.506813,1.27347},
						{-0.225226,0.515748,1.274},
						{-0.174609,0.525257,1.27353},
						{-0.125836,0.527765,1.27185},
						{-0.0759701,0.536755,1.27302},
						{-0.0251543,0.542789,1.27418},
						{0.0166818,0.547341,1.282},
						{0.0319878,0.550805,1.26528},
						{0.0756242,0.554287,1.27292},
						{0.120095,0.560631,1.26932},
						{0.168435,0.566696,1.26139},
						{0.222692,0.572058,1.27179},
						{0.256333,0.576667,1.257},
						{0.33475,0.58975,1.254},
						{0.839917,-0.515792,1.30973},
						{0.835397,-0.474742,1.31253},
						{0.835246,-0.436018,1.31504},
						{0.8245,-0.379611,1.30844},
						{0.817346,-0.313269,1.31135},
						{0.812466,-0.274683,1.32094},
						{0.811039,-0.225331,1.32287},
						{0.799,-0.154,1.347},
						{0.808006,-0.174513,1.32645},
						{0.797691,-0.123357,1.34331},
						{0.804326,-0.126621,1.32334},
						{0.795822,-0.0705445,1.33439},
						{0.801261,-0.0827102,1.32096},
						{0.793732,-0.024949,1.33124},
						{0.8,-0.048,1.3275},
						{0.789351,0.0248831,1.32989},
						{0.78462,0.0739467,1.32892},
						{0.780882,0.12425,1.3268},
						{0.776992,0.173644,1.32528},
						{0.773891,0.222446,1.32598},
						{0.770711,0.263,1.33497},
						{0.769,0.31,1.349},
						{-0.478018,0.487873,1.31951},
						{-0.40425,0.488,1.3455},
						{-0.371723,0.493015,1.32632},
						{-0.33586,0.498023,1.32391},
						{-0.314547,0.500812,1.32517},
						{-0.275354,0.503894,1.32296},
						{-0.224478,0.51258,1.32314},
						{-0.175854,0.519364,1.32158},
						{-0.125825,0.523349,1.3224},
						{-0.0777983,0.531378,1.31876},
						{-0.0285513,0.537692,1.31387},
						{0.0191184,0.543289,1.31403},
						{0.053,0.548,1.311},
						{0.1035,0.549,1.348},
						{0.047,0.55,1.3},
						{0.076,0.5514,1.305},
						{0.13425,0.55725,1.303},
						{0.234,0.5682,1.3032},
						{0.834,-0.427,1.3575},
						{0.813456,-0.215978,1.36548},
						{0.7982,-0.154667,1.3694},
						{0.804463,-0.180098,1.36173},
						{0.79594,-0.124168,1.37229},
						{0.792224,-0.075392,1.3736},
						{0.787264,-0.0248722,1.37359},
						{0.782591,0.024781,1.37237},
						{0.778162,0.0749118,1.37099},
						{0.775013,0.12472,1.37099},
						{0.772173,0.173899,1.37039},
						{0.769523,0.224387,1.36829},
						{0.766827,0.273776,1.37092},
						{0.765644,0.321856,1.37137},
						{0.770222,0.353333,1.379},
						{-0.552,0.471,1.387},
						{-0.4888,0.483,1.3572},
						{-0.4153,0.4868,1.366},
						{-0.365333,0.492786,1.3674},
						{-0.331487,0.497462,1.37112},
						{-0.30975,0.50035,1.36785},
						{-0.281702,0.503433,1.36928},
						{-0.226303,0.510037,1.37207},
						{-0.174892,0.51618,1.37182},
						{-0.126859,0.522484,1.36608},
						{-0.0920526,0.526105,1.36416},
						{0.845,-0.542,1.449},
						{0.834429,-0.47,1.44314},
						{0.834,-0.434667,1.44533},
						{0.813125,-0.211125,1.43313},
						{0.806069,-0.190897,1.42062},
						{0.7955,-0.12,1.41648},
						{0.792263,-0.0749649,1.42285},
						{0.787136,-0.0236562,1.42174},
						{0.782041,0.0253559,1.42387},
						{0.777634,0.0749032,1.42626},
						{0.772365,0.1206,1.42004},
						{0.7695,0.163022,1.42657},
						{0.764737,0.215263,1.42363},
						{0.763,0.253,1.44},
						{-0.514445,0.475889,1.439},
						{-0.404167,0.488167,1.42817},
						{-0.385379,0.489448,1.421},
						{-0.319,0.4975,1.42667},
						{-0.305,0.500111,1.42411},
						{-0.293308,0.501231,1.42338},
						{-0.232632,0.506026,1.42695},
						{-0.17182,0.51528,1.42236},
						{-0.126324,0.522059,1.42415},
						{-0.095,0.5215,1.426},
						{0.866,-0.75575,1.461},
						{0.859644,-0.721511,1.46038},
						{0.847333,-0.656667,1.4675},
						{0.857125,-0.6775,1.46481},
						{0.84564,-0.6206,1.47168},
						{0.853915,-0.623234,1.46681},
						{0.843661,-0.57178,1.47929},
						{0.852053,-0.579868,1.46887},
						{0.841576,-0.525085,1.4745},
						{0.85,-0.547,1.464},
						{0.836567,-0.47475,1.47717},
						{0.832297,-0.430752,1.48035},
						{0.814726,-0.218306,1.48019},
						{0.81109,-0.17706,1.48243},
						{0.798111,-0.116,1.45611},
						{0.806018,-0.126255,1.48353},
						{0.795379,-0.0727027,1.46292},
						{0.802259,-0.0780371,1.48956},
						{0.792729,-0.0238475,1.47369},
						{0.8004,-0.0434,1.4964},
						{0.786812,0.0247812,1.47205},
						{0.781043,0.0743571,1.47117},
						{0.776448,0.122155,1.47347},
						{0.771095,0.174,1.47249},
						{0.767525,0.217625,1.47795},
						{0.7634,0.26375,1.48675},
						{0.7586,0.3184,1.4956},
						{0.754333,0.369333,1.49167},
						{-0.523333,0.474833,1.4715},
						{-0.398,0.492,1.498},
						{-0.315,0.499,1.4525},
						{-0.316778,0.501,1.47678},
						{-0.2534,0.5092,1.4624},
						{-0.2341,0.5046,1.4554},
						{-0.171486,0.515568,1.47586},
						{-0.130382,0.522206,1.47165},
						{0.8245,-0.4365,1.501},
						{0.808445,-0.215622,1.50364},
						{0.798833,-0.16,1.51183},
						{0.805479,-0.175845,1.50558},
						{0.796897,-0.122138,1.51376},
						{0.804482,-0.127536,1.50707},
						{0.794948,-0.0745689,1.51812},
						{0.802582,-0.0773023,1.5074},
						{0.791178,-0.0239068,1.51893},
						{0.80025,-0.0404167,1.506},
						{0.787566,0.0245659,1.51991},
						{0.783539,0.0746538,1.52292},
						{0.779412,0.123863,1.52455},
						{0.774163,0.173489,1.52772},
						{0.770168,0.224115,1.53023},
						{0.765492,0.274523,1.53268},
						{0.762928,0.324036,1.53473},
						{0.74875,0.37275,1.54025},
						{0.758391,0.361219,1.52966},
						{-0.553,0.47,1.543},
						{-0.539333,0.473,1.516},
						{-0.414786,0.489857,1.52893},
						{-0.31572,0.50356,1.52948},
						{-0.279778,0.506778,1.53883},
						{-0.2452,0.5039,1.5286},
						{-0.166979,0.516596,1.52772},
						{-0.13731,0.522643,1.52302},
						{0.87,-0.807,1.596},
						{0.86975,-0.7724,1.59455},
						{0.866909,-0.724182,1.59427},
						{0.861667,-0.6705,1.5955},
						{0.85725,-0.625833,1.59475},
						{0.8482,-0.5568,1.5902},
						{0.851833,-0.580083,1.59242},
						{0.845529,-0.529529,1.58994},
						{0.839538,-0.483308,1.58954},
						{0.795,-0.073,1.599},
						{0.782,0.019,1.55},
						{0.767375,0.175375,1.57275},
						{0.7636,0.2216,1.5502},
						{0.758,0.2725,1.55267},
						{0.753937,0.330313,1.55634},
						{0.751833,0.357167,1.55717},
						{-0.566905,0.468333,1.57638},
						{-0.425773,0.490273,1.57382},
						{-0.384444,0.496889,1.59711},
						{-0.366667,0.501056,1.59111},
						{-0.324125,0.503453,1.57602},
						{-0.276506,0.505265,1.57604},
						{-0.241043,0.502348,1.57626},
						{-0.167418,0.517073,1.57425},
						{-0.130691,0.52225,1.57537},
						{0.7375,0.54,1.5965},
						{0.734,0.590333,1.59133},
						{0.87,-0.80375,1.60275},
						{0.867815,-0.774185,1.60678},
						{0.864591,-0.725068,1.60816},
						{0.859212,-0.673673,1.61085},
						{0.855207,-0.623431,1.61522},
						{0.847296,-0.567926,1.62556},
						{0.852439,-0.580439,1.61383},
						{0.845736,-0.525458,1.62015},
						{0.840804,-0.477639,1.62026},
						{0.837,-0.449,1.603},
						{0.814769,-0.205231,1.63254},
						{0.811631,-0.175662,1.63262},
						{0.807867,-0.126667,1.6325},
						{0.797375,-0.07275,1.60888},
						{0.803395,-0.0783023,1.63388},
						{0.797606,-0.0211212,1.63645},
						{0.8,-0.0485,1.6385},
						{0.792,0.0251538,1.63231},
						{0.787636,0.0719091,1.63527},
						{0.780514,0.127243,1.63105},
						{0.775133,0.176022,1.6282},
						{0.771458,0.224667,1.63625},
						{0.7682,0.2676,1.6433},
						{0.765,0.301,1.644},
						{-0.573,0.467,1.603},
						{-0.45525,0.48925,1.646},
						{-0.422566,0.489487,1.62042},
						{-0.376145,0.493783,1.6206},
						{-0.331744,0.497233,1.62928},
						{-0.36275,0.500625,1.60363},
						{-0.318452,0.501572,1.61369},
						{-0.275353,0.50275,1.62374},
						{-0.238048,0.502952,1.62671},
						{-0.172714,0.516,1.62583},
						{-0.129565,0.519903,1.62592},
						{0.737,0.54325,1.60725},
						{0.735,0.573742,1.62113},
						{0.813,-0.204,1.654},
						{0.808935,-0.174484,1.65648},
						{0.805226,-0.124245,1.66153},
						{0.798,-0.064875,1.67131},
						{0.802074,-0.0783704,1.66032},
						{0.796125,-0.0238281,1.66597},
						{0.8,-0.039,1.6525},
						{0.792441,0.0253676,1.66574},
						{0.787127,0.0749747,1.66847},
						{0.782869,0.124655,1.67185},
						{0.778464,0.173833,1.67367},
						{0.77407,0.224118,1.67473},
						{0.769701,0.274046,1.67897},
						{0.766628,0.312349,1.67702},
						{-0.459381,0.486857,1.67776},
						{-0.424593,0.488831,1.67607},
						{-0.374377,0.493443,1.6762},
						{-0.327534,0.496914,1.67817},
						{-0.291889,0.498667,1.68},
						{-0.2355,0.49875,1.6945},
						{-0.308636,0.500182,1.66736},
						{-0.272692,0.5024,1.67668},
						{-0.228977,0.502442,1.67526},
						{-0.174817,0.514073,1.6769},
						{-0.13029,0.5165,1.67339},
						{0.738,0.5465,1.6815},
						{0.734911,0.574929,1.67896},
						{0.765333,0.288333,1.70133},
						{0.767,0.321,1.701},
						{-0.867667,0.459333,1.74733},
						{-0.729,0.4685,1.7455},
						{-0.610444,0.477778,1.735},
						{-0.5782,0.4849,1.7439},
						{-0.521111,0.492,1.73515},
						{-0.470784,0.484922,1.73},
						{-0.425671,0.484939,1.7261},
						{-0.374598,0.488696,1.72615},
						{-0.325175,0.492612,1.72758},
						{-0.275519,0.495506,1.72577},
						{-0.238645,0.497806,1.72858},
						{-0.196,0.499,1.749},
						{-0.278333,0.5,1.70567},
						{-0.21329,0.501613,1.72597},
						{-0.175598,0.50777,1.72667},
						{-0.130273,0.509376,1.72494},
						{0.738,0.547,1.701},
						{0.73865,0.57945,1.72215},
						{0.914207,-1.02045,1.78952},
						{0.911032,-0.976548,1.79306},
						{0.898167,-0.909833,1.79567},
						{0.9046,-0.9299,1.78957},
						{0.897188,-0.875438,1.79438},
						{0.903889,-0.879333,1.79244},
						{0.892483,-0.824552,1.79255},
						{0.901,-0.8404,1.7932},
						{0.889839,-0.776097,1.79529},
						{0.887,-0.7252,1.7982},
						{0.881455,-0.686818,1.79691},
						{-0.424149,0.244,1.79123},
						{-0.377088,0.245735,1.79321},
						{-0.328191,0.246857,1.79452},
						{-0.289833,0.248,1.79583},
						{-0.154833,0.2455,1.7835},
						{-0.123776,0.242612,1.77294},
						{-0.0935294,0.243647,1.77324},
						{-0.453,0.253,1.7985},
						{-0.423225,0.257796,1.79084},
						{-0.373162,0.26105,1.78844},
						{-0.3255,0.268779,1.78909},
						{-0.284882,0.266788,1.78938},
						{-0.168648,0.275648,1.77661},
						{-0.125033,0.274446,1.7701},
						{-0.094425,0.274075,1.7706},
						{-0.452,0.348,1.798},
						{-0.438167,0.347833,1.7975},
						{-0.355063,0.329,1.79531},
						{-0.330527,0.327242,1.79569},
						{-0.2915,0.34175,1.79656},
						{-0.172,0.324826,1.77913},
						{-0.125306,0.323842,1.772},
						{-0.097,0.320125,1.77425},
						{-0.4573,0.370567,1.79163},
						{-0.425182,0.371724,1.78469},
						{-0.375143,0.374566,1.78299},
						{-0.326657,0.380709,1.78522},
						{-0.29146,0.38274,1.78776},
						{-0.174765,0.374624,1.78027},
						{-0.125702,0.37449,1.77423},
						{-0.0991538,0.374308,1.77831},
						{-0.9058,0.4432,1.7922},
						{-0.883357,0.447357,1.79143},
						{-0.460214,0.431071,1.78771},
						{-0.42441,0.425735,1.78253},
						{-0.375045,0.425688,1.78309},
						{-0.324863,0.426467,1.78372},
						{-0.278609,0.429293,1.78259},
						{-0.203,0.43375,1.79588},
						{-0.175425,0.424436,1.77903},
						{-0.126937,0.424714,1.77578},
						{-0.1,0.405,1.785},
						{-0.869408,0.45537,1.77522},
						{-0.824817,0.46055,1.78467},
						{-0.773941,0.461235,1.78378},
						{-0.724699,0.462904,1.77804},
						{-0.67315,0.46825,1.78528},
						{-0.620981,0.473463,1.77802},
						{-0.574636,0.478937,1.77624},
						{-0.524699,0.482313,1.77463},
						{-0.474011,0.473979,1.77872},
						{-0.424796,0.467705,1.77291},
						{-0.374667,0.469544,1.76976},
						{-0.324526,0.471091,1.76769},
						{-0.274794,0.473165,1.7659},
						{-0.225285,0.484021,1.77635},
						{-0.175701,0.476523,1.77492},
						{-0.126887,0.475424,1.77437},
						{-0.212,0.5,1.7515},
						{-0.1706,0.5018,1.7582},
						{-0.128069,0.502276,1.75728},
						{0.73975,0.58575,1.7675},
						{0.911667,-1.0195,1.806},
						{0.906,-0.973333,1.80511},
						{0.898,-0.907,1.809},
						{0.9021,-0.9325,1.8108},
						{0.895368,-0.875368,1.80705},
						{0.890609,-0.82387,1.81443},
						{0.885291,-0.775161,1.81194},
						{0.882302,-0.724868,1.8096},
						{0.875635,-0.671058,1.80919},
						{0.87305,-0.627067,1.815},
						{0.867017,-0.574983,1.81559},
						{0.86169,-0.523948,1.82121},
						{0.85722,-0.474373,1.82281},
						{0.858111,-0.447778,1.83189},
						{0.8425,-0.253,1.845},
						{0.834981,-0.226412,1.83837},
						{0.828612,-0.175429,1.83871},
						{0.8233,-0.126,1.83932},
						{0.818854,-0.0767708,1.84183},
						{0.817471,-0.0315294,1.84606},
						{-0.853,0.02775,1.84225},
						{-0.826731,0.0363881,1.83724},
						{-0.79025,0.0358,1.84195},
						{-0.511615,0.0274,1.84154},
						{-0.494444,0.0448889,1.84333},
						{0.8115,0.0314,1.8479},
						{-0.816091,0.0830909,1.83991},
						{-0.771115,0.0909508,1.83808},
						{-0.510167,0.0540833,1.84467},
						{-0.471904,0.0639759,1.84194},
						{-0.431917,0.0684583,1.84675},
						{-0.373412,0.0787059,1.84475},
						{-0.33995,0.0868,1.84385},
						{0.806182,0.0709091,1.84745},
						{-0.807364,0.121136,1.84186},
						{-0.779258,0.123403,1.83736},
						{0.801714,0.111143,1.84657},
						{-0.805333,0.154,1.84833},
						{-0.782567,0.161784,1.84635},
						{-0.453857,0.192143,1.84671},
						{-0.423873,0.191309,1.84224},
						{-0.372648,0.187409,1.84005},
						{-0.325336,0.18176,1.83889},
						{-0.2975,0.19625,1.836},
						{-0.457219,0.227656,1.83722},
						{-0.425473,0.22113,1.83177},
						{-0.37488,0.222596,1.82448},
						{-0.324731,0.222819,1.81699},
						{-0.291025,0.226329,1.82496},
						{-0.161,0.247,1.803},
						{-0.57451,0.282959,1.8469},
						{-0.51791,0.27966,1.84687},
						{-0.473603,0.274644,1.83448},
						{-0.42575,0.283894,1.81398},
						{-0.375475,0.287088,1.81094},
						{-0.318594,0.2915,1.80441},
						{-0.277732,0.281366,1.81202},
						{-0.186,0.262,1.813},
						{-0.611211,0.333684,1.84811},
						{-0.575629,0.324818,1.84487},
						{-0.525406,0.324055,1.84012},
						{-0.476643,0.326063,1.83306},
						{-0.424698,0.323515,1.80656},
						{-0.378531,0.322701,1.80512},
						{-0.318755,0.320786,1.80406},
						{-0.284182,0.320647,1.81184},
						{-0.197333,0.325167,1.80783},
						{-0.813154,0.394,1.84869},
						{-0.796,0.394,1.849},
						{-0.610029,0.3796,1.84826},
						{-0.56906,0.372554,1.84629},
						{-0.52549,0.374647,1.84124},
						{-0.483462,0.374846,1.83029},
						{-0.4168,0.3604,1.8002},
						{-0.368583,0.352417,1.80237},
						{-0.323105,0.358562,1.80309},
						{-0.279446,0.367631,1.81691},
						{-0.20375,0.38775,1.81025},
						{-0.199,0.3596,1.8022},
						{-0.917217,0.427145,1.83129},
						{-0.875453,0.434,1.82866},
						{-0.82624,0.436594,1.83179},
						{-0.775673,0.436627,1.83541},
						{-0.731246,0.440721,1.8403},
						{-0.6745,0.4485,1.844},
						{-0.615875,0.42225,1.84796},
						{-0.571039,0.42767,1.84658},
						{-0.52614,0.424752,1.84262},
						{-0.481033,0.419652,1.82177},
						{-0.438933,0.405333,1.80067},
						{-0.262045,0.418939,1.82315},
						{-0.217059,0.432588,1.81876},
						{-0.853,0.45,1.802},
						{-0.8272,0.4526,1.80572},
						{-0.77464,0.45248,1.8052},
						{-0.720192,0.454385,1.81581},
						{-0.677059,0.458212,1.82393},
						{-0.624052,0.46001,1.8219},
						{-0.574848,0.461276,1.8238},
						{-0.527254,0.465,1.82766},
						{-0.48793,0.462326,1.81463},
						{-0.233,0.470387,1.82106},
						{0.75,0.5875,1.837},
						{0.9475,-1.321,1.8985},
						{0.944,-1.292,1.895},
						{0.945,-1.11617,1.89417},
						{0.938429,-1.07543,1.89329},
						{0.856,-0.455,1.854},
						{1.2225,-0.4652,1.88515},
						{1.25,-0.47,1.899},
						{0.858,-0.45,1.86},
						{0.840167,-0.254333,1.871},
						{0.831739,-0.224391,1.86196},
						{0.826826,-0.17087,1.86652},
						{0.819968,-0.124806,1.86403},
						{0.8155,-0.0736471,1.86615},
						{-0.51975,-0.003,1.87025},
						{0.812982,-0.0239107,1.86407},
						{-0.861833,0.0365,1.86367},
						{-0.822421,0.0244737,1.86095},
						{-0.788167,0.0346667,1.86094},
						{-0.512947,0.0180526,1.86195},
						{-0.484667,0.0444667,1.86947},
						{0.80741,0.024623,1.86195},
						{-0.864567,0.0731667,1.88777},
						{-0.828227,0.0880455,1.87464},
						{-0.767222,0.079,1.86663},
						{-0.740286,0.0917714,1.87406},
						{-0.671543,0.0656,1.88217},
						{-0.620061,0.074449,1.87327},
						{-0.575817,0.0841936,1.86733},
						{-0.52217,0.0810982,1.85813},
						{-0.476613,0.0826,1.85488},
						{-0.424257,0.0803465,1.85621},
						{-0.378037,0.0856296,1.85972},
						{-0.338,0.0902,1.85813},
						{0.798545,0.085,1.868},
						{0.801938,0.0729167,1.86527},
						{-0.864786,0.126839,1.88961},
						{-0.833597,0.128274,1.8695},
						{-0.756817,0.128067,1.86005},
						{-0.731978,0.123,1.88284},
						{-0.611425,0.125957,1.88383},
						{-0.57607,0.125076,1.87418},
						{-0.52482,0.123756,1.86543},
						{-0.476671,0.123598,1.86983},
						{-0.425375,0.126068,1.87897},
						{-0.37165,0.1273,1.88366},
						{-0.33052,0.126415,1.88488},
						{-0.2958,0.1258,1.8976},
						{0.795538,0.124846,1.864},
						{0.8008,0.1231,1.858},
						{-0.868,0.181404,1.89549},
						{-0.827411,0.1774,1.88232},
						{-0.767217,0.173906,1.85862}};


		std::vector <detailed_node> detailed_nodes;
		detailed_nodes.resize(test_points.size());
		std::vector<int> x_sort_ids(test_points.size());
		std::vector<int> y_sort_ids(test_points.size());
		std::vector<int> z_sort_ids(test_points.size());
		std::vector<int> end_list(test_points.size());

		// point_with_id point_with_ids[test_points.size()];
		for(int i=0;i<test_points.size();i++){
			point_with_ids[i].id = i;
			point_with_ids[i].pos[0] = test_points[i][0];
			point_with_ids[i].pos[1] = test_points[i][1];
			point_with_ids[i].pos[2] = test_points[i][2];
			detailed_nodes[i].ready = false;
			end_list[i] = -1;
		}
		for(sort_axis=0; sort_axis<3; sort_axis++){
			qsort(point_with_ids, test_points.size(), sizeof(point_with_id), AxisSort);
			for (int i=0 ; i < test_points.size(); i++){
				if(sort_axis==0){
					x_sort_ids[i]=point_with_ids[i].id;
				}
				if(sort_axis==1){
					y_sort_ids[i]=point_with_ids[i].id;
				}
				if(sort_axis==2){
					z_sort_ids[i]=point_with_ids[i].id;
				}
			}
		}
		//最初のmedian特定
		size_t root_middle = ((test_points.size() - 1) / 2);
		int root_median_id = x_sort_ids[root_middle];//最初はx

		detailed_nodes[root_median_id].ready = true;
		detailed_nodes[root_median_id].node_is_right = false;
		detailed_nodes[root_median_id].parent_id = -1;
		detailed_nodes[root_median_id].depth = 0;
		detailed_nodes[root_median_id].axis = 0;
		detailed_nodes[root_median_id].middle = root_middle;
		detailed_nodes[root_median_id].group_size = test_points.size();

		int *d_end_list;
		detailed_node *d_detailed_nodes;
		hipMalloc((void **)&detailed_nodes[root_median_id].x_sort_ids, test_points.size() * sizeof(int));
		hipMalloc((void **)&detailed_nodes[root_median_id].y_sort_ids, test_points.size() * sizeof(int));
		hipMalloc((void **)&detailed_nodes[root_median_id].z_sort_ids, test_points.size() * sizeof(int));
		hipMalloc((void **)&d_end_list, test_points.size() * sizeof(int));
		hipMalloc((void **)&d_detailed_nodes, test_points.size() * sizeof(detailed_node));

		hipMemcpy(detailed_nodes[root_median_id].x_sort_ids, &x_sort_ids[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(detailed_nodes[root_median_id].y_sort_ids, &y_sort_ids[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(detailed_nodes[root_median_id].z_sort_ids, &z_sort_ids[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_end_list, &end_list[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_detailed_nodes, &detailed_nodes[0], test_points.size() * sizeof(detailed_node), hipMemcpyHostToDevice);

		hipDeviceSetLimit(hipLimitStackSize, 1024*1024);
		int dimx_create_node = 32;
		dim3 block_create_node(dimx_create_node, 1);
		dim3 grid_create_node((test_points.size() + block_create_node.x - 1) / block_create_node.x, 1);
		float estimate_depth = log2((test_points.size()+1.0f)/2.0f);
		int depth_count = 0;

		// std::cout << "frames" << frames <<"------------------------------------------------------------------------------------------------------------"<< std::endl;
		// if(frames==114) std::cout<<"dead point is ("<<test_points[684][0]<<","<<test_points[684][1]<<","<<test_points[684][2]<<")"<<std::endl;
		// if(frames==114) std::cout<<"around point is ("<<test_points[683][0]<<","<<test_points[683][1]<<","<<test_points[683][2]<<")"<<std::endl;
		// if(frames==114) std::cout<<"around point is ("<<test_points[682][0]<<","<<test_points[682][1]<<","<<test_points[682][2]<<")"<<std::endl;

		// if(frames==114){
		// 	std::cout<<"test_points = {";
		// 	for(int i=0;i<test_points.size();i++){
		// 		std::cout<<"{";
		// 		for(int j=0;j<3;j++){
		// 			std::cout<<test_points[i][j];
		// 			if(j!=(3-1)) std::cout<<",";
		// 		}
		// 		std::cout<<"}";
		// 		if(i!=(test_points.size()-1)) std::cout<<",";
		// 		std::cout<<std::endl;
		// 	}
		// 	std::cout<<"};"<<std::endl;
		// }
		std::cout<<"estimate_depth = "<<estimate_depth<<std::endl;
		while(1){
			std::cout<<"call depth = "<< depth_count <<std::endl;
			// std::cout<<"create kernel start"<<std::endl;
			d_DepthCreateNode<<<grid_create_node,block_create_node>>>(test_points.size(),d_detailed_nodes,d_end_list);
			// std::cout<<"create kernel end"<<std::endl;
			if(depth_count >= estimate_depth){
				std::cout<<"limit termination"<<std::endl;
				break;
			} 
			depth_count++;
			
			// if(all_end) break;
		}
		hipMemcpy(&end_list[0], d_end_list, test_points.size() * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&detailed_nodes[0], d_detailed_nodes, test_points.size() * sizeof(detailed_node), hipMemcpyDeviceToHost);

		hipFree(detailed_nodes[root_median_id].x_sort_ids);
		hipFree(detailed_nodes[root_median_id].y_sort_ids);
		hipFree(detailed_nodes[root_median_id].z_sort_ids);
		hipFree(d_end_list);
		hipFree(d_detailed_nodes);
		bool all_end = std::all_of(end_list.begin(), end_list.end(), [](int end) { return 0 < end; });
		if(all_end) std::cout<<"successful termination"<<std::endl;


		// first=false;
	}
	// root_id=root_median_id;
	//表示用スクリプト

	/////////////////////////////////////////////////////////////////////////////////////////施工
	
	build_end = clock();
	// printf("create tree time is %.5fs\n",(double)(build_end-build_start)/CLOCKS_PER_SEC);
	//root_id表示
	// std::cout << "root_id = " << root_id << std::endl;
	//nodes表示

	// std::cout<<"search kernel start"<<std::endl;
	// std::cout<<"3.01"<<std::endl;
    //ホスト1次配列宣言
	//kd
	std::vector<int> h_parent_ids(points_array.size());
	std::vector<int> h_left_ids(points_array.size());
	std::vector<int> h_right_ids(points_array.size());
	std::vector<int> h_axes(points_array.size());

	std::vector<int> h_point_neighbor(points_array.size());
	std::vector<int> h_point_neighbor_size(1);
	std::vector<long long int> h_neighbor_time(points_array.size());

	//normal
    std::vector<float> h_points(points_array.size() * 3);
    std::vector<int> h_neighbor_points_indices(neighbor_points_count);
    std::vector<float> h_normals(points_array.size() * 3);
    std::vector<float> h_curvatures(points_array.size());
    std::vector<long long int> h_covariance_compute_time(points_array.size());
    std::vector<long long int> h_eigen_compute_time(points_array.size());
    // std::cout<<"3.02"<<std::endl;
    //デバイス1次配列宣言
	//kd
	int *d_parent_ids,*d_left_ids,*d_right_ids,*d_axes;
	int *d_point_neighbor,*d_point_neighbor_size;
	long long int *d_neighbor_time;

	//normal
    float *d_points,*d_normals,*d_curvatures;
    int *d_neighbor_points_indices,*d_neighbor_start_indices;
    long long int *d_covariance_compute_time,*d_eigen_compute_time;


    // std::cout<<"3.03"<<std::endl;
    //メモリ確保
	//kd
	hipMalloc((void **)&d_parent_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_left_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_right_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_axes, points_array.size() * sizeof(int));

	hipMalloc((void **)&d_point_neighbor, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_point_neighbor_size, sizeof(int));
	hipMalloc((void **)&d_neighbor_time, points_array.size() * sizeof(long long int));
	//normal
    hipMalloc((void **)&d_points, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_neighbor_points_indices, neighbor_points_count * sizeof(int));
    hipMalloc((void **)&d_neighbor_start_indices, points_array.size() * sizeof(int));
    hipMalloc((void **)&d_normals, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_curvatures, points_array.size() * sizeof(float));
    hipMalloc((void **)&d_covariance_compute_time, points_array.size() * sizeof(long long int));
    hipMalloc((void **)&d_eigen_compute_time, points_array.size() * sizeof(long long int));
    // std::cout<<"3.04"<<std::endl;
    //1次配列化
    int k=0,l=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            h_points[k]=points_array[i][j];
            k++;
        }
        for(int j=0;j<neighbor_points_indices[i].size();j++){//近傍
            h_neighbor_points_indices[l]=neighbor_points_indices[i][j];
            l++;
        }
		h_parent_ids[i]=nodes[i].parent_id;
		h_left_ids[i]=nodes[i].left_id;
		h_right_ids[i]=nodes[i].right_id;
		h_axes[i]=nodes[i].axis;
    }



    // std::cout<<"3.05"<<std::endl;
    //コピー
	//kd
	hipMemcpy(d_parent_ids, &h_parent_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_left_ids, &h_left_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_right_ids, &h_right_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_axes, &h_axes[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	//normal
    hipMemcpy(d_points, &h_points[0], points_array.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_points_indices, &h_neighbor_points_indices[0], neighbor_points_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_start_indices, &neighbor_start_indices[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
    // std::cout<<"3.06"<<std::endl;
    //並列スレッド設定
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((points_array.size() + block.x - 1) / block.x, 1);
    // std::cout<<"3.07"<<std::endl;
    // std::cout<<"normalsGPUstart"<<std::endl;
    // hipDeviceSetLimit(hipLimitStackSize, 1024*8);
    //実行
    NormalsGPU<<<grid,block>>>(/*d_detailed_nodes,*/d_neighbor_time,d_point_neighbor_size,d_point_neighbor,d_parent_ids,d_left_ids,d_right_ids,d_axes,root_id,d_points,points_array.size(),d_neighbor_points_indices,d_neighbor_start_indices,neighbor_points_count,d_normals,d_curvatures,d_covariance_compute_time,d_eigen_compute_time);
    // std::cout<<"normalsGPUend"<<std::endl;
    // std::cout<<"3.08"<<std::endl;
    //コピー
	//kd
	hipMemcpy(&h_point_neighbor[0], d_point_neighbor, points_array.size() * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_point_neighbor_size[0], d_point_neighbor_size, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_neighbor_time[0], d_neighbor_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
	//normal
    hipMemcpy(&h_normals[0], d_normals, points_array.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_curvatures[0], d_curvatures, points_array.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_covariance_compute_time[0], d_covariance_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_compute_time[0], d_eigen_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    // std::cout<<"3.09"<<std::endl;
    //2次配列化
	for(int i=0;i<h_point_neighbor_size[0];i++){
		point_neighbor[i]=h_point_neighbor[i];
	}
	
	point_neighbor.resize(h_point_neighbor_size[0]);
	// std::cout<<"host cu size "<<h_point_neighbor_size[0]<<std::endl;

    k=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            normals_array[i][j]=h_normals[k];
            k++;
        }
        curvatures_array[i]=h_curvatures[i];
        covariance_compute_time[i]=h_covariance_compute_time[i];
        eigen_compute_time[i]=h_eigen_compute_time[i];
		neighbor_time[i]=h_neighbor_time[i];
    }
	// std::cout<<"search kernel end"<<std::endl;
    // std::cout<<"cu_normals : "<<normals_array[0][0]<<","<<normals_array[0][1]<<","<<normals_array[0][2]<<std::endl;
    // std::cout<<"3.10"<<std::endl;
    //メモリ解放
	//kd2

	//kd
	hipFree(d_parent_ids);
	hipFree(d_left_ids);
	hipFree(d_right_ids);
	hipFree(d_axes);

	hipFree(d_point_neighbor);
	hipFree(d_point_neighbor_size);
	hipFree(d_neighbor_time);

	//normal
    hipFree(d_points);
    hipFree(d_neighbor_points_indices);
    hipFree(d_neighbor_start_indices);
    hipFree(d_normals);
    hipFree(d_curvatures);
    hipFree(d_covariance_compute_time);
    hipFree(d_eigen_compute_time);


	frames++;
}