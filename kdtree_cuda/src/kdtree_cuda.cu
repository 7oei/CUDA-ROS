#include "hip/hip_runtime.h"


#include <string.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kdtree_cuda/kdtree_cuda.hpp"
#include <vector>
#include <iterator>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int sort_axis=0;
int frames=0;
typedef struct
{
	int	id;
	float pos[3];
} point_with_id;

typedef struct
{
	int	parent_id;
	int left_id;
	int right_id;
	int axis;
} node;

bool first=true;

//	年齢(昇順)
__host__ int AxisSort(const void * n1, const void * n2)
{
	if (((point_with_id *)n1)->pos[sort_axis] > ((point_with_id *)n2)->pos[sort_axis])
	{
		return 1;
	}
	else if (((point_with_id *)n1)->pos[sort_axis] < ((point_with_id *)n2)->pos[sort_axis])
	{
		return -1;
	}
	else
	{
		return 0;
	}
}

__host__ int CreateTree(int* root_id,std::vector <node>& nodes, std::vector<std::vector<float>> points,std::vector<int> group_indices,int parent_id,bool node_is_right)
{
	//入力データ初期化
	int group_size = group_indices.size();
	// std::cout<<"group_size"<<group_size<<std::endl;
	point_with_id point_with_ids[group_size];
	std::vector<std::vector<int>> axis_sort_ids(3, std::vector<int>(group_size));
	// std::cout<<"oppai 1 "<<std::endl;
	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////3*points
		point_with_ids[i].id=group_indices[i];
		point_with_ids[i].pos[0]=points[group_indices[i]][0];
		point_with_ids[i].pos[1]=points[group_indices[i]][1];
		point_with_ids[i].pos[2]=points[group_indices[i]][2];
	}
	// std::cout<<"oppai 2 "<<std::endl;

	//ソート
	float max[3],min[3],median[3],length[3];
	int axis_median_id[3];
	int median_id;
	for(sort_axis=0; sort_axis<3; sort_axis++){//x,y,zそれぞれにソート
		// std::cout<<"sort_axis = "<<sort_axis<<std::endl;
		qsort(point_with_ids, group_size, sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < group_size ; i++){///////////////////////////////////////////////////////////////////////////////////////////3*points
			axis_sort_ids[sort_axis][i]=point_with_ids[i].id;
			// printf("%d, %f, %f, %f \n", point_with_ids[i].id, point_with_ids[i].pos[0], point_with_ids[i].pos[1], point_with_ids[i].pos[2]);
		}
		// std::cout<<std::endl;
		//max,min,median,axis_median_id取得
		max[sort_axis]=point_with_ids[group_size-1].pos[sort_axis];//minとmaxいらんかも
		min[sort_axis]=point_with_ids[0].pos[sort_axis];
		length[sort_axis]=max[sort_axis]-min[sort_axis];
		median[sort_axis]=point_with_ids[(group_size-1)/2].pos[sort_axis];//偶数なら小さい方(-1)消せば大きい方
		axis_median_id[sort_axis]=point_with_ids[(group_size-1)/2].id;
	}
	// std::cout<<"oppai 3 "<<std::endl;
	// std::cout<<"x_length = "<< length[0] <<", x_median["<<axis_median_id[0]<<"] = "<<median[0]<<std::endl;
	// std::cout<<"y_length = "<< length[1] <<", y_median["<<axis_median_id[1]<<"] = "<<median[1]<<std::endl;
	// std::cout<<"z_length = "<< length[2] <<", z_median["<<axis_median_id[2]<<"] = "<<median[2]<<std::endl;
	// std::cout<<std::endl;
	// for(int i=0;i<group_size;i++){
	// 	std::cout<<"point_id["<<i<<"] = "<<point_with_ids[i].id<<std::endl;
	// }
	

	//中央値id設定、長軸設定
	if(length[0]>=length[1]&&length[0]>=length[2]){
		median_id=axis_median_id[0];
		nodes[median_id].axis=0;
	}
	if(length[1]>=length[0]&&length[1]>=length[2]){
		median_id=axis_median_id[1];
		nodes[median_id].axis=1;
	}
	if(length[2]>=length[0]&&length[2]>=length[1]){
		median_id=axis_median_id[2];
		nodes[median_id].axis=2;
	}
	// std::cout<<"oppai 4 "<<std::endl;

	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////points
		group_indices[i]=axis_sort_ids[nodes[median_id].axis][i];
	}
	// std::cout<<"oppai 5 "<<std::endl;
	// memcpy(&group_indices[0], axis_sort_ids[nodes[median_id].axis], group_size*sizeof(int));
	// std::vector<int> group_indices2;
	// group_indices.resize(0);
	// copy(axis_sort_ids[nodes[median_id].axis].begin(), axis_sort_ids[nodes[median_id].axis].end(), back_inserter(group_indices) );



	//node初期化
	nodes[median_id].left_id=-1;
	nodes[median_id].right_id=-1;

	//親設定、親の左右設定
	nodes[median_id].parent_id=parent_id;
	if(parent_id>=0){//親あり
		if(!node_is_right) nodes[parent_id].left_id=median_id;
		if(node_is_right) nodes[parent_id].right_id=median_id;
	}
	else{//親なし
		*root_id=median_id;
	}
	// std::cout<<"oppai 6 "<<std::endl;

	// std::vector<int> right_group(group_size);
	// std::vector<int> left_group(group_size);
	// int right_count=0;
	// int left_count=0;

	// for(int i=0;i<=((group_size-1)/2)-1;i++){////////////////////////////////////////////////////////////////////////////////////points
	// 	left_group[left_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	left_count++;
	// }
	// left_group.resize(left_count);///////////////////////////////////////////////////////////////////////////////////////////////points
	// for(int i=((group_size-1)/2)+1;i<group_size;i++){
	// 	right_group[right_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	right_count++;
	// }
	// right_group.resize(right_count);

	// std::vector<int> v0{1,2,3,4};
	// std::vector<int> v9(v0.begin(),v0.end());  // [1,2,3,4]
	// std::vector<int> v10(v0.begin(),v0.end());  

	// group_indices.resize(8);
	// for(int i=0;i<8;i++){
	// 	group_indices[i]=i;
	// }
	// size_t middle = ((8-1)/2);

	size_t middle = ((group_size-1)/2);
	std::vector<int>::iterator middleIter(group_indices.begin());
	std::advance(middleIter, middle);
	// std::cout<<"advance end"<<std::endl;

	std::vector<int> left_group(group_indices.begin(), middleIter);
	++middleIter;
	std::vector<int> right_group(middleIter, group_indices.end());
	// std::cout<<"oppai 7 "<<std::endl;
	// std::cout<<"group end"<<std::endl;
	// std::cout<<"left group is [";
	// for(int i=0;i<left_group.size();i++){
	// 	if(i<10) std::cout<<left_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	// std::cout<<"right group is [";
	// for(int i=0;i<right_group.size();i++){
	// 	if(i<10) std::cout<<right_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	//


	// std::cout<<"median_id"<<median_id<<std::endl;
	// std::cout<<"parent_id"<<parent_id<<std::endl;
	// std::cout<<"left_id"<<nodes[median_id].left_id<<std::endl;
	// std::cout<<"right_id"<<nodes[median_id].right_id<<std::endl;
	// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;

	//right,left group表示
	// std::cout<<"left_group is (";
	// for(int i=0;i<left_group.size();i++){
	// 	std::cout<<left_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<"right_group is (";
	// for(int i=0;i<right_group.size();i++){
	// 	std::cout<<right_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<"--------------------------------------------------------------------------------"<<std::endl;
	bool left=false;
	bool right=false;
	if(group_size>1){//子がいる
		if(left_group.size()>0){//左に子がいる
			left= CreateTree(root_id,nodes,points,left_group,median_id,false);
		}
		else left=true;
		if(right_group.size()>0){//右に子がいる
			right= CreateTree(root_id,nodes,points,right_group,median_id,true);
		}
		else right=true;
		// std::cout<<"oppai 8 "<<std::endl;
		if(right&&left) return 1;
	}
	else return 1;//子がいない
}

__host__ int CreateNode(int* root_id,int point_size,std::vector <node>& nodes, std::vector<std::vector<int>> axis_sort_ids,int depth,int parent_id,bool node_is_right)
{

	// std::cout << "oppai 0" << std::endl;
	// std::cout << std::endl;
	int group_size = axis_sort_ids[0].size();
	int axis = depth % 3;
	size_t middle = ((group_size-1)/2);
	int median_id = axis_sort_ids[axis][middle];
	nodes[median_id].axis = axis;
	nodes[median_id].parent_id = parent_id;
	nodes[median_id].left_id = -1;
	nodes[median_id].right_id = -1;
	if(parent_id >= 0){ // 親あり
		if(!node_is_right) nodes[parent_id].left_id = median_id;
		if(node_is_right) nodes[parent_id].right_id = median_id;
	}
	else{ // 親なし
		*root_id = median_id;
	}
	// std::cout << "oppai 4" << std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout << "axis_sort_ids ="<<std::endl;
	// for(int j = 0; j < 3; j++){
	// 	if(j==0) std::cout << "x =";
	// 	if(j==1) std::cout << "y =";
	// 	if(j==2) std::cout << "z =";
	// 	for(int i = 0; i < group_size; i++){
	// 		std::cout << axis_sort_ids[j][i] << ",";
	// 	}
	// 	std::cout<<std::endl;
	// }

	if(group_size > 1){ // 子あり
		std::vector<int>::iterator middle_iter(axis_sort_ids[axis].begin());
		std::advance(middle_iter,middle);
		std::vector<int> left_group(axis_sort_ids[axis].begin(),middle_iter);
		++middle_iter;
		std::vector<int> right_group(middle_iter,axis_sort_ids[axis].end());

		// std::cout<<"median_id"<<median_id<<std::endl;
		// std::cout<<"middle"<<middle<<std::endl;
		// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;
		// std::cout<<"group is (";
		// for(int i=0;i<group_size;i++){
		// 	std::cout<<axis_sort_ids[axis][i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"left_group is (";
		// for(int i=0;i<left_group.size();i++){
		// 	std::cout<<left_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"right_group is (";
		// for(int i=0;i<right_group.size();i++){
		// 	std::cout<<right_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;

		// std::cout << "oppai 1" << std::endl;

		std::vector<std::vector<int>> left_axis_sort_ids(3,std::vector<int>(left_group.size()));
		std::vector<std::vector<int>> right_axis_sort_ids(3,std::vector<int>(right_group.size()));

		std::vector<int> next_group(point_size,0);/////////////これどうにかしたい
		std::vector<int> left_axis_count(3,0);
		std::vector<int> right_axis_count(3,0);
		// std::cout << "oppai 1.5" << std::endl;
		// std::cout << "next_group.size()" << next_group.size() <<std::endl;
		// std::cout << "left_group.size()" << left_group.size() <<std::endl;
		// std::cout << "right_group.size()" << right_group.size() <<std::endl;
		for(int i = 0; i < left_group.size(); i++){
			// std::cout << "oppai 1.51" << std::endl;
			left_axis_sort_ids[axis][i] = left_group[i];
			// std::cout << "oppai 1.52" << std::endl;
			// std::cout << "left_group[i]" << left_group[i] <<std::endl;
			next_group[left_group[i]] = -1;//これで死んでそう//left_group[i]がnext_groupのレンジを超えている//この式の参照indexおかしい//1段目では正しく作用
		}
		// std::cout << "oppai 1.6" << std::endl;
		for(int i = 0; i < right_group.size(); i++){
			right_axis_sort_ids[axis][i] = right_group[i];
			// std::cout << "right_group[i]" << right_group[i] <<std::endl;
			next_group[right_group[i]] = 1;
		}
		// std::cout << "oppai 2" << std::endl;
		for(int i = 0; i < group_size; i++){
			for(int j = 0; j < 3; j++){
				if(j==axis) continue;
				if(next_group[axis_sort_ids[j][i]] == -1){
					left_axis_sort_ids[j][left_axis_count[j]] = axis_sort_ids[j][i];
					left_axis_count[j]++;
					// std::cout << "left_axis_count["<<j<<"] = "<<left_axis_count[j]<<std::endl;
				}
				else if(next_group[axis_sort_ids[j][i]] == 1){
					right_axis_sort_ids[j][right_axis_count[j]] = axis_sort_ids[j][i];
					right_axis_count[j]++;
					// std::cout << "right_axis_count["<<j<<"] = "<<right_axis_count[j]<<std::endl;
				}
			}
		}

		bool left = false;
		bool right = false;
		if(left_group.size() > 0) left = CreateNode(root_id,point_size,nodes,left_axis_sort_ids,depth+1,median_id,false);
		else left = true;

		if(right_group.size() > 0) right = CreateNode(root_id,point_size,nodes,right_axis_sort_ids,depth+1,median_id,true);
		else right = true;

		if(right&&left) return 1;
	}
	else return 1;
}

__global__ void d_CreateNode(int point_size,int group_size,int depth,int parent_id,bool node_is_right,int *x_sort_ids,int *y_sort_ids,int *z_sort_ids,int *root_id,node* nodes)
{

	// printf("create node open\n");
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
	printf("idx = %d, ",idx);
	// printf("group size = %d\n",group_size);
	int axis = depth % 3;
	size_t middle = ((group_size-1)/2);
	int median_id;
	if(axis==0) median_id = x_sort_ids[middle];
	if(axis==1) median_id = y_sort_ids[middle];
	if(axis==2) median_id = z_sort_ids[middle];

	// if(median_id==1||median_id==13||median_id==14||median_id==15||median_id==19||median_id==36||median_id==45){//ここでは正しい
	// 	printf("1 x_sort_ids[] = ");
	// 	for(int i=0;i<group_size;i++){
	// 		printf("%d,",x_sort_ids[i]);
	// 	}
	// 	printf("\n");
	// }

	int *copy_x_sort_ids,*copy_y_sort_ids,*copy_z_sort_ids;
	copy_x_sort_ids = (int *)malloc(group_size * sizeof(int));
	copy_y_sort_ids = (int *)malloc(group_size * sizeof(int));
	copy_z_sort_ids = (int *)malloc(group_size * sizeof(int));
	memcpy(copy_x_sort_ids, x_sort_ids, group_size * sizeof(int));
	memcpy(copy_y_sort_ids, y_sort_ids, group_size * sizeof(int));
	memcpy(copy_z_sort_ids, z_sort_ids, group_size * sizeof(int));

	nodes[median_id].axis = axis;
	nodes[median_id].parent_id = parent_id;
	nodes[median_id].left_id = -1;
	nodes[median_id].right_id = -1;
	// if(median_id==1||median_id==13||median_id==14||median_id==15||median_id==19||median_id==36||median_id==45){//ここでは正しい
	// 	printf("2 copy_x_sort_ids[] = ");
	// 	for(int i=0;i<group_size;i++){
	// 		printf("%d,",copy_x_sort_ids[i]);
	// 	}
	// 	printf("\n");
	// }

	// printf("1");
	if(parent_id >= 0){ // 親あり
		free(x_sort_ids);
		free(y_sort_ids);
		free(z_sort_ids);
		if(!node_is_right) nodes[parent_id].left_id = median_id;
		if(node_is_right) nodes[parent_id].right_id = median_id;
	}
	else{ // 親なし
		printf("root update\n");
		*root_id = median_id;
	}
	// printf("2");
	if(group_size > 1){ // 子あり
		int left_group_size = 0;
		int right_group_size = 0;
		int *left_x_sort_ids,*left_y_sort_ids,*left_z_sort_ids;
		int *right_x_sort_ids,*right_y_sort_ids,*right_z_sort_ids;
		int *next_group;
		next_group = (int *)malloc(point_size * sizeof(int));
		left_x_sort_ids = (int *)malloc(middle * sizeof(int));
		left_y_sort_ids = (int *)malloc(middle * sizeof(int));
		left_z_sort_ids = (int *)malloc(middle * sizeof(int));
		right_x_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		right_y_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		right_z_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		int left_axis_count[3]={0,0,0};
		int right_axis_count[3]={0,0,0};
		// printf("\n\n\n");
		// printf("median_id = %d\n",median_id);
		// if(!node_is_right) printf("node is left\n");
		// else printf("node is right\n");
		// printf("parent_id = %d\n",parent_id);
		// printf("middle = %d\n",middle);
		// printf("axis = %d\n",nodes[median_id].axis);

		// printf("3");
		// printf("axis = %d",axis);
		// if(median_id==1||median_id==14||median_id==19||median_id==36||median_id==45){//ここでは正しい
		// 	printf("3 copy_x_sort_ids[] = ");
		// 	for(int i=0;i<group_size;i++){
		// 		printf("%d,",copy_x_sort_ids[i]);
		// 	}
		// 	printf("\n");
		// }


		if(axis==0){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_x_sort_ids[left_axis_count[0]] = copy_x_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[0]++;
					// printf("3.02 ");
					// printf("parent_id = %d",parent_id);
					// printf("copy_x_sort_ids[%d] = %d ",i,copy_x_sort_ids[i]);
					next_group[copy_x_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_x_sort_ids[i],next_group[copy_x_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_x_sort_ids[right_axis_count[0]] = copy_x_sort_ids[i];
					right_axis_count[0]++;
					next_group[copy_x_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_x_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}
		}
		else if(axis==1){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_y_sort_ids[left_axis_count[1]] = copy_y_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[1]++;
					// printf("3.02 ");
					// printf("copy_y_sort_ids[%d] = %d ",i,copy_y_sort_ids[i]);
					next_group[copy_y_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_y_sort_ids[i],next_group[copy_y_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_y_sort_ids[right_axis_count[1]] = copy_y_sort_ids[i];
					right_axis_count[1]++;
					next_group[copy_y_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_y_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}

		}
		else if(axis==2){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_z_sort_ids[left_axis_count[2]] = copy_z_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[2]++;
					// printf("3.02 ");
					// printf("copy_z_sort_ids[%d] = %d ",i,copy_z_sort_ids[i]);
					next_group[copy_z_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_z_sort_ids[i],next_group[copy_z_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_z_sort_ids[right_axis_count[2]] = copy_z_sort_ids[i];
					right_axis_count[2]++;
					next_group[copy_z_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_z_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}
		}
		// printf("\n");
		// printf("4");
		left_group_size = left_axis_count[axis];
		right_group_size = right_axis_count[axis];

		// printf("group is (");
		// for(int i=0;i<group_size;i++){
		// 	if(axis==0) printf("%d,",copy_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",copy_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",copy_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("left_group is (");
		// for(int i=0;i<left_group_size;i++){
		// 	if(axis==0) printf("%d,",left_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",left_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",left_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("right_group is (");
		// for(int i=0;i<right_group_size;i++){
		// 	if(axis==0) printf("%d,",right_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",right_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",right_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("\n\n\n");

		for(int i = 0; i < group_size; i++){
			for(int j = 0; j < 3; j++){
				if(j==axis) continue;
				if(j==0){//x実装
					if(next_group[copy_x_sort_ids[i]] == -1){
						left_x_sort_ids[left_axis_count[j]] = copy_x_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_x_sort_ids[i]] == 1){
						right_x_sort_ids[right_axis_count[j]] = copy_x_sort_ids[i];
						right_axis_count[j]++;
					}
				}
				if(j==1){//y実装
					if(next_group[copy_y_sort_ids[i]] == -1){
						left_y_sort_ids[left_axis_count[j]] = copy_y_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_y_sort_ids[i]] == 1){
						right_y_sort_ids[right_axis_count[j]] = copy_y_sort_ids[i];
						right_axis_count[j]++;
					}
				}
				if(j==2){//z実装
					if(next_group[copy_z_sort_ids[i]] == -1){
						left_z_sort_ids[left_axis_count[j]] = copy_z_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_z_sort_ids[i]] == 1){
						right_z_sort_ids[right_axis_count[j]] = copy_z_sort_ids[i];
						right_axis_count[j]++;
					}
				}
			}
		}
		free(copy_x_sort_ids);
		free(copy_y_sort_ids);
		free(copy_z_sort_ids);
		free(next_group);
		// printf("5");
		// if(median_id==18) printf("Hit!!!!!!!!!!!!!!! left_x_sort_ids[16] = %d\n\n\n",left_x_sort_ids[16]);
		// if(median_id==33) printf("Hit!!!!!!!!!!!!!!! right_x_sort_ids[16] = %d\n\n\n",right_x_sort_ids[16]);
		// if(median_id==3145) printf("Hit!!!!!!!!!!!!!!! left_x_sort_ids[156] = %d\n\n\n",left_x_sort_ids[156]);
		// if(median_id==3877) printf("Hit!!!!!!!!!!!!!!! left_y_sort_ids[1] = %d\n\n\n",left_x_sort_ids[1]);
		// if(median_id==3888) printf("Hit!!!!!!!!!!!!!!! left_y_sort_ids[1] = %d\n\n\n",left_y_sort_ids[1]);
		// hipDeviceSynchronize();
		if(left_group_size > 0) d_CreateNode<<<1, 1>>>(point_size,left_group_size,depth+1,median_id,false,left_x_sort_ids,left_y_sort_ids,left_z_sort_ids,root_id,nodes);
		else {
			free(left_x_sort_ids);
			free(left_y_sort_ids);
			free(left_z_sort_ids);
		}
		if(right_group_size > 0) d_CreateNode<<<1, 1>>>(point_size,right_group_size,depth+1,median_id,true,right_x_sort_ids,right_y_sort_ids,right_z_sort_ids,root_id,nodes);
		else {
			free(right_x_sort_ids);
			free(right_y_sort_ids);
			free(right_z_sort_ids);
		}
		// hipDeviceSynchronize();
		// free(left_x_sort_ids);
		// free(left_y_sort_ids);
		// free(left_z_sort_ids);
		// free(right_x_sort_ids);
		// free(right_y_sort_ids);
		// free(right_z_sort_ids);
		
	}
}

__device__ void d_PointRangeCheckAndAdd(int *range_indices_size,int *range_indices,int head_id,float* points,float* search_point,float range_sq)
{
	float dist_sq = powf(points[head_id*3+0]-search_point[0],2)+powf(points[head_id*3+1]-search_point[1],2)+powf(points[head_id*3+2]-search_point[2],2);
	if(dist_sq<range_sq){
		// printf("device inside");
		range_indices[*range_indices_size] = head_id;
		*range_indices_size+=1;
	} 
}

__device__ int d_SearchSubTree(int *range_indices_size,int *range_indices,int root_id,node* nodes,float* points,float* search_point,float range_sq)
{
	int head_id = root_id;
	bool cross,next_is_right;
	//潜り
	while(1){
		if(search_point[nodes[head_id].axis]>points[head_id*3+nodes[head_id].axis]) next_is_right = true;
		else next_is_right = false;

		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
			if(next_is_right){//right
				head_id = nodes[head_id].right_id;
			}
			else{//left
				head_id = nodes[head_id].left_id;
			}
		}
		else break;
	}
	//rootが底
	if(head_id==root_id) {
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		return 1;
	}
	//昇り
	int last_id;
	while(1){
		cross = false;
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		//昇る
		last_id = head_id;
		head_id = nodes[head_id].parent_id;
		//交差判定　1軸のみ低効率
		float axis_diff_sq = powf(points[head_id*3+nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
		if(axis_diff_sq < range_sq) cross = true;
		int sub_tree=0;
		if(cross){
			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
		}
		if(head_id==root_id){
			d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
			break;
		}
	}

	return 1;
}

__device__ int EigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 

__global__ void NormalsGPU(long long int* neighbor_time,int *point_neighbor_size,int* point_neighbor,int* d_parent_ids,int* d_left_ids,int* d_right_ids,int* d_axes,int root_id,float* points,int point_size,int* neighbor_points_indices,int* neighbor_start_indices,int neighbor_points_count,float* normals,float* curvatures,long long int* covariance_time,long long int* eigen_time)
{
    // printf("normalsGPU");
    //インデックス取得
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
    unsigned int output_id=50;
    // printf("idx = %d ,", idx);
	if(idx==output_id){
		long long int neighbor_start, neighbor_stop;
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_start));
		////////////////////////////////////////////////////////////////
		node *nodes = (node*)malloc(sizeof(node) * point_size);

		for(int i=0;i<point_size;i++){
			nodes[i].parent_id=d_parent_ids[i];
			nodes[i].left_id=d_left_ids[i];
			nodes[i].right_id=d_right_ids[i];
			nodes[i].axis=d_axes[i];
		}
		float search_point[3];
		search_point[0]=points[idx*3+0];
		search_point[1]=points[idx*3+1];
		search_point[2]=points[idx*3+2];
		
		int *range_indices = (int*)malloc(sizeof(int) * point_size);
		int range_indices_size = 0;

		float range_sq = 0.15*0.15;

		//探索関数の実行
		int range_search = d_SearchSubTree(&range_indices_size,range_indices,root_id,nodes,points,search_point,range_sq);
		// std::cout<<"range_indices.size()"<<range_indices.size()<<std::endl;
		if(range_search==1) {
			// printf("device range_indices size is =%d",range_indices_size);
			// printf("device range_indices is [");
			for(int i=0;i<range_indices_size;i++){
				// printf("%d,",range_indices[i]);
				point_neighbor[i]=range_indices[i];
			}
			// printf("]\n");
			point_neighbor_size[0]=range_indices_size;
			// printf("device size %d\n",range_indices_size);
		}
		
		free (nodes);
		free (range_indices);
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_stop));
		neighbor_time[idx]=neighbor_stop - neighbor_start;
		////////////////////////////////////////////////////////////////////
	}


    if(idx<point_size-1){//対象スレッド内のみ計算
        //デバッグ用
        // if(idx==0||idx==10||idx==20) printf("points(%d) = %f,%f,%f\n",idx,points[idx*3+0],points[idx*3+1],points[idx*3+2]);
        
        // printf("idx<point_size");
        //近傍点終点インデックスの定義
        int end_indices;
        if(idx==(point_size-1)) end_indices = neighbor_points_count;
        else end_indices = neighbor_start_indices[idx+1]-1;

        int neighbor_size=(end_indices-neighbor_start_indices[idx]+1)/3;
        // if(idx==0||idx==10||idx==20) printf("neighbor(%d) start = %d, end = %d,size = %d\n",idx,neighbor_start_indices[idx],end_indices,neighbor_size);
        // printf("neighbor_size = %d\n", neighbor_size);
        if(neighbor_size>=3){//近傍点数3以上
            long long int covariance_start, covariance_stop;
            long long int eigen_start,eigen_stop;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_start));
            // printf("neighbor_size>=3");
            //平均計算
            float x_average=0,y_average=0,z_average=0;
            // if(idx==output_id) printf("neighbor_size = %d\n",neighbor_size);

            // if(idx==output_id) printf("neighbor_points(%d) = {\n",idx);
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                // デバッグ用
                // if((idx==0||idx==10||idx==20)&&(i==(neighbor_start_indices[idx]))) printf("neighbor_points_indices(%d) = %d\n",idx,neighbor_points_indices[i]);
                // if((idx==0||idx==10||idx==20)&&i==neighbor_start_indices[idx]) printf("neighbor_points(%d) = %f,%f,%f\n",idx,points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                // if(idx==output_id) printf("{%f, %f, %f},\n",points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                x_average+=points[neighbor_points_indices[i]*3+0];
                y_average+=points[neighbor_points_indices[i]*3+1];
                z_average+=points[neighbor_points_indices[i]*3+2];
            }
            // if(idx==output_id) printf("};\n");
            x_average/=neighbor_size;
            y_average/=neighbor_size;
            z_average/=neighbor_size;

            // //要素計算
            float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                sxx+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+0]-x_average);
                syy+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                szz+=(points[neighbor_points_indices[i]*3+2]-z_average)*(points[neighbor_points_indices[i]*3+2]-z_average);

                sxy+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                sxz+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
                syz+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
            }

            sxx/=neighbor_size;
            syy/=neighbor_size;
            szz/=neighbor_size;
            sxy/=neighbor_size;
            sxz/=neighbor_size;
            syz/=neighbor_size;

            //共分散行列
            float a[3*3]={
                sxx,sxy,sxz,
                sxy,syy,syz,
                sxz,syz,szz,
            };

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_stop));
            covariance_time[idx]=covariance_stop - covariance_start;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_start));
            
            // __syncthreads();
            // if(idx==output_id){
            //     printf("                          %f ,%f ,%f \ncovariance matrix(%d)=   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,idx,sxy,syy,syz,sxz,syz,szz);
            // }

            //固有値計算
            float eigen_vector[3 * 3];
            EigenJacobiMethod(a, eigen_vector, 3);

            // __syncthreads();
            // if(neighbor_size<3){
            //     printf("               %f ,%f ,%f \neigen_value=   %f ,%f ,%f \n               %f ,%f ,%f \n\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7],a[8]);
            // }

            float eigen_value[3];
            eigen_value[0]=a[0];
            eigen_value[1]=a[4];
            eigen_value[2]=a[8];

            int min_eigen_axis=0;
            float min_eigen_value=eigen_value[0];
            float eigen_sum=0;
            for(int i=1;i<3;i++){//x,y,z
                if(eigen_value[i]<min_eigen_value){
                    min_eigen_value=eigen_value[i];
                    min_eigen_axis=i;
                }
                //正規化用にnorm計算しておく
                eigen_sum += eigen_value[i];
            }

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_stop));
            eigen_time[idx]=eigen_stop - eigen_start;

            normals[idx*3+0]=eigen_vector[min_eigen_axis+0];
            normals[idx*3+1]=eigen_vector[min_eigen_axis+3];
            normals[idx*3+2]=eigen_vector[min_eigen_axis+6];

            curvatures[idx]=min_eigen_value/eigen_sum;

            // if(idx==output_id){
            //     printf("normals(%d) = %f, %f, %f\n\n\n\n",idx,normals[idx*3+0],normals[idx*3+1],normals[idx*3+2]);
            //     printf("curvature(%d) = %f\n",idx,curvatures[idx]);
            // }

            //デバッグ用
            // normals[idx*3+0]=idx*10+0;
            // normals[idx*3+1]=idx*10+1;
            // normals[idx*3+2]=idx*10+2;
            // printf("normal_x = %f ,normal_y = %f ,normal_z = %f \n", normals[idx*3+1],normals[idx*3+2],normals[idx*3+3]);
        }
        else{
            normals[idx*3+0]=0;
            normals[idx*3+1]=0;
            normals[idx*3+2]=0;
            curvatures[idx]=0;
            covariance_time[idx]=0;
            eigen_time[idx]=0;
        }
    }
    
}

// __global__ void ChildKernel(void* data)
// {

// 	printf("child : %d, %d\n", blockIdx.x, threadIdx.x);

// }

// __global__ void ParentKernel(void* data)
// {

// 	printf("parent: %d, %d\n", blockIdx.x, threadIdx.x);

// 	ChildKernel<<<1, 2>>>(data);
// 	hipDeviceSynchronize();

// }

// __global__ void KernelFunctionArgumentTypeCheck(node* nodes)
// {
// 	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
//     unsigned int idx = ix;
// 	printf("KernelFunctionArgumentTypeCheck: %d, %d\n", blockIdx.x, threadIdx.x);
// 	for(int i=0;i<3;i++){
// 		nodes[i].parent_id = i; nodes[i].left_id = i; nodes[i].right_id = i; nodes[i].axis = i;
// 	}
// }

// __global__ void MyKernel(float* devPtr, size_t pitch, int width, int height)
// {
//   for (int r = 0; r < height; ++r) {
//     float* row = (float*)((char*)devPtr + r * pitch);
//     for (int c = 0; c < width; ++c) {
//       float element = row[c];
//     }
//   }
// }

extern void ComputeNormals(std::vector<long long int>& neighbor_time,std::vector<int>& point_neighbor,std::vector<std::vector<float>> points_array,std::vector<std::vector<int>> neighbor_points_indices,std::vector<int> neighbor_start_indices,int neighbor_points_count,std::vector<std::vector<float>>& normals_array,std::vector<float>& curvatures_array,std::vector<long long int>& covariance_compute_time,std::vector<long long int>& eigen_compute_time)
{
	//minimum recursive function in cuda
	// int nbytes = 1024;
	// int *data_dev = 0;
	// hipMalloc((void**)&data_dev, nbytes);
	// hipMemset(data_dev, 255, nbytes);
	// ParentKernel<<<1, 2>>>(data_dev);

	//Kernel Function Argument Type Check
	// std::vector<node> h_nodes_test(3);
	// node *d_nodes_test;
	// hipMalloc((void **)&d_nodes_test, 3 * sizeof(node));
	// KernelFunctionArgumentTypeCheck<<<1, 1>>>(d_nodes_test);

	// hipMemcpy(&h_nodes_test[0], d_nodes_test, 3 * sizeof(node), hipMemcpyDeviceToHost);
	// hipFree(d_nodes_test);
	// for(int i=0;i<3;i++){
	// 	std::cout<<"nodes = "<<h_nodes_test[i].parent_id<<","<<h_nodes_test[i].left_id<<","<<h_nodes_test[i].right_id<<","<<h_nodes_test[i].axis<<std::endl;
	// }

	//2d確保
	// int width = 64, height = 64;
	// float* devPtr;
	// size_t pitch;
	// hipMallocPitch(&devPtr, &pitch, width * sizeof(float), height);
	// MyKernel<<<100, 512>>>(devPtr, pitch, width, height);

	// points_array.clear();
	// points_array.resize(9);
	// points_array = {{6, 0, 0}, 
	// 				{5, 3, 0},
	// 				{3, 4, 0},
	// 				{4, 6, 0},
	// 				{2, 5, 0},
	// 				{1, 2, 0},
	// 				{0, 1, 0},
	// 				{0, 0, 0},
	// 				{0, 0, 0}};
	// points_array.resize(7);

	// points_array.resize(77);
	std::vector <node> nodes;
	nodes.resize(points_array.size());
	int root_id=-1;
	clock_t build_start,build_end;
	build_start = clock();
	// if(frames==43) std::cout<<"dead point is ("<<points_array[77][0]<<","<<points_array[77][0]<<","<<points_array[77][0]<<")"<<std::endl;


	/////////////////////////////////////////////////////////////////////////////////////////
	// std::vector<std::vector<int>> axis_sort_ids(3,std::vector<int>(points_array.size()));
	// point_with_id point_with_ids[points_array.size()];
	// for(int i=0;i<points_array.size();i++){
	// 	point_with_ids[i].id = i;
	// 	point_with_ids[i].pos[0] = points_array[i][0];
	// 	point_with_ids[i].pos[1] = points_array[i][1];
	// 	point_with_ids[i].pos[2] = points_array[i][2];
	// }
	// for(sort_axis=0; sort_axis<3; sort_axis++){
	// 	qsort(point_with_ids, points_array.size(), sizeof(point_with_id), AxisSort);
	// 	for (int i=0 ; i < points_array.size() ; i++){
	// 		axis_sort_ids[sort_axis][i]=point_with_ids[i].id;
	// 	}
	// }
	// int create_end = CreateNode(&root_id,points_array.size(),nodes,axis_sort_ids,0,-1,false);
	/////////////////////////////////////////////////////////////////////////////////////////

	/////////////////////////////////////////////////////////////////////////////////////////
	std::vector<int> x_sort_ids(points_array.size());
	std::vector<int> y_sort_ids(points_array.size());
	std::vector<int> z_sort_ids(points_array.size());
	point_with_id point_with_ids[points_array.size()];
	for(int i=0;i<points_array.size();i++){
		point_with_ids[i].id = i;
		point_with_ids[i].pos[0] = points_array[i][0];
		point_with_ids[i].pos[1] = points_array[i][1];
		point_with_ids[i].pos[2] = points_array[i][2];
	}
	for(sort_axis=0; sort_axis<3; sort_axis++){
		qsort(point_with_ids, points_array.size(), sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < points_array.size() ; i++){
			if(sort_axis==0){
				x_sort_ids[i]=point_with_ids[i].id;
			}
			if(sort_axis==1){
				y_sort_ids[i]=point_with_ids[i].id;
			}
			if(sort_axis==2){
				z_sort_ids[i]=point_with_ids[i].id;
			}
		}
	}
	int *d_x_sort_ids,*d_y_sort_ids,*d_z_sort_ids,*d_root_id;
	node *d_nodes;
	hipMalloc((void **)&d_x_sort_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_y_sort_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_z_sort_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_root_id, sizeof(int));
	hipMalloc((void **)&d_nodes, points_array.size() * sizeof(node));
	hipMemcpy(d_x_sort_ids, &x_sort_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_y_sort_ids, &y_sort_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_z_sort_ids, &z_sort_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSetLimit(hipLimitStackSize, 1024*1024);
	// std::cout << "frames" << frames <<"------------------------------------------------------------------------------------------------------------"<< std::endl;
	d_CreateNode<<<1, 1>>>(points_array.size(),points_array.size(),0,-1,false,d_x_sort_ids,d_y_sort_ids,d_z_sort_ids,d_root_id,d_nodes);
	hipMemcpy(&root_id, d_root_id, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&nodes[0], d_nodes, points_array.size() * sizeof(node), hipMemcpyDeviceToHost);
	hipFree(d_x_sort_ids);
	hipFree(d_y_sort_ids);
	hipFree(d_z_sort_ids);
	hipFree(d_root_id);
	hipFree(d_nodes);
	/////////////////////////////////////////////////////////////////////////////////////////
	
	build_end = clock();
	printf("create tree time is %.5fs\n",(double)(build_end-build_start)/CLOCKS_PER_SEC);
	//root_id表示
	std::cout << "root_id = " << root_id << std::endl;
	//nodes表示


	// std::cout<<"3.01"<<std::endl;
    //ホスト1次配列宣言
	//kd
	std::vector<int> h_parent_ids(points_array.size());
	std::vector<int> h_left_ids(points_array.size());
	std::vector<int> h_right_ids(points_array.size());
	std::vector<int> h_axes(points_array.size());

	std::vector<int> h_point_neighbor(points_array.size());
	std::vector<int> h_point_neighbor_size(1);
	std::vector<long long int> h_neighbor_time(points_array.size());

	//normal
    std::vector<float> h_points(points_array.size() * 3);
    std::vector<int> h_neighbor_points_indices(neighbor_points_count);
    std::vector<float> h_normals(points_array.size() * 3);
    std::vector<float> h_curvatures(points_array.size());
    std::vector<long long int> h_covariance_compute_time(points_array.size());
    std::vector<long long int> h_eigen_compute_time(points_array.size());
    // std::cout<<"3.02"<<std::endl;
    //デバイス1次配列宣言
	//kd
	int *d_parent_ids,*d_left_ids,*d_right_ids,*d_axes;
	int *d_point_neighbor,*d_point_neighbor_size;
	long long int *d_neighbor_time;

	//normal
    float *d_points,*d_normals,*d_curvatures;
    int *d_neighbor_points_indices,*d_neighbor_start_indices;
    long long int *d_covariance_compute_time,*d_eigen_compute_time;


    // std::cout<<"3.03"<<std::endl;
    //メモリ確保
	//kd
	hipMalloc((void **)&d_parent_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_left_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_right_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_axes, points_array.size() * sizeof(int));

	hipMalloc((void **)&d_point_neighbor, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_point_neighbor_size, sizeof(int));
	hipMalloc((void **)&d_neighbor_time, points_array.size() * sizeof(long long int));
	//normal
    hipMalloc((void **)&d_points, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_neighbor_points_indices, neighbor_points_count * sizeof(int));
    hipMalloc((void **)&d_neighbor_start_indices, points_array.size() * sizeof(int));
    hipMalloc((void **)&d_normals, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_curvatures, points_array.size() * sizeof(float));
    hipMalloc((void **)&d_covariance_compute_time, points_array.size() * sizeof(long long int));
    hipMalloc((void **)&d_eigen_compute_time, points_array.size() * sizeof(long long int));
    // std::cout<<"3.04"<<std::endl;
    //1次配列化
    int k=0,l=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            h_points[k]=points_array[i][j];
            k++;
        }
        for(int j=0;j<neighbor_points_indices[i].size();j++){//近傍
            h_neighbor_points_indices[l]=neighbor_points_indices[i][j];
            l++;
        }
		h_parent_ids[i]=nodes[i].parent_id;
		h_left_ids[i]=nodes[i].left_id;
		h_right_ids[i]=nodes[i].right_id;
		h_axes[i]=nodes[i].axis;
    }



    // std::cout<<"3.05"<<std::endl;
    //コピー
	//kd
	hipMemcpy(d_parent_ids, &h_parent_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_left_ids, &h_left_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_right_ids, &h_right_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_axes, &h_axes[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	//normal
    hipMemcpy(d_points, &h_points[0], points_array.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_points_indices, &h_neighbor_points_indices[0], neighbor_points_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_start_indices, &neighbor_start_indices[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
    // std::cout<<"3.06"<<std::endl;
    //並列スレッド設定
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((points_array.size() + block.x - 1) / block.x, 1);
    // std::cout<<"3.07"<<std::endl;
    // std::cout<<"normalsGPUstart"<<std::endl;
    // hipDeviceSetLimit(hipLimitStackSize, 1024*8);
    //実行
    NormalsGPU<<<grid,block>>>(d_neighbor_time,d_point_neighbor_size,d_point_neighbor,d_parent_ids,d_left_ids,d_right_ids,d_axes,root_id,d_points,points_array.size(),d_neighbor_points_indices,d_neighbor_start_indices,neighbor_points_count,d_normals,d_curvatures,d_covariance_compute_time,d_eigen_compute_time);
    // std::cout<<"normalsGPUend"<<std::endl;
    // std::cout<<"3.08"<<std::endl;
    //コピー
	//kd
	hipMemcpy(&h_point_neighbor[0], d_point_neighbor, points_array.size() * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_point_neighbor_size[0], d_point_neighbor_size, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_neighbor_time[0], d_neighbor_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
	//normal
    hipMemcpy(&h_normals[0], d_normals, points_array.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_curvatures[0], d_curvatures, points_array.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_covariance_compute_time[0], d_covariance_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_compute_time[0], d_eigen_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    // std::cout<<"3.09"<<std::endl;
    //2次配列化
	for(int i=0;i<h_point_neighbor_size[0];i++){
		point_neighbor[i]=h_point_neighbor[i];
	}
	
	point_neighbor.resize(h_point_neighbor_size[0]);
	// std::cout<<"host cu size "<<h_point_neighbor_size[0]<<std::endl;

    k=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            normals_array[i][j]=h_normals[k];
            k++;
        }
        curvatures_array[i]=h_curvatures[i];
        covariance_compute_time[i]=h_covariance_compute_time[i];
        eigen_compute_time[i]=h_eigen_compute_time[i];
		neighbor_time[i]=h_neighbor_time[i];
    }
    // std::cout<<"cu_normals : "<<normals_array[0][0]<<","<<normals_array[0][1]<<","<<normals_array[0][2]<<std::endl;
    // std::cout<<"3.10"<<std::endl;
    //メモリ解放
	//kd
	hipFree(d_parent_ids);
	hipFree(d_left_ids);
	hipFree(d_right_ids);
	hipFree(d_axes);

	hipFree(d_point_neighbor);
	hipFree(d_point_neighbor_size);
	hipFree(d_neighbor_time);

	//normal
    hipFree(d_points);
    hipFree(d_neighbor_points_indices);
    hipFree(d_neighbor_start_indices);
    hipFree(d_normals);
    hipFree(d_curvatures);
    hipFree(d_covariance_compute_time);
    hipFree(d_eigen_compute_time);
	frames++;
}