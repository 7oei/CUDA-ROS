#include "hip/hip_runtime.h"


#include <string.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kdtree_cuda/kdtree_cuda.hpp"
#include <vector>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

int sort_axis=0;

typedef struct
{
	int	id;
	float pos[3];
} point_with_id;

//	年齢(昇順)
int axis_sort(const void * n1, const void * n2)
{
	if (((point_with_id *)n1)->pos[sort_axis] > ((point_with_id *)n2)->pos[sort_axis])
	{
		return 1;
	}
	else if (((point_with_id *)n1)->pos[sort_axis] < ((point_with_id *)n2)->pos[sort_axis])
	{
		return -1;
	}
	else
	{
		return 0;
	}
}

__device__ int eigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 

__global__ void normalsGPU(float* points,int point_size,int* neighbor_points_indices,int* neighbor_start_indices,int neighbor_points_count,float* normals,float* curvatures,long long int* covariance_time,long long int* eigen_time) {
    // printf("normalsGPU");
    //インデックス取得
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
    unsigned int output_id=50;
    // printf("idx = %d ,", idx);


    if(idx<point_size-1){//対象スレッド内のみ計算
        //デバッグ用
        // if(idx==0||idx==10||idx==20) printf("points(%d) = %f,%f,%f\n",idx,points[idx*3+0],points[idx*3+1],points[idx*3+2]);
        
        // printf("idx<point_size");
        //近傍点終点インデックスの定義
        int end_indices;
        if(idx==(point_size-1)) end_indices = neighbor_points_count;
        else end_indices = neighbor_start_indices[idx+1]-1;

        int neighbor_size=(end_indices-neighbor_start_indices[idx]+1)/3;
        // if(idx==0||idx==10||idx==20) printf("neighbor(%d) start = %d, end = %d,size = %d\n",idx,neighbor_start_indices[idx],end_indices,neighbor_size);
        // printf("neighbor_size = %d\n", neighbor_size);
        if(neighbor_size>=3){//近傍点数3以上
            long long int covariance_start, covariance_stop;
            long long int eigen_start,eigen_stop;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_start));
            // printf("neighbor_size>=3");
            //平均計算
            float x_average=0,y_average=0,z_average=0;
            // if(idx==output_id) printf("neighbor_size = %d\n",neighbor_size);

            // if(idx==output_id) printf("neighbor_points(%d) = {\n",idx);
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                // デバッグ用
                // if((idx==0||idx==10||idx==20)&&(i==(neighbor_start_indices[idx]))) printf("neighbor_points_indices(%d) = %d\n",idx,neighbor_points_indices[i]);
                // if((idx==0||idx==10||idx==20)&&i==neighbor_start_indices[idx]) printf("neighbor_points(%d) = %f,%f,%f\n",idx,points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                // if(idx==output_id) printf("{%f, %f, %f},\n",points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                x_average+=points[neighbor_points_indices[i]*3+0];
                y_average+=points[neighbor_points_indices[i]*3+1];
                z_average+=points[neighbor_points_indices[i]*3+2];
            }
            // if(idx==output_id) printf("};\n");
            x_average/=neighbor_size;
            y_average/=neighbor_size;
            z_average/=neighbor_size;

            // //要素計算
            float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                sxx+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+0]-x_average);
                syy+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                szz+=(points[neighbor_points_indices[i]*3+2]-z_average)*(points[neighbor_points_indices[i]*3+2]-z_average);

                sxy+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                sxz+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
                syz+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
            }

            sxx/=neighbor_size;
            syy/=neighbor_size;
            szz/=neighbor_size;
            sxy/=neighbor_size;
            sxz/=neighbor_size;
            syz/=neighbor_size;

            //共分散行列
            float a[3*3]={
                sxx,sxy,sxz,
                sxy,syy,syz,
                sxz,syz,szz,
            };

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_stop));
            covariance_time[idx]=covariance_stop - covariance_start;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_start));
            
            // __syncthreads();
            // if(idx==output_id){
            //     printf("                          %f ,%f ,%f \ncovariance matrix(%d)=   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,idx,sxy,syy,syz,sxz,syz,szz);
            // }

            //固有値計算
            float eigen_vector[3 * 3];
            eigenJacobiMethod(a, eigen_vector, 3);

            // __syncthreads();
            // if(neighbor_size<3){
            //     printf("               %f ,%f ,%f \neigen_value=   %f ,%f ,%f \n               %f ,%f ,%f \n\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7],a[8]);
            // }

            float eigen_value[3];
            eigen_value[0]=a[0];
            eigen_value[1]=a[4];
            eigen_value[2]=a[8];

            int min_eigen_axis=0;
            float min_eigen_value=eigen_value[0];
            float eigen_sum=0;
            for(int i=1;i<3;i++){//x,y,z
                if(eigen_value[i]<min_eigen_value){
                    min_eigen_value=eigen_value[i];
                    min_eigen_axis=i;
                }
                //正規化用にnorm計算しておく
                eigen_sum += eigen_value[i];
            }

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_stop));
            eigen_time[idx]=eigen_stop - eigen_start;

            normals[idx*3+0]=eigen_vector[min_eigen_axis+0];
            normals[idx*3+1]=eigen_vector[min_eigen_axis+3];
            normals[idx*3+2]=eigen_vector[min_eigen_axis+6];

            curvatures[idx]=min_eigen_value/eigen_sum;

            // if(idx==output_id){
            //     printf("normals(%d) = %f, %f, %f\n\n\n\n",idx,normals[idx*3+0],normals[idx*3+1],normals[idx*3+2]);
            //     printf("curvature(%d) = %f\n",idx,curvatures[idx]);
            // }

            //デバッグ用
            // normals[idx*3+0]=idx*10+0;
            // normals[idx*3+1]=idx*10+1;
            // normals[idx*3+2]=idx*10+2;
            // printf("normal_x = %f ,normal_y = %f ,normal_z = %f \n", normals[idx*3+1],normals[idx*3+2],normals[idx*3+3]);
        }
        else{
            normals[idx*3+0]=0;
            normals[idx*3+1]=0;
            normals[idx*3+2]=0;
            curvatures[idx]=0;
            covariance_time[idx]=0;
            eigen_time[idx]=0;
        }
    }
    
}

extern void compute_normals(std::vector<std::vector<float>> points_array,std::vector<std::vector<int>> neighbor_points_indices,std::vector<int> neighbor_start_indices,int neighbor_points_count,std::vector<std::vector<float>>& normals_array,std::vector<float>& curvatures_array,std::vector<long long int>& covariance_compute_time,std::vector<long long int>& eigen_compute_time){
	point_with_id point_with_ids[] =
	{
		{0,3,1,4},
		{1,8,3,6},
		{2,6,2,8},
		{3,4,3,8},
	};

	float max[3],min[3],median[3],length[3];
	int median_id[3];
	for(sort_axis=0; sort_axis<3; sort_axis++){//x,y,zそれぞれにソート
		std::cout<<"sort_axis = "<<sort_axis<<std::endl;

		qsort(point_with_ids, sizeof(point_with_ids) / sizeof(point_with_ids[0]), sizeof(point_with_id), axis_sort);
		for (int j=0 ; j < sizeof(point_with_ids) / sizeof(point_with_ids[0]) ; j++)
		{
			printf("%d, %f, %f, %f \n", point_with_ids[j].id, point_with_ids[j].pos[0], point_with_ids[j].pos[1], point_with_ids[j].pos[2]);
		}
		std::cout<<std::endl;
		//max,min,median,median_id取得
		max[sort_axis]=point_with_ids[(sizeof(point_with_ids) / sizeof(point_with_ids[0]))-1].pos[sort_axis];//minとmaxいらんかも
		min[sort_axis]=point_with_ids[0].pos[sort_axis];
		length[sort_axis]=max[sort_axis]-min[sort_axis];
		median[sort_axis]=point_with_ids[((sizeof(point_with_ids) / sizeof(point_with_ids[0]))-1)/2].pos[sort_axis];//偶数なら小さい方(-1)消せば大きい方
		median_id[sort_axis]=point_with_ids[((sizeof(point_with_ids) / sizeof(point_with_ids[0]))-1)/2].id;
	}
	std::cout<<"x_length = "<< length[0] <<", x_median["<<median_id[0]<<"] = "<<median[0]<<std::endl;
	std::cout<<"y_length = "<< length[1] <<", y_median["<<median_id[1]<<"] = "<<median[1]<<std::endl;
	std::cout<<"z_length = "<< length[2] <<", z_median["<<median_id[2]<<"] = "<<median[2]<<std::endl;
	std::cout<<std::endl;
	std::cout<<std::endl;
	std::cout<<std::endl;
    // std::cout<<"3.01"<<std::endl;
    //ホスト1次配列宣言
    std::vector<float> h_points(points_array.size() * 3);
    std::vector<int> h_neighbor_points_indices(neighbor_points_count);
    std::vector<float> h_normals(points_array.size() * 3);
    std::vector<float> h_curvatures(points_array.size());
    std::vector<long long int> h_covariance_compute_time(points_array.size());
    std::vector<long long int> h_eigen_compute_time(points_array.size());
    // std::cout<<"3.02"<<std::endl;
    //デバイス1次配列宣言
    float *d_points,*d_normals,*d_curvatures;
    int *d_neighbor_points_indices,*d_neighbor_start_indices;
    long long int *d_covariance_compute_time,*d_eigen_compute_time;
    // std::cout<<"3.03"<<std::endl;
    //メモリ確保
    hipMalloc((void **)&d_points, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_neighbor_points_indices, neighbor_points_count * sizeof(int));
    hipMalloc((void **)&d_neighbor_start_indices, points_array.size() * sizeof(int));
    hipMalloc((void **)&d_normals, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_curvatures, points_array.size() * sizeof(float));
    hipMalloc((void **)&d_covariance_compute_time, points_array.size() * sizeof(long long int));
    hipMalloc((void **)&d_eigen_compute_time, points_array.size() * sizeof(long long int));
    // std::cout<<"3.04"<<std::endl;
    //1次配列化
    int k=0,l=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            h_points[k]=points_array[i][j];
            k++;
        }
        for(int j=0;j<neighbor_points_indices[i].size();j++){//近傍
            h_neighbor_points_indices[l]=neighbor_points_indices[i][j];
            l++;
        }
    }
    // std::cout<<"3.05"<<std::endl;
    //コピー
    hipMemcpy(d_points, &h_points[0], points_array.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_points_indices, &h_neighbor_points_indices[0], neighbor_points_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_start_indices, &neighbor_start_indices[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
    // std::cout<<"3.06"<<std::endl;
    //並列スレッド設定
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((points_array.size() + block.x - 1) / block.x, 1);
    // std::cout<<"3.07"<<std::endl;
    // std::cout<<"normalsGPUstart"<<std::endl;
    hipDeviceSetLimit(hipLimitStackSize, 1024*8);
    //実行
    normalsGPU<<<grid,block>>>(d_points,points_array.size(),d_neighbor_points_indices,d_neighbor_start_indices,neighbor_points_count,d_normals,d_curvatures,d_covariance_compute_time,d_eigen_compute_time);
    // std::cout<<"normalsGPUend"<<std::endl;
    // std::cout<<"3.08"<<std::endl;
    //コピー
    hipMemcpy(&h_normals[0], d_normals, points_array.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_curvatures[0], d_curvatures, points_array.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_covariance_compute_time[0], d_covariance_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_compute_time[0], d_eigen_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    // std::cout<<"3.09"<<std::endl;
    //2次配列化
    k=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            normals_array[i][j]=h_normals[k];
            k++;
        }
        curvatures_array[i]=h_curvatures[i];
        covariance_compute_time[i]=h_covariance_compute_time[i];
        eigen_compute_time[i]=h_eigen_compute_time[i];
    }
    // std::cout<<"cu_normals : "<<normals_array[0][0]<<","<<normals_array[0][1]<<","<<normals_array[0][2]<<std::endl;
    // std::cout<<"3.10"<<std::endl;
    //メモリ解放
    hipFree(d_points);
    hipFree(d_neighbor_points_indices);
    hipFree(d_neighbor_start_indices);
    hipFree(d_normals);
    hipFree(d_curvatures);
    hipFree(d_covariance_compute_time);
    hipFree(d_eigen_compute_time);
}