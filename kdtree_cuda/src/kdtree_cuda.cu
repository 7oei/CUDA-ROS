#include "hip/hip_runtime.h"


#include <string.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kdtree_cuda/kdtree_cuda.hpp"
#include <vector>
#include <iostream>
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include <string.h>
#include <math.h>

struct kdhyperrect {
	int dim;
	double *min, *max;              /* minimum/maximum coords */
};

struct kdnode {
	double *pos;
	int dir;
	void *data;

	struct kdnode *left, *right;	/* negative/positive side */
};

struct res_node {
	struct kdnode *item;
	double dist_sq;
	struct res_node *next;
};

struct kdtree {
	int dim;
	struct kdnode *root;
	struct kdhyperrect *rect;
	void (*destr)(void*);
};

struct kdres {
	struct kdtree *tree;
	struct res_node *rlist, *riter;
	int size;
};

#define SQ(x)			((x) * (x))

//device
__device__ struct kdtree *d_kd_create(int k);
__device__ void d_kd_free(struct kdtree *tree);
__device__ void d_kd_clear(struct kdtree *tree);
__device__ void d_kd_data_destructor(struct kdtree *tree, void (*destr)(void*));

__device__ int d_kd_insert(struct kdtree *tree, const double *pos, void *data);
__device__ int d_kd_insertf(struct kdtree *tree, const float *pos, void *data);
__device__ int d_kd_insert3(struct kdtree *tree, double x, double y, double z, void *data);
__device__ int d_kd_insert3f(struct kdtree *tree, float x, float y, float z, void *data);

__device__ static int d_find_nearest(struct kdnode *node, const double *pos, double range, struct res_node *list, int ordered, int dim);

#if 0
__device__ static int d_find_nearest_n(struct kdnode *node, const double *pos, double range, int num, struct rheap *heap, int dim);
#endif

__device__ struct kdres *d_kd_nearest(struct kdtree *tree, const double *pos);
__device__ struct kdres *d_kd_nearestf(struct kdtree *tree, const float *pos);
__device__ struct kdres *d_kd_nearest3(struct kdtree *tree, double x, double y, double z);
__device__ struct kdres *d_kd_nearest3f(struct kdtree *tree, float x, float y, float z);

__device__ struct kdres *d_kd_nearest_range(struct kdtree *tree, const double *pos, double range);
__device__ struct kdres *d_kd_nearest_rangef(struct kdtree *tree, const float *pos, float range);
__device__ struct kdres *d_kd_nearest_range3(struct kdtree *tree, double x, double y, double z, double range);
__device__ struct kdres *d_kd_nearest_range3f(struct kdtree *tree, float x, float y, float z, float range);

__device__ void d_kd_res_free(struct kdres *set);
__device__ int d_kd_res_size(struct kdres *set);
__device__ void d_kd_res_rewind(struct kdres *set);
__device__ int d_kd_res_end(struct kdres *set);
__device__ int d_kd_res_next(struct kdres *set);

__device__ void *d_kd_res_item(struct kdres *set, double *pos);
__device__ void *d_kd_res_itemf(struct kdres *set, float *pos);
__device__ void *d_kd_res_item3(struct kdres *set, double *x, double *y, double *z);
__device__ void *d_kd_res_item3f(struct kdres *set, float *x, float *y, float *z);
__device__ void *d_kd_res_item_data(struct kdres *set);

__device__ static void d_clear_rec(struct kdnode *node, void (*destr)(void*));
__device__ static int d_insert_rec(struct kdnode **node, const double *pos, void *data, int dir, int dim);
__device__ static int d_rlist_insert(struct res_node *list, struct kdnode *item, double dist_sq);
__device__ static void d_clear_results(struct kdres *set);

__device__ static struct kdhyperrect* d_hyperrect_create(int dim, const double *min, const double *max);
__device__ static void d_hyperrect_free(struct kdhyperrect *rect);
__device__ static struct kdhyperrect* d_hyperrect_duplicate(const struct kdhyperrect *rect);
__device__ static void d_hyperrect_extend(struct kdhyperrect *rect, const double *pos);
__device__ static double d_hyperrect_dist_sq(struct kdhyperrect *rect, const double *pos);

//host
struct kdtree *h_kd_create(int k);
void h_kd_free(struct kdtree *tree);
void h_kd_clear(struct kdtree *tree);
void h_kd_data_destructor(struct kdtree *tree, void (*destr)(void*));

int h_kd_insert(struct kdtree *tree, const double *pos, void *data);
int h_kd_insertf(struct kdtree *tree, const float *pos, void *data);
int h_kd_insert3(struct kdtree *tree, double x, double y, double z, void *data);
int h_kd_insert3f(struct kdtree *tree, float x, float y, float z, void *data);

static int h_find_nearest(struct kdnode *node, const double *pos, double range, struct res_node *list, int ordered, int dim);

#if 0
static int h_find_nearest_n(struct kdnode *node, const double *pos, double range, int num, struct rheap *heap, int dim);
#endif

struct kdres *h_kd_nearest(struct kdtree *tree, const double *pos);
struct kdres *h_kd_nearestf(struct kdtree *tree, const float *pos);
struct kdres *h_kd_nearest3(struct kdtree *tree, double x, double y, double z);
struct kdres *h_kd_nearest3f(struct kdtree *tree, float x, float y, float z);

struct kdres *h_kd_nearest_range(struct kdtree *tree, const double *pos, double range);
struct kdres *h_kd_nearest_rangef(struct kdtree *tree, const float *pos, float range);
struct kdres *h_kd_nearest_range3(struct kdtree *tree, double x, double y, double z, double range);
struct kdres *h_kd_nearest_range3f(struct kdtree *tree, float x, float y, float z, float range);

void h_kd_res_free(struct kdres *set);
int h_kd_res_size(struct kdres *set);
void h_kd_res_rewind(struct kdres *set);
int h_kd_res_end(struct kdres *set);
int h_kd_res_next(struct kdres *set);

void *h_kd_res_item(struct kdres *set, double *pos);
void *h_kd_res_itemf(struct kdres *set, float *pos);
void *h_kd_res_item3(struct kdres *set, double *x, double *y, double *z);
void *h_kd_res_item3f(struct kdres *set, float *x, float *y, float *z);
void *h_kd_res_item_data(struct kdres *set);

static void h_clear_rec(struct kdnode *node, void (*destr)(void*));
static int h_insert_rec(struct kdnode **node, const double *pos, void *data, int dir, int dim);
static int h_rlist_insert(struct res_node *list, struct kdnode *item, double dist_sq);
static void h_clear_results(struct kdres *set);

static struct kdhyperrect* h_hyperrect_create(int dim, const double *min, const double *max);
static void h_hyperrect_free(struct kdhyperrect *rect);
static struct kdhyperrect* h_hyperrect_duplicate(const struct kdhyperrect *rect);
static void h_hyperrect_extend(struct kdhyperrect *rect, const double *pos);
static double h_hyperrect_dist_sq(struct kdhyperrect *rect, const double *pos);


//device
__device__ struct kdtree *d_kd_create(int k)
{
	struct kdtree *tree;

	if(!(tree = (struct kdtree *)malloc(sizeof *tree))) {
		return 0;
	}

	tree->dim = k;
	tree->root = 0;
	tree->destr = 0;
	tree->rect = 0;

	return tree;
}

__device__ void d_kd_free(struct kdtree *tree)
{
	if(tree) {
		d_kd_clear(tree);
		free(tree);
	}
}

__device__ static void d_clear_rec(struct kdnode *node, void (*destr)(void*))
{
	if(!node) return;

	d_clear_rec(node->left, destr);
	d_clear_rec(node->right, destr);
	
	if(destr) {
		destr(node->data);
	}
	free(node->pos);
	free(node);
}

__device__ void d_kd_clear(struct kdtree *tree)
{
	d_clear_rec(tree->root, tree->destr);
	tree->root = 0;

	if (tree->rect) {
		d_hyperrect_free(tree->rect);
		tree->rect = 0;
	}
}

__device__ void d_kd_data_destructor(struct kdtree *tree, void (*destr)(void*))
{
	tree->destr = destr;
}


__device__ static int d_insert_rec(struct kdnode **nptr, const double *pos, void *data, int dir, int dim)
{
	int new_dir;
	struct kdnode *node;

	if(!*nptr) {
		if(!(node = (struct kdnode *)malloc(sizeof *node))) {
			return -1;
		}
		if(!(node->pos = (double *)malloc(dim * sizeof *node->pos))) {
			free(node);
			return -1;
		}
		memcpy(node->pos, pos, dim * sizeof *node->pos);
		node->data = data;
		node->dir = dir;
		node->left = node->right = 0;
		*nptr = node;
		return 0;
	}

	node = *nptr;
	new_dir = (node->dir + 1) % dim;
	if(pos[node->dir] < node->pos[node->dir]) {
		return d_insert_rec(&(*nptr)->left, pos, data, new_dir, dim);
	}
	return d_insert_rec(&(*nptr)->right, pos, data, new_dir, dim);
}

__device__ int d_kd_insert(struct kdtree *tree, const double *pos, void *data)
{
	if (d_insert_rec(&tree->root, pos, data, 0, tree->dim)) {
		return -1;
	}

	if (tree->rect == 0) {
		tree->rect = d_hyperrect_create(tree->dim, pos, pos);
	} else {
		d_hyperrect_extend(tree->rect, pos);
	}

	return 0;
}

__device__ int d_kd_insertf(struct kdtree *tree, const float *pos, void *data)
{
	static double sbuf[16];
	double *bptr, *buf = 0;
	int res, dim = tree->dim;

	if(dim > 16) {
        if(!(bptr = buf = (double *)malloc(dim * sizeof *bptr))) {
            return -1;
        }
	} else {
		bptr = buf = sbuf;
	}

	while(dim-- > 0) {
		*bptr++ = *pos++;
	}

	res = d_kd_insert(tree, buf, data);
	if(tree->dim > 16) free(buf);
	return res;
}

__device__ int d_kd_insert3(struct kdtree *tree, double x, double y, double z, void *data)
{
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return d_kd_insert(tree, buf, data);
}

__device__ int d_kd_insert3f(struct kdtree *tree, float x, float y, float z, void *data)
{
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return d_kd_insert(tree, buf, data);
}

__device__ static int d_find_nearest(struct kdnode *node, const double *pos, double range, struct res_node *list, int ordered, int dim)
{
    printf("d_find_nearest open\n");
	double dist_sq, dx;
	int i, ret, added_res = 0;

	if(!node) return 0;
    printf("fn 1\n");
	dist_sq = 0;
	for(i=0; i<dim; i++) {
		dist_sq += SQ(node->pos[i] - pos[i]);
	}
    printf("fn 2\n");
	if(dist_sq <= SQ(range)) {
		if(d_rlist_insert(list, node, ordered ? dist_sq : -1.0) == -1) {
			return -1;
		}
		added_res = 1;
	}
    printf("fn 3\n");

	dx = pos[node->dir] - node->pos[node->dir];
    printf("fn 4\n");
	ret = d_find_nearest(dx <= 0.0 ? node->left : node->right, pos, range, list, ordered, dim);
    printf("fn 5\n");
	if(ret >= 0 && fabs(dx) < range) {
		added_res += ret;
		ret = d_find_nearest(dx <= 0.0 ? node->right : node->left, pos, range, list, ordered, dim);
	}
    printf("fn 6\n");
	if(ret == -1) {
		return -1;
	}
    printf("fn 7\n");
	added_res += ret;
    printf("fn 8\n");
	return added_res;
}

#if 0
__device__ static int d_find_nearest_n(struct kdnode *node, const double *pos, double range, int num, struct rheap *heap, int dim)
{
	double dist_sq, dx;
	int i, ret, added_res = 0;

	if(!node) return 0;
	
	/* if the photon is close enough, add it to the result heap */
	dist_sq = 0;
	for(i=0; i<dim; i++) {
		dist_sq += SQ(node->pos[i] - pos[i]);
	}
	if(dist_sq <= range_sq) {
		if(heap->size >= num) {
			/* get furthest element */
			struct res_node *maxelem = rheap_get_max(heap);

			/* and check if the new one is closer than that */
			if(maxelem->dist_sq > dist_sq) {
				rheap_remove_max(heap);

				if(rheap_insert(heap, node, dist_sq) == -1) {
					return -1;
				}
				added_res = 1;

				range_sq = dist_sq;
			}
		} else {
			if(rheap_insert(heap, node, dist_sq) == -1) {
				return =1;
			}
			added_res = 1;
		}
	}


	/* find signed distance from the splitting plane */
	dx = pos[node->dir] - node->pos[node->dir];

	ret = d_find_nearest_n(dx <= 0.0 ? node->left : node->right, pos, range, num, heap, dim);
	if(ret >= 0 && fabs(dx) < range) {
		added_res += ret;
		ret = d_find_nearest_n(dx <= 0.0 ? node->right : node->left, pos, range, num, heap, dim);
	}

}
#endif

__device__ static void d_kd_nearest_i(struct kdnode *node, const double *pos, struct kdnode **result, double *result_dist_sq, struct kdhyperrect* rect)
{
	int dir = node->dir;
	int i;
	double dummy, dist_sq;
	struct kdnode *nearer_subtree, *farther_subtree;
	double *nearer_hyperrect_coord, *farther_hyperrect_coord;

	/* Decide whether to go left or right in the tree */
	dummy = pos[dir] - node->pos[dir];
	if (dummy <= 0) {
		nearer_subtree = node->left;
		farther_subtree = node->right;
		nearer_hyperrect_coord = rect->max + dir;
		farther_hyperrect_coord = rect->min + dir;
	} else {
		nearer_subtree = node->right;
		farther_subtree = node->left;
		nearer_hyperrect_coord = rect->min + dir;
		farther_hyperrect_coord = rect->max + dir;
	}

	if (nearer_subtree) {
		/* Slice the hyperrect to get the hyperrect of the nearer subtree */
		dummy = *nearer_hyperrect_coord;
		*nearer_hyperrect_coord = node->pos[dir];
		/* Recurse down into nearer subtree */
		d_kd_nearest_i(nearer_subtree, pos, result, result_dist_sq, rect);
		/* Undo the slice */
		*nearer_hyperrect_coord = dummy;
	}

	/* Check the distance of the point at the current node, compare it
	 * with our best so far */
	dist_sq = 0;
	for(i=0; i < rect->dim; i++) {
		dist_sq += SQ(node->pos[i] - pos[i]);
	}
	if (dist_sq < *result_dist_sq) {
		*result = node;
		*result_dist_sq = dist_sq;
	}

	if (farther_subtree) {
		/* Get the hyperrect of the farther subtree */
		dummy = *farther_hyperrect_coord;
		*farther_hyperrect_coord = node->pos[dir];
		/* Check if we have to recurse down by calculating the closest
		 * point of the hyperrect and see if it's closer than our
		 * minimum distance in result_dist_sq. */
		if (d_hyperrect_dist_sq(rect, pos) < *result_dist_sq) {
			/* Recurse down into farther subtree */
			d_kd_nearest_i(farther_subtree, pos, result, result_dist_sq, rect);
		}
		/* Undo the slice on the hyperrect */
		*farther_hyperrect_coord = dummy;
	}
}

__device__ struct kdres *d_kd_nearest(struct kdtree *kd, const double *pos)
{
	struct kdhyperrect *rect;
	struct kdnode *result;
	struct kdres *rset;
	double dist_sq;
	int i;

	if (!kd) return 0;
	if (!kd->rect) return 0;

	/* Allocate result set */
	if(!(rset = (struct kdres *)malloc(sizeof *rset))) {
		return 0;
	}
	if(!(rset->rlist = (struct res_node *)malloc(sizeof(struct res_node)))) {
		free(rset);
		return 0;
	}
	rset->rlist->next = 0;
	rset->tree = kd;

	/* Duplicate the bounding hyperrectangle, we will work on the copy */
	if (!(rect = d_hyperrect_duplicate(kd->rect))) {
		d_kd_res_free(rset);
		return 0;
	}

	/* Our first guesstimate is the root node */
	result = kd->root;
	dist_sq = 0;
	for (i = 0; i < kd->dim; i++)
		dist_sq += SQ(result->pos[i] - pos[i]);

	/* Search for the nearest neighbour recursively */
	d_kd_nearest_i(kd->root, pos, &result, &dist_sq, rect);

	/* Free the copy of the hyperrect */
	d_hyperrect_free(rect);

	/* Store the result */
	if (result) {
		if (d_rlist_insert(rset->rlist, result, -1.0) == -1) {
			d_kd_res_free(rset);
			return 0;
		}
		rset->size = 1;
		d_kd_res_rewind(rset);
		return rset;
	} else {
		d_kd_res_free(rset);
		return 0;
	}
}

__device__ struct kdres *d_kd_nearestf(struct kdtree *tree, const float *pos)
{
	static double sbuf[16];
	double *bptr, *buf = 0;
	int dim = tree->dim;
	struct kdres *res;

	if(dim > 16) {
        if(!(bptr = buf = (double *)malloc(dim * sizeof *bptr))) {
            return 0;
        }
	} else {
		bptr = buf = sbuf;
	}

	while(dim-- > 0) {
		*bptr++ = *pos++;
	}

	res = d_kd_nearest(tree, buf);
	if(tree->dim > 16) free(buf);
	return res;
}

__device__ struct kdres *d_kd_nearest3(struct kdtree *tree, double x, double y, double z)
{
	double pos[3];
	pos[0] = x;
	pos[1] = y;
	pos[2] = z;
	return d_kd_nearest(tree, pos);
}

__device__ struct kdres *d_kd_nearest3f(struct kdtree *tree, float x, float y, float z)
{
	double pos[3];
	pos[0] = x;
	pos[1] = y;
	pos[2] = z;
	return d_kd_nearest(tree, pos);
}

__device__ struct kdres *d_kd_nearest_range(struct kdtree *kd, const double *pos, double range)
{
    printf("d_kd_nearest_range open\n");
	int ret;
	struct kdres *rset;

	if(!(rset = (struct kdres *)malloc(sizeof *rset))) {
		return 0;
	}
	if(!(rset->rlist = (struct res_node *)malloc(sizeof(struct res_node)))) {
		free(rset);
		return 0;
	}
	rset->rlist->next = 0;
	rset->tree = kd;

	if((ret = d_find_nearest(kd->root, pos, range, rset->rlist, 0, kd->dim)) == -1) {
		d_kd_res_free(rset);
		return 0;
	}
    printf("fn close\n");
	rset->size = ret;
	d_kd_res_rewind(rset);
	return rset;
}

__device__ struct kdres *d_kd_nearest_rangef(struct kdtree *kd, const float *pos, float range)
{
	static double sbuf[16];
	double *bptr, *buf = 0;
	int dim = kd->dim;
	struct kdres *res;

	if(dim > 16) {
        if(!(bptr = buf = (double *)malloc(dim * sizeof *bptr))) {
            return 0;
        }
	} else {
		bptr = buf = sbuf;
	}

	while(dim-- > 0) {
		*bptr++ = *pos++;
	}

	res = d_kd_nearest_range(kd, buf, range);
	if(kd->dim > 16) free(buf);
	return res;
}

__device__ struct kdres *d_kd_nearest_range3(struct kdtree *tree, double x, double y, double z, double range)
{
    printf("d_kd_nearest_range3 open\n");
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return d_kd_nearest_range(tree, buf, range);
}

__device__ struct kdres *d_kd_nearest_range3f(struct kdtree *tree, float x, float y, float z, float range)
{
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return d_kd_nearest_range(tree, buf, range);
}

__device__ void d_kd_res_free(struct kdres *rset)
{
    printf("d_kd_res_free open\n");
	d_clear_results(rset);
	free(rset->rlist);
	free(rset);
}

__device__ int d_kd_res_size(struct kdres *set)
{
	return (set->size);
}

__device__ void d_kd_res_rewind(struct kdres *rset)
{
    printf("d_kd_res_rewind open\n");
	rset->riter = rset->rlist->next;
}

__device__ int d_kd_res_end(struct kdres *rset)
{
	return rset->riter == 0;
}

__device__ int d_kd_res_next(struct kdres *rset)
{
	rset->riter = rset->riter->next;
	return rset->riter != 0;
}

__device__ void *d_kd_res_item(struct kdres *rset, double *pos)
{
	if(rset->riter) {
		if(pos) {
			memcpy(pos, rset->riter->item->pos, rset->tree->dim * sizeof *pos);
		}
		return rset->riter->item->data;
	}
	return 0;
}

__device__ void *d_kd_res_itemf(struct kdres *rset, float *pos)
{
	if(rset->riter) {
		if(pos) {
			int i;
			for(i=0; i<rset->tree->dim; i++) {
				pos[i] = rset->riter->item->pos[i];
			}
		}
		return rset->riter->item->data;
	}
	return 0;
}

__device__ void *d_kd_res_item3(struct kdres *rset, double *x, double *y, double *z)
{
	if(rset->riter) {
		if(x) *x = rset->riter->item->pos[0];
		if(y) *y = rset->riter->item->pos[1];
		if(z) *z = rset->riter->item->pos[2];
		return rset->riter->item->data;
	}
	return 0;
}

__device__ void *d_kd_res_item3f(struct kdres *rset, float *x, float *y, float *z)
{
	if(rset->riter) {
		if(x) *x = rset->riter->item->pos[0];
		if(y) *y = rset->riter->item->pos[1];
		if(z) *z = rset->riter->item->pos[2];
		return rset->riter->item->data;
	}
	return 0;
}

__device__ void *d_kd_res_item_data(struct kdres *set)
{
	return d_kd_res_item(set, 0);
}

/* ---- hyperrectangle helpers ---- */
__device__ static struct kdhyperrect* d_hyperrect_create(int dim, const double *min, const double *max)
{
	size_t size = dim * sizeof(double);
	struct kdhyperrect* rect = 0;

	if (!(rect = (struct kdhyperrect*)malloc(sizeof(struct kdhyperrect)))) {
		return 0;
	}

	rect->dim = dim;
	if (!(rect->min = (double *)malloc(size))) {
		free(rect);
		return 0;
	}
	if (!(rect->max = (double *)malloc(size))) {
		free(rect->min);
		free(rect);
		return 0;
	}
	memcpy(rect->min, min, size);
	memcpy(rect->max, max, size);

	return rect;
}

__device__ static void d_hyperrect_free(struct kdhyperrect *rect)
{
	free(rect->min);
	free(rect->max);
	free(rect);
}

__device__ static struct kdhyperrect* d_hyperrect_duplicate(const struct kdhyperrect *rect)
{
	return d_hyperrect_create(rect->dim, rect->min, rect->max);
}

__device__ static void d_hyperrect_extend(struct kdhyperrect *rect, const double *pos)
{
	int i;

	for (i=0; i < rect->dim; i++) {
		if (pos[i] < rect->min[i]) {
			rect->min[i] = pos[i];
		}
		if (pos[i] > rect->max[i]) {
			rect->max[i] = pos[i];
		}
	}
}

__device__ static double d_hyperrect_dist_sq(struct kdhyperrect *rect, const double *pos)
{
	int i;
	double result = 0;

	for (i=0; i < rect->dim; i++) {
		if (pos[i] < rect->min[i]) {
			result += SQ(rect->min[i] - pos[i]);
		} else if (pos[i] > rect->max[i]) {
			result += SQ(rect->max[i] - pos[i]);
		}
	}

	return result;
}


/* inserts the item. if dist_sq is >= 0, then do an ordered insert */
/* TODO make the ordering code use heapsort */
__device__ static int d_rlist_insert(struct res_node *list, struct kdnode *item, double dist_sq)
{
    printf("d_rlist_insert open\n");
	struct res_node *rnode;

	if(!(rnode = (struct res_node *)malloc(sizeof(struct res_node)))) {
		return -1;
	}
	rnode->item = item;
	rnode->dist_sq = dist_sq;

	if(dist_sq >= 0.0) {
		while(list->next && list->next->dist_sq < dist_sq) {
			list = list->next;
		}
	}
	rnode->next = list->next;
	list->next = rnode;
	return 0;
}

__device__ static void d_clear_results(struct kdres *rset)
{
	struct res_node *tmp, *node = rset->rlist->next;

	while(node) {
		tmp = node;
		node = node->next;
		free(tmp);
	}

	rset->rlist->next = 0;
}




//host
struct kdtree *h_kd_create(int k)
{
	struct kdtree *tree;

	if(!(tree = (struct kdtree *)malloc(sizeof *tree))) {
		return 0;
	}

	tree->dim = k;
	tree->root = 0;
	tree->destr = 0;
	tree->rect = 0;

	return tree;
}

void h_kd_free(struct kdtree *tree)
{
	if(tree) {
		h_kd_clear(tree);
		free(tree);
	}
}

static void h_clear_rec(struct kdnode *node, void (*destr)(void*))
{
	if(!node) return;

	h_clear_rec(node->left, destr);
	h_clear_rec(node->right, destr);
	
	if(destr) {
		destr(node->data);
	}
	free(node->pos);
	free(node);
}

void h_kd_clear(struct kdtree *tree)
{
	h_clear_rec(tree->root, tree->destr);
	tree->root = 0;

	if (tree->rect) {
		h_hyperrect_free(tree->rect);
		tree->rect = 0;
	}
}

void h_kd_data_destructor(struct kdtree *tree, void (*destr)(void*))
{
	tree->destr = destr;
}


static int h_insert_rec(struct kdnode **nptr, const double *pos, void *data, int dir, int dim)
{
	int new_dir;
	struct kdnode *node;

	if(!*nptr) {
		if(!(node = (struct kdnode *)malloc(sizeof *node))) {
			return -1;
		}
		if(!(node->pos = (double *)malloc(dim * sizeof *node->pos))) {
			free(node);
			return -1;
		}
		memcpy(node->pos, pos, dim * sizeof *node->pos);
		node->data = data;
		node->dir = dir;
		node->left = node->right = 0;
		*nptr = node;
		return 0;
	}

	node = *nptr;
	new_dir = (node->dir + 1) % dim;
	if(pos[node->dir] < node->pos[node->dir]) {
		return h_insert_rec(&(*nptr)->left, pos, data, new_dir, dim);
	}
	return h_insert_rec(&(*nptr)->right, pos, data, new_dir, dim);
}

int h_kd_insert(struct kdtree *tree, const double *pos, void *data)
{
	if (h_insert_rec(&tree->root, pos, data, 0, tree->dim)) {
		return -1;
	}

	if (tree->rect == 0) {
		tree->rect = h_hyperrect_create(tree->dim, pos, pos);
	} else {
		h_hyperrect_extend(tree->rect, pos);
	}

	return 0;
}

int h_kd_insertf(struct kdtree *tree, const float *pos, void *data)
{
	static double sbuf[16];
	double *bptr, *buf = 0;
	int res, dim = tree->dim;

	if(dim > 16) {
        if(!(bptr = buf = (double *)malloc(dim * sizeof *bptr))) {
            return -1;
        }
	} else {
		bptr = buf = sbuf;
	}

	while(dim-- > 0) {
		*bptr++ = *pos++;
	}

	res = h_kd_insert(tree, buf, data);
	if(tree->dim > 16) free(buf);
	return res;
}

int h_kd_insert3(struct kdtree *tree, double x, double y, double z, void *data)
{
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return h_kd_insert(tree, buf, data);
}

int h_kd_insert3f(struct kdtree *tree, float x, float y, float z, void *data)
{
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return h_kd_insert(tree, buf, data);
}

static int h_find_nearest(struct kdnode *node, const double *pos, double range, struct res_node *list, int ordered, int dim)
{
    printf("h_find_nearest open\n");
	double dist_sq, dx;
	int i, ret, added_res = 0;

	if(!node) return 0;
    printf("fn 1\n");
	dist_sq = 0;
	for(i=0; i<dim; i++) {
		dist_sq += SQ(node->pos[i] - pos[i]);
	}
    printf("fn 2\n");
	if(dist_sq <= SQ(range)) {
		if(h_rlist_insert(list, node, ordered ? dist_sq : -1.0) == -1) {
			return -1;
		}
		added_res = 1;
	}
    printf("fn 3\n");

	dx = pos[node->dir] - node->pos[node->dir];
    printf("fn 4\n");
	ret = h_find_nearest(dx <= 0.0 ? node->left : node->right, pos, range, list, ordered, dim);
    printf("fn 5\n");
	if(ret >= 0 && fabs(dx) < range) {
		added_res += ret;
		ret = h_find_nearest(dx <= 0.0 ? node->right : node->left, pos, range, list, ordered, dim);
	}
    printf("fn 6\n");
	if(ret == -1) {
		return -1;
	}
    printf("fn 7\n");
	added_res += ret;
    printf("fn 8\n");
	return added_res;
}

#if 0
static int h_find_nearest_n(struct kdnode *node, const double *pos, double range, int num, struct rheap *heap, int dim)
{
	double dist_sq, dx;
	int i, ret, added_res = 0;

	if(!node) return 0;
	
	/* if the photon is close enough, add it to the result heap */
	dist_sq = 0;
	for(i=0; i<dim; i++) {
		dist_sq += SQ(node->pos[i] - pos[i]);
	}
	if(dist_sq <= range_sq) {
		if(heap->size >= num) {
			/* get furthest element */
			struct res_node *maxelem = rheap_get_max(heap);

			/* and check if the new one is closer than that */
			if(maxelem->dist_sq > dist_sq) {
				rheap_remove_max(heap);

				if(rheap_insert(heap, node, dist_sq) == -1) {
					return -1;
				}
				added_res = 1;

				range_sq = dist_sq;
			}
		} else {
			if(rheap_insert(heap, node, dist_sq) == -1) {
				return =1;
			}
			added_res = 1;
		}
	}


	/* find signed distance from the splitting plane */
	dx = pos[node->dir] - node->pos[node->dir];

	ret = h_find_nearest_n(dx <= 0.0 ? node->left : node->right, pos, range, num, heap, dim);
	if(ret >= 0 && fabs(dx) < range) {
		added_res += ret;
		ret = h_find_nearest_n(dx <= 0.0 ? node->right : node->left, pos, range, num, heap, dim);
	}

}
#endif

static void h_kd_nearest_i(struct kdnode *node, const double *pos, struct kdnode **result, double *result_dist_sq, struct kdhyperrect* rect)
{
	int dir = node->dir;
	int i;
	double dummy, dist_sq;
	struct kdnode *nearer_subtree, *farther_subtree;
	double *nearer_hyperrect_coord, *farther_hyperrect_coord;

	/* Decide whether to go left or right in the tree */
	dummy = pos[dir] - node->pos[dir];
	if (dummy <= 0) {
		nearer_subtree = node->left;
		farther_subtree = node->right;
		nearer_hyperrect_coord = rect->max + dir;
		farther_hyperrect_coord = rect->min + dir;
	} else {
		nearer_subtree = node->right;
		farther_subtree = node->left;
		nearer_hyperrect_coord = rect->min + dir;
		farther_hyperrect_coord = rect->max + dir;
	}

	if (nearer_subtree) {
		/* Slice the hyperrect to get the hyperrect of the nearer subtree */
		dummy = *nearer_hyperrect_coord;
		*nearer_hyperrect_coord = node->pos[dir];
		/* Recurse down into nearer subtree */
		h_kd_nearest_i(nearer_subtree, pos, result, result_dist_sq, rect);
		/* Undo the slice */
		*nearer_hyperrect_coord = dummy;
	}

	/* Check the distance of the point at the current node, compare it
	 * with our best so far */
	dist_sq = 0;
	for(i=0; i < rect->dim; i++) {
		dist_sq += SQ(node->pos[i] - pos[i]);
	}
	if (dist_sq < *result_dist_sq) {
		*result = node;
		*result_dist_sq = dist_sq;
	}

	if (farther_subtree) {
		/* Get the hyperrect of the farther subtree */
		dummy = *farther_hyperrect_coord;
		*farther_hyperrect_coord = node->pos[dir];
		/* Check if we have to recurse down by calculating the closest
		 * point of the hyperrect and see if it's closer than our
		 * minimum distance in result_dist_sq. */
		if (h_hyperrect_dist_sq(rect, pos) < *result_dist_sq) {
			/* Recurse down into farther subtree */
			h_kd_nearest_i(farther_subtree, pos, result, result_dist_sq, rect);
		}
		/* Undo the slice on the hyperrect */
		*farther_hyperrect_coord = dummy;
	}
}

struct kdres *h_kd_nearest(struct kdtree *kd, const double *pos)
{
	struct kdhyperrect *rect;
	struct kdnode *result;
	struct kdres *rset;
	double dist_sq;
	int i;

	if (!kd) return 0;
	if (!kd->rect) return 0;

	/* Allocate result set */
	if(!(rset = (struct kdres *)malloc(sizeof *rset))) {
		return 0;
	}
	if(!(rset->rlist = (struct res_node *)malloc(sizeof(struct res_node)))) {
		free(rset);
		return 0;
	}
	rset->rlist->next = 0;
	rset->tree = kd;

	/* Duplicate the bounding hyperrectangle, we will work on the copy */
	if (!(rect = h_hyperrect_duplicate(kd->rect))) {
		h_kd_res_free(rset);
		return 0;
	}

	/* Our first guesstimate is the root node */
	result = kd->root;
	dist_sq = 0;
	for (i = 0; i < kd->dim; i++)
		dist_sq += SQ(result->pos[i] - pos[i]);

	/* Search for the nearest neighbour recursively */
	h_kd_nearest_i(kd->root, pos, &result, &dist_sq, rect);

	/* Free the copy of the hyperrect */
	h_hyperrect_free(rect);

	/* Store the result */
	if (result) {
		if (h_rlist_insert(rset->rlist, result, -1.0) == -1) {
			h_kd_res_free(rset);
			return 0;
		}
		rset->size = 1;
		h_kd_res_rewind(rset);
		return rset;
	} else {
		h_kd_res_free(rset);
		return 0;
	}
}

struct kdres *h_kd_nearestf(struct kdtree *tree, const float *pos)
{
	static double sbuf[16];
	double *bptr, *buf = 0;
	int dim = tree->dim;
	struct kdres *res;

	if(dim > 16) {
        if(!(bptr = buf = (double *)malloc(dim * sizeof *bptr))) {
            return 0;
        }
	} else {
		bptr = buf = sbuf;
	}

	while(dim-- > 0) {
		*bptr++ = *pos++;
	}

	res = h_kd_nearest(tree, buf);
	if(tree->dim > 16) free(buf);
	return res;
}

struct kdres *h_kd_nearest3(struct kdtree *tree, double x, double y, double z)
{
	double pos[3];
	pos[0] = x;
	pos[1] = y;
	pos[2] = z;
	return h_kd_nearest(tree, pos);
}

struct kdres *h_kd_nearest3f(struct kdtree *tree, float x, float y, float z)
{
	double pos[3];
	pos[0] = x;
	pos[1] = y;
	pos[2] = z;
	return h_kd_nearest(tree, pos);
}

struct kdres *h_kd_nearest_range(struct kdtree *kd, const double *pos, double range)
{
    printf("h_kd_nearest_range open\n");
	int ret;
	struct kdres *rset;

	if(!(rset = (struct kdres *)malloc(sizeof *rset))) {
		return 0;
	}
	if(!(rset->rlist = (struct res_node *)malloc(sizeof(struct res_node)))) {
		free(rset);
		return 0;
	}
	rset->rlist->next = 0;
	rset->tree = kd;

	if((ret = h_find_nearest(kd->root, pos, range, rset->rlist, 0, kd->dim)) == -1) {
		h_kd_res_free(rset);
		return 0;
	}
    printf("fn close\n");
	rset->size = ret;
	h_kd_res_rewind(rset);
	return rset;
}

struct kdres *h_kd_nearest_rangef(struct kdtree *kd, const float *pos, float range)
{
	static double sbuf[16];
	double *bptr, *buf = 0;
	int dim = kd->dim;
	struct kdres *res;

	if(dim > 16) {
        if(!(bptr = buf = (double *)malloc(dim * sizeof *bptr))) {
            return 0;
        }
	} else {
		bptr = buf = sbuf;
	}

	while(dim-- > 0) {
		*bptr++ = *pos++;
	}

	res = h_kd_nearest_range(kd, buf, range);
	if(kd->dim > 16) free(buf);
	return res;
}

struct kdres *h_kd_nearest_range3(struct kdtree *tree, double x, double y, double z, double range)
{
    printf("h_kd_nearest_range3 open\n");
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return h_kd_nearest_range(tree, buf, range);
}

struct kdres *h_kd_nearest_range3f(struct kdtree *tree, float x, float y, float z, float range)
{
	double buf[3];
	buf[0] = x;
	buf[1] = y;
	buf[2] = z;
	return h_kd_nearest_range(tree, buf, range);
}

void h_kd_res_free(struct kdres *rset)
{
    printf("h_kd_res_free open\n");
	h_clear_results(rset);
	free(rset->rlist);
	free(rset);
}

int h_kd_res_size(struct kdres *set)
{
	return (set->size);
}

void h_kd_res_rewind(struct kdres *rset)
{
    printf("h_kd_res_rewind open\n");
	rset->riter = rset->rlist->next;
}

int h_kd_res_end(struct kdres *rset)
{
	return rset->riter == 0;
}

int h_kd_res_next(struct kdres *rset)
{
	rset->riter = rset->riter->next;
	return rset->riter != 0;
}

void *h_kd_res_item(struct kdres *rset, double *pos)
{
	if(rset->riter) {
		if(pos) {
			memcpy(pos, rset->riter->item->pos, rset->tree->dim * sizeof *pos);
		}
		return rset->riter->item->data;
	}
	return 0;
}

void *h_kd_res_itemf(struct kdres *rset, float *pos)
{
	if(rset->riter) {
		if(pos) {
			int i;
			for(i=0; i<rset->tree->dim; i++) {
				pos[i] = rset->riter->item->pos[i];
			}
		}
		return rset->riter->item->data;
	}
	return 0;
}

void *h_kd_res_item3(struct kdres *rset, double *x, double *y, double *z)
{
	if(rset->riter) {
		if(x) *x = rset->riter->item->pos[0];
		if(y) *y = rset->riter->item->pos[1];
		if(z) *z = rset->riter->item->pos[2];
		return rset->riter->item->data;
	}
	return 0;
}

void *h_kd_res_item3f(struct kdres *rset, float *x, float *y, float *z)
{
	if(rset->riter) {
		if(x) *x = rset->riter->item->pos[0];
		if(y) *y = rset->riter->item->pos[1];
		if(z) *z = rset->riter->item->pos[2];
		return rset->riter->item->data;
	}
	return 0;
}

void *h_kd_res_item_data(struct kdres *set)
{
	return h_kd_res_item(set, 0);
}

/* ---- hyperrectangle helpers ---- */
static struct kdhyperrect* h_hyperrect_create(int dim, const double *min, const double *max)
{
	size_t size = dim * sizeof(double);
	struct kdhyperrect* rect = 0;

	if (!(rect = (struct kdhyperrect*)malloc(sizeof(struct kdhyperrect)))) {
		return 0;
	}

	rect->dim = dim;
	if (!(rect->min = (double *)malloc(size))) {
		free(rect);
		return 0;
	}
	if (!(rect->max = (double *)malloc(size))) {
		free(rect->min);
		free(rect);
		return 0;
	}
	memcpy(rect->min, min, size);
	memcpy(rect->max, max, size);

	return rect;
}

static void h_hyperrect_free(struct kdhyperrect *rect)
{
	free(rect->min);
	free(rect->max);
	free(rect);
}

static struct kdhyperrect* h_hyperrect_duplicate(const struct kdhyperrect *rect)
{
	return h_hyperrect_create(rect->dim, rect->min, rect->max);
}

static void h_hyperrect_extend(struct kdhyperrect *rect, const double *pos)
{
	int i;

	for (i=0; i < rect->dim; i++) {
		if (pos[i] < rect->min[i]) {
			rect->min[i] = pos[i];
		}
		if (pos[i] > rect->max[i]) {
			rect->max[i] = pos[i];
		}
	}
}

static double h_hyperrect_dist_sq(struct kdhyperrect *rect, const double *pos)
{
	int i;
	double result = 0;

	for (i=0; i < rect->dim; i++) {
		if (pos[i] < rect->min[i]) {
			result += SQ(rect->min[i] - pos[i]);
		} else if (pos[i] > rect->max[i]) {
			result += SQ(rect->max[i] - pos[i]);
		}
	}

	return result;
}


/* inserts the item. if dist_sq is >= 0, then do an ordered insert */
/* TODO make the ordering code use heapsort */
static int h_rlist_insert(struct res_node *list, struct kdnode *item, double dist_sq)
{
    printf("h_rlist_insert open\n");
	struct res_node *rnode;

	if(!(rnode = (struct res_node *)malloc(sizeof(struct res_node)))) {
		return -1;
	}
	rnode->item = item;
	rnode->dist_sq = dist_sq;

	if(dist_sq >= 0.0) {
		while(list->next && list->next->dist_sq < dist_sq) {
			list = list->next;
		}
	}
	rnode->next = list->next;
	list->next = rnode;
	return 0;
}

static void h_clear_results(struct kdres *rset)
{
	struct res_node *tmp, *node = rset->rlist->next;

	while(node) {
		tmp = node;
		node = node->next;
		free(tmp);
	}

	rset->rlist->next = 0;
}


__device__ unsigned int Rand(unsigned int randx)
{
    randx = randx*1103515245+12345;
    return randx&2147483647;
}

static double dist_sq( double *a1, double *a2, int dims ) {
  double dist_sq = 0, diff;
  while( --dims >= 0 ) {
    diff = (a1[dims] - a2[dims]);
    dist_sq += diff*diff;
  }
  return dist_sq;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ int eigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 

__global__ void normalsGPU(float* points,int point_size,int* neighbor_points_indices,int* neighbor_start_indices,int neighbor_points_count,float* normals,float* curvatures,long long int* covariance_time,long long int* eigen_time) {
    // printf("normalsGPU");
    //インデックス取得
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
    unsigned int output_id=50;
    // printf("idx = %d ,", idx);

    // if(idx==output_id) {
    //     int i, vcount = 50;
    //     void *kd, *set;
    //     printf("inserting %d random vectors... ", vcount);
    //     kd = d_kd_create(3);
    //     printf("kdcreate ok\n");
    //     for(i=0; i<vcount; i++) {
    //         float x, y, z;
    //         /*
    //         x = ((float)Rand(idx) / 4294967295.0f) * 10.0 - 5.0;
    //         y = ((float)Rand(idx) / 4294967295.0f) * 10.0 - 5.0;
    //         z = ((float)Rand(idx) / 4294967295.0f) * 10.0 - 5.0;
    //         */
    //         x = (float)i/(float)vcount *10.0 - 5.0;
    //         y = (float)i/(float)vcount *10.0 - 5.0;
    //         z = (float)i/(float)vcount *10.0 - 5.0;
    //         printf("rand ok\n");
    //         d_kd_insert3f((struct kdtree *)kd, x, y, z, 0);
    //         printf("insert ok\n");
    //     }
    //     printf("kdset ok\n");
    //     set = d_kd_nearest_range3f((struct kdtree *)kd, 0, 0, 0, 1);
    //     printf("range query returned %d items\n", d_kd_res_size((struct kdres *)set));
    //     d_kd_res_free((struct kdres *)set);
    //     d_kd_free((struct kdtree *)kd);
    // }

    if(idx<point_size-1){//対象スレッド内のみ計算
        //デバッグ用
        // if(idx==0||idx==10||idx==20) printf("points(%d) = %f,%f,%f\n",idx,points[idx*3+0],points[idx*3+1],points[idx*3+2]);
        
        // printf("idx<point_size");
        //近傍点終点インデックスの定義
        int end_indices;
        if(idx==(point_size-1)) end_indices = neighbor_points_count;
        else end_indices = neighbor_start_indices[idx+1]-1;

        int neighbor_size=(end_indices-neighbor_start_indices[idx]+1)/3;
        // if(idx==0||idx==10||idx==20) printf("neighbor(%d) start = %d, end = %d,size = %d\n",idx,neighbor_start_indices[idx],end_indices,neighbor_size);
        // printf("neighbor_size = %d\n", neighbor_size);
        if(neighbor_size>=3){//近傍点数3以上
            long long int covariance_start, covariance_stop;
            long long int eigen_start,eigen_stop;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_start));
            // printf("neighbor_size>=3");
            //平均計算
            float x_average=0,y_average=0,z_average=0;
            // if(idx==output_id) printf("neighbor_size = %d\n",neighbor_size);

            // if(idx==output_id) printf("neighbor_points(%d) = {\n",idx);
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                // デバッグ用
                // if((idx==0||idx==10||idx==20)&&(i==(neighbor_start_indices[idx]))) printf("neighbor_points_indices(%d) = %d\n",idx,neighbor_points_indices[i]);
                // if((idx==0||idx==10||idx==20)&&i==neighbor_start_indices[idx]) printf("neighbor_points(%d) = %f,%f,%f\n",idx,points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                // if(idx==output_id) printf("{%f, %f, %f},\n",points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                x_average+=points[neighbor_points_indices[i]*3+0];
                y_average+=points[neighbor_points_indices[i]*3+1];
                z_average+=points[neighbor_points_indices[i]*3+2];
            }
            // if(idx==output_id) printf("};\n");
            x_average/=neighbor_size;
            y_average/=neighbor_size;
            z_average/=neighbor_size;

            // //要素計算
            float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                sxx+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+0]-x_average);
                syy+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                szz+=(points[neighbor_points_indices[i]*3+2]-z_average)*(points[neighbor_points_indices[i]*3+2]-z_average);

                sxy+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                sxz+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
                syz+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
            }

            sxx/=neighbor_size;
            syy/=neighbor_size;
            szz/=neighbor_size;
            sxy/=neighbor_size;
            sxz/=neighbor_size;
            syz/=neighbor_size;

            //共分散行列
            float a[3*3]={
                sxx,sxy,sxz,
                sxy,syy,syz,
                sxz,syz,szz,
            };

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_stop));
            covariance_time[idx]=covariance_stop - covariance_start;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_start));
            
            // __syncthreads();
            // if(idx==output_id){
            //     printf("                          %f ,%f ,%f \ncovariance matrix(%d)=   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,idx,sxy,syy,syz,sxz,syz,szz);
            // }

            //固有値計算
            float eigen_vector[3 * 3];
            eigenJacobiMethod(a, eigen_vector, 3);

            // __syncthreads();
            // if(neighbor_size<3){
            //     printf("               %f ,%f ,%f \neigen_value=   %f ,%f ,%f \n               %f ,%f ,%f \n\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7],a[8]);
            // }

            float eigen_value[3];
            eigen_value[0]=a[0];
            eigen_value[1]=a[4];
            eigen_value[2]=a[8];

            int min_eigen_axis=0;
            float min_eigen_value=eigen_value[0];
            float eigen_sum=0;
            for(int i=1;i<3;i++){//x,y,z
                if(eigen_value[i]<min_eigen_value){
                    min_eigen_value=eigen_value[i];
                    min_eigen_axis=i;
                }
                //正規化用にnorm計算しておく
                eigen_sum += eigen_value[i];
            }

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_stop));
            eigen_time[idx]=eigen_stop - eigen_start;

            normals[idx*3+0]=eigen_vector[min_eigen_axis+0];
            normals[idx*3+1]=eigen_vector[min_eigen_axis+3];
            normals[idx*3+2]=eigen_vector[min_eigen_axis+6];

            curvatures[idx]=min_eigen_value/eigen_sum;

            // if(idx==output_id){
            //     printf("normals(%d) = %f, %f, %f\n\n\n\n",idx,normals[idx*3+0],normals[idx*3+1],normals[idx*3+2]);
            //     printf("curvature(%d) = %f\n",idx,curvatures[idx]);
            // }

            //デバッグ用
            // normals[idx*3+0]=idx*10+0;
            // normals[idx*3+1]=idx*10+1;
            // normals[idx*3+2]=idx*10+2;
            // printf("normal_x = %f ,normal_y = %f ,normal_z = %f \n", normals[idx*3+1],normals[idx*3+2],normals[idx*3+3]);
        }
        else{
            normals[idx*3+0]=0;
            normals[idx*3+1]=0;
            normals[idx*3+2]=0;
            curvatures[idx]=0;
            covariance_time[idx]=0;
            eigen_time[idx]=0;
        }
    }
    
}

extern void compute_normals(std::vector<std::vector<float>> points_array,std::vector<std::vector<int>> neighbor_points_indices,std::vector<int> neighbor_start_indices,int neighbor_points_count,std::vector<std::vector<float>>& normals_array,std::vector<float>& curvatures_array,std::vector<long long int>& covariance_compute_time,std::vector<long long int>& eigen_compute_time){
	int i, vcount = 50;
	void *kd, *set;
	printf("inserting %d random vectors... ", vcount);
	kd = h_kd_create(3);
	printf("kdcreate ok\n");
	for(i=0; i<vcount; i++) {
		float x, y, z;
		/*
		x = ((float)Rand(idx) / 4294967295.0f) * 10.0 - 5.0;
		y = ((float)Rand(idx) / 4294967295.0f) * 10.0 - 5.0;
		z = ((float)Rand(idx) / 4294967295.0f) * 10.0 - 5.0;
		*/
		x = (float)i/(float)vcount *10.0 - 5.0;
		y = (float)i/(float)vcount *10.0 - 5.0;
		z = (float)i/(float)vcount *10.0 - 5.0;
		printf("rand ok\n");
		h_kd_insert3f((struct kdtree *)kd, x, y, z, 0);
		printf("insert ok\n");
	}
	printf("kdset ok\n");
	set = h_kd_nearest_range3f((struct kdtree *)kd, 0, 0, 0, 1);//tree,pos,radius
	printf("range query returned %d items\n", h_kd_res_size((struct kdres *)set));
	
	char *pch;
	double pos[3], dist;
	double pt[3] = { 0, 0, 0 };//中心のポイント

	while( !h_kd_res_end( (struct kdres *)set ) ) {
		/* get the data and position of the current result item */
		pch = (char*)h_kd_res_item( (struct kdres *)set, pos);

		/* compute the distance of the current result from the pt */
		dist = sqrt( dist_sq( pt, pos, 3 ) );

		/* print out the retrieved data */
		printf( "node at (%.3f, %.3f, %.3f) is %.3f away\n", 
			pos[0], pos[1], pos[2], dist);
		if(pch!=0) printf( "has data=%c\n", *pch);
		// 	pos[0], pos[1], pos[2], dist, *pch );
		// printf( "node at (%.3f, %.3f, %.3f) is %.3f away and has data=%c\n", 
		// 	pos[0], pos[1], pos[2], dist, *pch );

		/* go to the next entry */
		h_kd_res_next((struct kdres *)set);
	}

	h_kd_res_free((struct kdres *)set);
	h_kd_free((struct kdtree *)kd);

    // std::cout<<"3.01"<<std::endl;
    //ホスト1次配列宣言
    std::vector<float> h_points(points_array.size() * 3);
    std::vector<int> h_neighbor_points_indices(neighbor_points_count);
    std::vector<float> h_normals(points_array.size() * 3);
    std::vector<float> h_curvatures(points_array.size());
    std::vector<long long int> h_covariance_compute_time(points_array.size());
    std::vector<long long int> h_eigen_compute_time(points_array.size());
    // std::cout<<"3.02"<<std::endl;
    //デバイス1次配列宣言
    float *d_points,*d_normals,*d_curvatures;
    int *d_neighbor_points_indices,*d_neighbor_start_indices;
    long long int *d_covariance_compute_time,*d_eigen_compute_time;
    // std::cout<<"3.03"<<std::endl;
    //メモリ確保
    hipMalloc((void **)&d_points, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_neighbor_points_indices, neighbor_points_count * sizeof(int));
    hipMalloc((void **)&d_neighbor_start_indices, points_array.size() * sizeof(int));
    hipMalloc((void **)&d_normals, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_curvatures, points_array.size() * sizeof(float));
    hipMalloc((void **)&d_covariance_compute_time, points_array.size() * sizeof(long long int));
    hipMalloc((void **)&d_eigen_compute_time, points_array.size() * sizeof(long long int));
    // std::cout<<"3.04"<<std::endl;
    //1次配列化
    int k=0,l=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            h_points[k]=points_array[i][j];
            k++;
        }
        for(int j=0;j<neighbor_points_indices[i].size();j++){//近傍
            h_neighbor_points_indices[l]=neighbor_points_indices[i][j];
            l++;
        }
    }
    // std::cout<<"3.05"<<std::endl;
    //コピー
    hipMemcpy(d_points, &h_points[0], points_array.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_points_indices, &h_neighbor_points_indices[0], neighbor_points_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_start_indices, &neighbor_start_indices[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
    // std::cout<<"3.06"<<std::endl;
    //並列スレッド設定
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((points_array.size() + block.x - 1) / block.x, 1);
    // std::cout<<"3.07"<<std::endl;
    // std::cout<<"normalsGPUstart"<<std::endl;
    hipDeviceSetLimit(hipLimitStackSize, 1024*8);
    //実行
    normalsGPU<<<grid,block>>>(d_points,points_array.size(),d_neighbor_points_indices,d_neighbor_start_indices,neighbor_points_count,d_normals,d_curvatures,d_covariance_compute_time,d_eigen_compute_time);
    // std::cout<<"normalsGPUend"<<std::endl;
    // std::cout<<"3.08"<<std::endl;
    //コピー
    hipMemcpy(&h_normals[0], d_normals, points_array.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_curvatures[0], d_curvatures, points_array.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_covariance_compute_time[0], d_covariance_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_compute_time[0], d_eigen_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    // std::cout<<"3.09"<<std::endl;
    //2次配列化
    k=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            normals_array[i][j]=h_normals[k];
            k++;
        }
        curvatures_array[i]=h_curvatures[i];
        covariance_compute_time[i]=h_covariance_compute_time[i];
        eigen_compute_time[i]=h_eigen_compute_time[i];
    }
    // std::cout<<"cu_normals : "<<normals_array[0][0]<<","<<normals_array[0][1]<<","<<normals_array[0][2]<<std::endl;
    // std::cout<<"3.10"<<std::endl;
    //メモリ解放
    hipFree(d_points);
    hipFree(d_neighbor_points_indices);
    hipFree(d_neighbor_start_indices);
    hipFree(d_normals);
    hipFree(d_curvatures);
    hipFree(d_covariance_compute_time);
    hipFree(d_eigen_compute_time);
}