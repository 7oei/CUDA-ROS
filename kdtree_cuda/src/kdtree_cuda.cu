#include "hip/hip_runtime.h"


#include <string.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kdtree_cuda/kdtree_cuda.hpp"
#include <vector>
#include <iterator>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <fstream>

int sort_axis=0;
int frames=0;
typedef struct
{
	int	id;
	float pos[3];
} point_with_id;

typedef struct
{
	int	parent_id;
	int left_id;
	int right_id;
	int axis;
} node;

typedef struct
{
	int i;
	bool ready;
} int_with_ready;

typedef struct
{
	bool ready;
	bool node_is_right;
	int parent_id;
	int left_id;
	int right_id;
	int depth;
	int axis;
	int middle;
	int group_size;
	int *x_sort_ids;
	int *y_sort_ids;
	int *z_sort_ids;
} detailed_node;

bool first=true;

//	年齢(昇順)
__host__ int AxisSort(const void * n1, const void * n2)
{
	if (((point_with_id *)n1)->pos[sort_axis] > ((point_with_id *)n2)->pos[sort_axis])
	{
		return 1;
	}
	else if (((point_with_id *)n1)->pos[sort_axis] < ((point_with_id *)n2)->pos[sort_axis])
	{
		return -1;
	}
	else
	{
		return 0;
	}
}

__host__ int CreateTree(int* root_id,std::vector <node>& nodes, std::vector<std::vector<float>> points,std::vector<int> group_indices,int parent_id,bool node_is_right)
{
	//入力データ初期化
	int group_size = group_indices.size();
	// std::cout<<"group_size"<<group_size<<std::endl;
	point_with_id point_with_ids[group_size];
	std::vector<std::vector<int>> axis_sort_ids(3, std::vector<int>(group_size));
	// std::cout<<"oppai 1 "<<std::endl;
	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////3*points
		point_with_ids[i].id=group_indices[i];
		point_with_ids[i].pos[0]=points[group_indices[i]][0];
		point_with_ids[i].pos[1]=points[group_indices[i]][1];
		point_with_ids[i].pos[2]=points[group_indices[i]][2];
	}
	// std::cout<<"oppai 2 "<<std::endl;

	//ソート
	float max[3],min[3],median[3],length[3];
	int axis_median_id[3];
	int median_id;
	for(sort_axis=0; sort_axis<3; sort_axis++){//x,y,zそれぞれにソート
		// std::cout<<"sort_axis = "<<sort_axis<<std::endl;
		qsort(point_with_ids, group_size, sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < group_size ; i++){///////////////////////////////////////////////////////////////////////////////////////////3*points
			axis_sort_ids[sort_axis][i]=point_with_ids[i].id;
			// printf("%d, %f, %f, %f \n", point_with_ids[i].id, point_with_ids[i].pos[0], point_with_ids[i].pos[1], point_with_ids[i].pos[2]);
		}
		// std::cout<<std::endl;
		//max,min,median,axis_median_id取得
		max[sort_axis]=point_with_ids[group_size-1].pos[sort_axis];//minとmaxいらんかも
		min[sort_axis]=point_with_ids[0].pos[sort_axis];
		length[sort_axis]=max[sort_axis]-min[sort_axis];
		median[sort_axis]=point_with_ids[(group_size-1)/2].pos[sort_axis];//偶数なら小さい方(-1)消せば大きい方
		axis_median_id[sort_axis]=point_with_ids[(group_size-1)/2].id;
	}
	// std::cout<<"oppai 3 "<<std::endl;
	// std::cout<<"x_length = "<< length[0] <<", x_median["<<axis_median_id[0]<<"] = "<<median[0]<<std::endl;
	// std::cout<<"y_length = "<< length[1] <<", y_median["<<axis_median_id[1]<<"] = "<<median[1]<<std::endl;
	// std::cout<<"z_length = "<< length[2] <<", z_median["<<axis_median_id[2]<<"] = "<<median[2]<<std::endl;
	// std::cout<<std::endl;
	// for(int i=0;i<group_size;i++){
	// 	std::cout<<"point_id["<<i<<"] = "<<point_with_ids[i].id<<std::endl;
	// }
	

	//中央値id設定、長軸設定
	if(length[0]>=length[1]&&length[0]>=length[2]){
		median_id=axis_median_id[0];
		nodes[median_id].axis=0;
	}
	if(length[1]>=length[0]&&length[1]>=length[2]){
		median_id=axis_median_id[1];
		nodes[median_id].axis=1;
	}
	if(length[2]>=length[0]&&length[2]>=length[1]){
		median_id=axis_median_id[2];
		nodes[median_id].axis=2;
	}
	// std::cout<<"oppai 4 "<<std::endl;

	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////points
		group_indices[i]=axis_sort_ids[nodes[median_id].axis][i];
	}
	// std::cout<<"oppai 5 "<<std::endl;
	// memcpy(&group_indices[0], axis_sort_ids[nodes[median_id].axis], group_size*sizeof(int));
	// std::vector<int> group_indices2;
	// group_indices.resize(0);
	// copy(axis_sort_ids[nodes[median_id].axis].begin(), axis_sort_ids[nodes[median_id].axis].end(), back_inserter(group_indices) );



	//node初期化
	nodes[median_id].left_id=-1;
	nodes[median_id].right_id=-1;

	//親設定、親の左右設定
	nodes[median_id].parent_id=parent_id;
	if(parent_id>=0){//親あり
		if(!node_is_right) nodes[parent_id].left_id=median_id;
		if(node_is_right) nodes[parent_id].right_id=median_id;
	}
	else{//親なし
		*root_id=median_id;
	}
	// std::cout<<"oppai 6 "<<std::endl;

	// std::vector<int> right_group(group_size);
	// std::vector<int> left_group(group_size);
	// int right_count=0;
	// int left_count=0;

	// for(int i=0;i<=((group_size-1)/2)-1;i++){////////////////////////////////////////////////////////////////////////////////////points
	// 	left_group[left_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	left_count++;
	// }
	// left_group.resize(left_count);///////////////////////////////////////////////////////////////////////////////////////////////points
	// for(int i=((group_size-1)/2)+1;i<group_size;i++){
	// 	right_group[right_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	right_count++;
	// }
	// right_group.resize(right_count);

	// std::vector<int> v0{1,2,3,4};
	// std::vector<int> v9(v0.begin(),v0.end());  // [1,2,3,4]
	// std::vector<int> v10(v0.begin(),v0.end());  

	// group_indices.resize(8);
	// for(int i=0;i<8;i++){
	// 	group_indices[i]=i;
	// }
	// size_t middle = ((8-1)/2);

	size_t middle = ((group_size-1)/2);
	std::vector<int>::iterator middleIter(group_indices.begin());
	std::advance(middleIter, middle);
	// std::cout<<"advance end"<<std::endl;

	std::vector<int> left_group(group_indices.begin(), middleIter);
	++middleIter;
	std::vector<int> right_group(middleIter, group_indices.end());
	// std::cout<<"oppai 7 "<<std::endl;
	// std::cout<<"group end"<<std::endl;
	// std::cout<<"left group is [";
	// for(int i=0;i<left_group.size();i++){
	// 	if(i<10) std::cout<<left_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	// std::cout<<"right group is [";
	// for(int i=0;i<right_group.size();i++){
	// 	if(i<10) std::cout<<right_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	//


	// std::cout<<"median_id"<<median_id<<std::endl;
	// std::cout<<"parent_id"<<parent_id<<std::endl;
	// std::cout<<"left_id"<<nodes[median_id].left_id<<std::endl;
	// std::cout<<"right_id"<<nodes[median_id].right_id<<std::endl;
	// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;

	//right,left group表示
	// std::cout<<"left_group is (";
	// for(int i=0;i<left_group.size();i++){
	// 	std::cout<<left_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<"right_group is (";
	// for(int i=0;i<right_group.size();i++){
	// 	std::cout<<right_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<"--------------------------------------------------------------------------------"<<std::endl;
	bool left=false;
	bool right=false;
	if(group_size>1){//子がいる
		if(left_group.size()>0){//左に子がいる
			left= CreateTree(root_id,nodes,points,left_group,median_id,false);
		}
		else left=true;
		if(right_group.size()>0){//右に子がいる
			right= CreateTree(root_id,nodes,points,right_group,median_id,true);
		}
		else right=true;
		// std::cout<<"oppai 8 "<<std::endl;
		if(right&&left) return 1;
	}
	else return 1;//子がいない
}

__host__ int CreateNode(int* root_id,int point_size,std::vector <node>& nodes, std::vector<std::vector<int>> axis_sort_ids,int depth,int parent_id,bool node_is_right)
{

	// std::cout << "oppai 0" << std::endl;
	// std::cout << std::endl;
	int group_size = axis_sort_ids[0].size();
	int axis = depth % 3;
	size_t middle = ((group_size-1)/2);
	int median_id = axis_sort_ids[axis][middle];
	nodes[median_id].axis = axis;
	nodes[median_id].parent_id = parent_id;
	nodes[median_id].left_id = -1;
	nodes[median_id].right_id = -1;
	if(parent_id >= 0){ // 親あり
		if(!node_is_right) nodes[parent_id].left_id = median_id;
		if(node_is_right) nodes[parent_id].right_id = median_id;
	}
	else{ // 親なし
		*root_id = median_id;
	}
	// std::cout << "oppai 4" << std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout << "axis_sort_ids ="<<std::endl;
	// for(int j = 0; j < 3; j++){
	// 	if(j==0) std::cout << "x =";
	// 	if(j==1) std::cout << "y =";
	// 	if(j==2) std::cout << "z =";
	// 	for(int i = 0; i < group_size; i++){
	// 		std::cout << axis_sort_ids[j][i] << ",";
	// 	}
	// 	std::cout<<std::endl;
	// }

	if(group_size > 1){ // 子あり
		std::vector<int>::iterator middle_iter(axis_sort_ids[axis].begin());
		std::advance(middle_iter,middle);
		std::vector<int> left_group(axis_sort_ids[axis].begin(),middle_iter);
		++middle_iter;
		std::vector<int> right_group(middle_iter,axis_sort_ids[axis].end());

		// std::cout<<"median_id"<<median_id<<std::endl;
		// std::cout<<"middle"<<middle<<std::endl;
		// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;
		// std::cout<<"group is (";
		// for(int i=0;i<group_size;i++){
		// 	std::cout<<axis_sort_ids[axis][i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"left_group is (";
		// for(int i=0;i<left_group.size();i++){
		// 	std::cout<<left_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"right_group is (";
		// for(int i=0;i<right_group.size();i++){
		// 	std::cout<<right_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;

		// std::cout << "oppai 1" << std::endl;

		std::vector<std::vector<int>> left_axis_sort_ids(3,std::vector<int>(left_group.size()));
		std::vector<std::vector<int>> right_axis_sort_ids(3,std::vector<int>(right_group.size()));

		std::vector<int> next_group(point_size,0);/////////////これどうにかしたい
		std::vector<int> left_axis_count(3,0);
		std::vector<int> right_axis_count(3,0);
		// std::cout << "oppai 1.5" << std::endl;
		// std::cout << "next_group.size()" << next_group.size() <<std::endl;
		// std::cout << "left_group.size()" << left_group.size() <<std::endl;
		// std::cout << "right_group.size()" << right_group.size() <<std::endl;
		for(int i = 0; i < left_group.size(); i++){
			// std::cout << "oppai 1.51" << std::endl;
			left_axis_sort_ids[axis][i] = left_group[i];
			// std::cout << "oppai 1.52" << std::endl;
			// std::cout << "left_group[i]" << left_group[i] <<std::endl;
			next_group[left_group[i]] = -1;//これで死んでそう//left_group[i]がnext_groupのレンジを超えている//この式の参照indexおかしい//1段目では正しく作用
		}
		// std::cout << "oppai 1.6" << std::endl;
		for(int i = 0; i < right_group.size(); i++){
			right_axis_sort_ids[axis][i] = right_group[i];
			// std::cout << "right_group[i]" << right_group[i] <<std::endl;
			next_group[right_group[i]] = 1;
		}
		// std::cout << "oppai 2" << std::endl;
		for(int i = 0; i < group_size; i++){
			for(int j = 0; j < 3; j++){
				if(j==axis) continue;
				if(next_group[axis_sort_ids[j][i]] == -1){
					left_axis_sort_ids[j][left_axis_count[j]] = axis_sort_ids[j][i];
					left_axis_count[j]++;
					// std::cout << "left_axis_count["<<j<<"] = "<<left_axis_count[j]<<std::endl;
				}
				else if(next_group[axis_sort_ids[j][i]] == 1){
					right_axis_sort_ids[j][right_axis_count[j]] = axis_sort_ids[j][i];
					right_axis_count[j]++;
					// std::cout << "right_axis_count["<<j<<"] = "<<right_axis_count[j]<<std::endl;
				}
			}
		}

		bool left = false;
		bool right = false;
		if(left_group.size() > 0) left = CreateNode(root_id,point_size,nodes,left_axis_sort_ids,depth+1,median_id,false);
		else left = true;

		if(right_group.size() > 0) right = CreateNode(root_id,point_size,nodes,right_axis_sort_ids,depth+1,median_id,true);
		else right = true;

		if(right&&left) return 1;
	}
	else return 1;
}

__host__ int CreateNode2(int* root_id,int point_size,std::vector <node>& nodes, std::vector<std::vector<int>> axis_sort_ids,int depth,int parent_id,bool node_is_right)
{

	// std::cout << "oppai 0" << std::endl;
	// std::cout << std::endl;
	int group_size = axis_sort_ids[0].size();
	int axis = depth % 3;
	size_t middle = ((group_size-1)/2);
	int median_id = axis_sort_ids[axis][middle];
	// std::cout<<"layer["<<depth<<"] median_id is"<<median_id<<std::endl;
	// std::ofstream ofs1("/home/adachi/cpu_tree.csv",std::ios::app);
    // ofs1 << depth << "," << median_id << ","<<std::endl;
	//depth,median_id
	nodes[median_id].axis = axis;
	nodes[median_id].parent_id = parent_id;
	nodes[median_id].left_id = -1;
	nodes[median_id].right_id = -1;
	if(parent_id >= 0){ // 親あり
		if(!node_is_right) nodes[parent_id].left_id = median_id;
		if(node_is_right) nodes[parent_id].right_id = median_id;
	}
	else{ // 親なし
		*root_id = median_id;
	}
	// std::cout << "oppai 4" << std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout << "axis_sort_ids ="<<std::endl;
	// for(int j = 0; j < 3; j++){
	// 	if(j==0) std::cout << "x =";
	// 	if(j==1) std::cout << "y =";
	// 	if(j==2) std::cout << "z =";
	// 	for(int i = 0; i < group_size; i++){
	// 		std::cout << axis_sort_ids[j][i] << ",";
	// 	}
	// 	std::cout<<std::endl;
	// }

	if(group_size > 1){ // 子あり
		std::vector<int>::iterator middle_iter(axis_sort_ids[axis].begin());
		std::advance(middle_iter,middle);
		std::vector<int> left_group(axis_sort_ids[axis].begin(),middle_iter);
		++middle_iter;
		std::vector<int> right_group(middle_iter,axis_sort_ids[axis].end());

		// std::cout<<"median_id"<<median_id<<std::endl;
		// std::cout<<"middle"<<middle<<std::endl;
		// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;
		// std::cout<<"group is (";
		// for(int i=0;i<group_size;i++){
		// 	std::cout<<axis_sort_ids[axis][i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"left_group is (";
		// for(int i=0;i<left_group.size();i++){
		// 	std::cout<<left_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;
		// std::cout<<"right_group is (";
		// for(int i=0;i<right_group.size();i++){
		// 	std::cout<<right_group[i]<<",";
		// }
		// std::cout<<")"<<std::endl;

		// std::cout << "oppai 1" << std::endl;

		std::vector<std::vector<int>> left_axis_sort_ids(3,std::vector<int>(left_group.size()));
		std::vector<std::vector<int>> right_axis_sort_ids(3,std::vector<int>(right_group.size()));

		std::vector<int> next_group(point_size,0);/////////////これどうにかしたい
		std::vector<int> left_axis_count(3,0);
		std::vector<int> right_axis_count(3,0);
		// std::cout << "oppai 1.5" << std::endl;
		// std::cout << "next_group.size()" << next_group.size() <<std::endl;
		// std::cout << "left_group.size()" << left_group.size() <<std::endl;
		// std::cout << "right_group.size()" << right_group.size() <<std::endl;
		for(int i = 0; i < left_group.size(); i++){
			// std::cout << "oppai 1.51" << std::endl;
			left_axis_sort_ids[axis][i] = left_group[i];
			// std::cout << "oppai 1.52" << std::endl;
			// std::cout << "left_group[i]" << left_group[i] <<std::endl;
			next_group[left_group[i]] = -1;//これで死んでそう//left_group[i]がnext_groupのレンジを超えている//この式の参照indexおかしい//1段目では正しく作用
		}
		// std::cout << "oppai 1.6" << std::endl;
		for(int i = 0; i < right_group.size(); i++){
			right_axis_sort_ids[axis][i] = right_group[i];
			// std::cout << "right_group[i]" << right_group[i] <<std::endl;
			next_group[right_group[i]] = 1;
		}
		// std::cout << "oppai 2" << std::endl;
		for(int i = 0; i < group_size; i++){
			for(int j = 0; j < 3; j++){
				if(j==axis) continue;
				if(next_group[axis_sort_ids[j][i]] == -1){
					left_axis_sort_ids[j][left_axis_count[j]] = axis_sort_ids[j][i];
					left_axis_count[j]++;
					// std::cout << "left_axis_count["<<j<<"] = "<<left_axis_count[j]<<std::endl;
				}
				else if(next_group[axis_sort_ids[j][i]] == 1){
					right_axis_sort_ids[j][right_axis_count[j]] = axis_sort_ids[j][i];
					right_axis_count[j]++;
					// std::cout << "right_axis_count["<<j<<"] = "<<right_axis_count[j]<<std::endl;
				}
			}
		}

		bool left = false;
		bool right = false;
		if(left_group.size() > 0) left = CreateNode2(root_id,point_size,nodes,left_axis_sort_ids,depth+1,median_id,false);
		else left = true;

		if(right_group.size() > 0) right = CreateNode2(root_id,point_size,nodes,right_axis_sort_ids,depth+1,median_id,true);
		else right = true;

		if(right&&left) return 1;
	}
	else return 1;
}

__host__ void TreeOutCsv(std::vector<detailed_node> nodes,int depth,int median_id)
{
	std::ofstream ofs1("/home/adachi/gpu_large_heep_tree.csv",std::ios::app);
    ofs1 << depth << "," << median_id << ","<<std::endl;
	if(nodes[median_id].left_id>=0) TreeOutCsv(nodes,depth+1,nodes[median_id].left_id);
	if(nodes[median_id].right_id>=0) TreeOutCsv(nodes,depth+1,nodes[median_id].right_id);
}

__global__ void d_CreateNode(int point_size,int group_size,int depth,int parent_id,bool node_is_right,int *x_sort_ids,int *y_sort_ids,int *z_sort_ids,int *root_id,node* nodes)
{

	// printf("create node open\n");
	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
	// printf("idx = %d, ",idx);
	// printf("group size = %d\n",group_size);
	int axis = depth % 3;
	size_t middle = ((group_size-1)/2);
	int median_id;
	if(axis==0) median_id = x_sort_ids[middle];
	if(axis==1) median_id = y_sort_ids[middle];
	if(axis==2) median_id = z_sort_ids[middle];

	// if(median_id==1||median_id==13||median_id==14||median_id==15||median_id==19||median_id==36||median_id==45){//ここでは正しい
	// 	printf("1 x_sort_ids[] = ");
	// 	for(int i=0;i<group_size;i++){
	// 		printf("%d,",x_sort_ids[i]);
	// 	}
	// 	printf("\n");
	// }

	int *copy_x_sort_ids,*copy_y_sort_ids,*copy_z_sort_ids;
	copy_x_sort_ids = (int *)malloc(group_size * sizeof(int));
	copy_y_sort_ids = (int *)malloc(group_size * sizeof(int));
	copy_z_sort_ids = (int *)malloc(group_size * sizeof(int));
	memcpy(copy_x_sort_ids, x_sort_ids, group_size * sizeof(int));
	memcpy(copy_y_sort_ids, y_sort_ids, group_size * sizeof(int));
	memcpy(copy_z_sort_ids, z_sort_ids, group_size * sizeof(int));

	nodes[median_id].axis = axis;
	nodes[median_id].parent_id = parent_id;
	nodes[median_id].left_id = -1;
	nodes[median_id].right_id = -1;
	// if(median_id==1||median_id==13||median_id==14||median_id==15||median_id==19||median_id==36||median_id==45){//ここでは正しい
	// 	printf("2 copy_x_sort_ids[] = ");
	// 	for(int i=0;i<group_size;i++){
	// 		printf("%d,",copy_x_sort_ids[i]);
	// 	}
	// 	printf("\n");
	// }

	// printf("1");
	if(parent_id >= 0){ // 親あり
		free(x_sort_ids);
		free(y_sort_ids);
		free(z_sort_ids);
		if(!node_is_right) nodes[parent_id].left_id = median_id;
		if(node_is_right) nodes[parent_id].right_id = median_id;
	}
	else{ // 親なし
		printf("root update\n");
		*root_id = median_id;
	}
	// printf("2");
	if(group_size > 1){ // 子あり
		int left_group_size = 0;
		int right_group_size = 0;
		int *left_x_sort_ids,*left_y_sort_ids,*left_z_sort_ids;
		int *right_x_sort_ids,*right_y_sort_ids,*right_z_sort_ids;
		int *next_group;
		next_group = (int *)malloc(point_size * sizeof(int));
		left_x_sort_ids = (int *)malloc(middle * sizeof(int));
		left_y_sort_ids = (int *)malloc(middle * sizeof(int));
		left_z_sort_ids = (int *)malloc(middle * sizeof(int));
		right_x_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		right_y_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		right_z_sort_ids = (int *)malloc((group_size - (middle + 1)) * sizeof(int));
		int left_axis_count[3]={0,0,0};
		int right_axis_count[3]={0,0,0};
		// printf("\n\n\n");
		// printf("median_id = %d\n",median_id);
		// if(!node_is_right) printf("node is left\n");
		// else printf("node is right\n");
		// printf("parent_id = %d\n",parent_id);
		// printf("middle = %d\n",middle);
		// printf("axis = %d\n",nodes[median_id].axis);

		// printf("3");
		// printf("axis = %d",axis);
		// if(median_id==1||median_id==14||median_id==19||median_id==36||median_id==45){//ここでは正しい
		// 	printf("3 copy_x_sort_ids[] = ");
		// 	for(int i=0;i<group_size;i++){
		// 		printf("%d,",copy_x_sort_ids[i]);
		// 	}
		// 	printf("\n");
		// }


		if(axis==0){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_x_sort_ids[left_axis_count[0]] = copy_x_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[0]++;
					// printf("3.02 ");
					// printf("parent_id = %d",parent_id);
					// printf("copy_x_sort_ids[%d] = %d ",i,copy_x_sort_ids[i]);
					next_group[copy_x_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_x_sort_ids[i],next_group[copy_x_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_x_sort_ids[right_axis_count[0]] = copy_x_sort_ids[i];
					right_axis_count[0]++;
					next_group[copy_x_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_x_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}
		}
		else if(axis==1){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_y_sort_ids[left_axis_count[1]] = copy_y_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[1]++;
					// printf("3.02 ");
					// printf("copy_y_sort_ids[%d] = %d ",i,copy_y_sort_ids[i]);
					next_group[copy_y_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_y_sort_ids[i],next_group[copy_y_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_y_sort_ids[right_axis_count[1]] = copy_y_sort_ids[i];
					right_axis_count[1]++;
					next_group[copy_y_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_y_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}

		}
		else if(axis==2){
			// printf("for start");
			for(int i = 0; i < group_size; i++){
				if(point_size<copy_x_sort_ids[i]) printf("out of range copy_x_sort_ids[%d] = %d \n",i,copy_x_sort_ids[i]);
				if(point_size<copy_y_sort_ids[i]) printf("out of range copy_y_sort_ids[%d] = %d \n",i,copy_y_sort_ids[i]);
				if(point_size<copy_z_sort_ids[i]) printf("out of range copy_z_sort_ids[%d] = %d \n",i,copy_z_sort_ids[i]);
				// printf("i = %d ",i);
				// printf("middle = %d ",middle);
				if(i<middle){
					left_z_sort_ids[left_axis_count[2]] = copy_z_sort_ids[i];
					// printf("3.01 ");
					left_axis_count[2]++;
					// printf("3.02 ");
					// printf("copy_z_sort_ids[%d] = %d ",i,copy_z_sort_ids[i]);
					next_group[copy_z_sort_ids[i]] = -1;
					// printf("next_group[%d] = %d ",copy_z_sort_ids[i],next_group[copy_z_sort_ids[i]]);
					// printf("3.1 ");
				}
				else if(i>middle){
					right_z_sort_ids[right_axis_count[2]] = copy_z_sort_ids[i];
					right_axis_count[2]++;
					next_group[copy_z_sort_ids[i]] = 1;
					// printf("3.3");
				}
				else{
					next_group[copy_z_sort_ids[i]] = 0;
					// printf("3.2");
				}
			}
		}
		// printf("\n");
		// printf("4");
		left_group_size = left_axis_count[axis];
		right_group_size = right_axis_count[axis];

		// printf("group is (");
		// for(int i=0;i<group_size;i++){
		// 	if(axis==0) printf("%d,",copy_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",copy_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",copy_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("left_group is (");
		// for(int i=0;i<left_group_size;i++){
		// 	if(axis==0) printf("%d,",left_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",left_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",left_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("right_group is (");
		// for(int i=0;i<right_group_size;i++){
		// 	if(axis==0) printf("%d,",right_x_sort_ids[i]);
		// 	if(axis==1) printf("%d,",right_y_sort_ids[i]);
		// 	if(axis==2) printf("%d,",right_z_sort_ids[i]);
		// }
		// printf(")");
		// printf("\n\n\n");

		for(int i = 0; i < group_size; i++){
			for(int j = 0; j < 3; j++){
				if(j==axis) continue;
				if(j==0){//x実装
					if(next_group[copy_x_sort_ids[i]] == -1){
						left_x_sort_ids[left_axis_count[j]] = copy_x_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_x_sort_ids[i]] == 1){
						right_x_sort_ids[right_axis_count[j]] = copy_x_sort_ids[i];
						right_axis_count[j]++;
					}
				}
				if(j==1){//y実装
					if(next_group[copy_y_sort_ids[i]] == -1){
						left_y_sort_ids[left_axis_count[j]] = copy_y_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_y_sort_ids[i]] == 1){
						right_y_sort_ids[right_axis_count[j]] = copy_y_sort_ids[i];
						right_axis_count[j]++;
					}
				}
				if(j==2){//z実装
					if(next_group[copy_z_sort_ids[i]] == -1){
						left_z_sort_ids[left_axis_count[j]] = copy_z_sort_ids[i];
						left_axis_count[j]++;
					}
					else if(next_group[copy_z_sort_ids[i]] == 1){
						right_z_sort_ids[right_axis_count[j]] = copy_z_sort_ids[i];
						right_axis_count[j]++;
					}
				}
			}
		}
		free(copy_x_sort_ids);
		free(copy_y_sort_ids);
		free(copy_z_sort_ids);
		free(next_group);
		// printf("5");
		// if(median_id==18) printf("Hit!!!!!!!!!!!!!!! left_x_sort_ids[16] = %d\n\n\n",left_x_sort_ids[16]);
		// if(median_id==33) printf("Hit!!!!!!!!!!!!!!! right_x_sort_ids[16] = %d\n\n\n",right_x_sort_ids[16]);
		// if(median_id==3145) printf("Hit!!!!!!!!!!!!!!! left_x_sort_ids[156] = %d\n\n\n",left_x_sort_ids[156]);
		// if(median_id==3877) printf("Hit!!!!!!!!!!!!!!! left_y_sort_ids[1] = %d\n\n\n",left_x_sort_ids[1]);
		// if(median_id==3888) printf("Hit!!!!!!!!!!!!!!! left_y_sort_ids[1] = %d\n\n\n",left_y_sort_ids[1]);
		// hipDeviceSynchronize();
		if(left_group_size > 0) d_CreateNode<<<1, 1>>>(point_size,left_group_size,depth+1,median_id,false,left_x_sort_ids,left_y_sort_ids,left_z_sort_ids,root_id,nodes);
		else {
			free(left_x_sort_ids);
			free(left_y_sort_ids);
			free(left_z_sort_ids);
		}
		if(right_group_size > 0) d_CreateNode<<<1, 1>>>(point_size,right_group_size,depth+1,median_id,true,right_x_sort_ids,right_y_sort_ids,right_z_sort_ids,root_id,nodes);
		else {
			free(right_x_sort_ids);
			free(right_y_sort_ids);
			free(right_z_sort_ids);
		}
		// hipDeviceSynchronize();
		// free(left_x_sort_ids);
		// free(left_y_sort_ids);
		// free(left_z_sort_ids);
		// free(right_x_sort_ids);
		// free(right_y_sort_ids);
		// free(right_z_sort_ids);
		
	}
}

__global__ void d_DepthCreateNode(int point_size,detailed_node* nodes,int* end_list)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
	// printf("idx = %d\n",idx);
	if(idx < point_size){//計算領域
		// printf("idx = %d\n",idx);
		// bool limit = true;
		// if(nodes[idx].depth == 9 && !nodes[idx].node_is_right) limit = false;//left or right limit
		// int id_th = 1895;//2464o,1000x,1732x,2098o,1915o,1823x,1869x,1892x,1903o,1897o,1894x,1895x,1896o
		// if(nodes[idx].depth == 9 && (idx<id_th)) limit = false;//left or right limit
		// if(nodes[idx].depth == 9 && limit) printf("layer9 limit in\n");
		// if(nodes[idx].depth == 9 && limit && (nodes[idx].ready && (0 > end_list[idx]))) printf("layer9 new in\n");
		// if(nodes[idx].depth == 9 && limit && nodes[idx].ready) printf("layer9 limit in and ready\n");
		// if(nodes[idx].depth == 9 && limit && (0 > end_list[idx])) printf("layer9 limit in and not end\n");
		// if(nodes[idx].depth == 9 && nodes[idx].ready) printf("layer9 ready\n");
		if(/*limit && */(nodes[idx].ready && (0 > end_list[idx]))){//該当ノード
			// if(nodes[idx].depth == 9) printf("in\n");
			printf("median_id = %d \n",idx);
			// printf("\n\n\n");
			// printf("\nmedian_id = %d\n",idx);特異
			// if(!nodes[idx].node_is_right) printf("node is left\n");
			// else printf("node is right\n");
			// printf("parent_id = %d\n",nodes[idx].parent_id);
			// printf("middle = %d\n",nodes[idx].middle);
			// printf("axis = %d\n",nodes[idx].axis);
			// if(nodes[idx].node_is_right) printf("device depth = %d\n",nodes[idx].depth);//間引きのため右のみ
			// if(nodes[idx].depth==9) printf("start idx = %d\n",idx);
			// if(idx==3040) printf("start idx = %d\n",idx);
			// printf("0");
			nodes[idx].left_id = -1;
			nodes[idx].right_id = -1;
			if(nodes[idx].group_size>1){//子あり
				// printf("\nmedian_id = %d\n",idx);
				int left_group_size = 0;
				int right_group_size = 0;
				int *left_x_sort_ids,*left_y_sort_ids,*left_z_sort_ids;
				int *right_x_sort_ids,*right_y_sort_ids,*right_z_sort_ids;
				int *next_group;
				next_group = (int *)malloc(point_size * sizeof(int));
				left_x_sort_ids = (int *)malloc(nodes[idx].middle * sizeof(int));
				left_y_sort_ids = (int *)malloc(nodes[idx].middle * sizeof(int));
				left_z_sort_ids = (int *)malloc(nodes[idx].middle * sizeof(int));
				right_x_sort_ids = (int *)malloc((nodes[idx].group_size - (nodes[idx].middle + 1)) * sizeof(int));
				right_y_sort_ids = (int *)malloc((nodes[idx].group_size - (nodes[idx].middle + 1)) * sizeof(int));
				right_z_sort_ids = (int *)malloc((nodes[idx].group_size - (nodes[idx].middle + 1)) * sizeof(int));
				int left_axis_count[3]={0,0,0};
				int right_axis_count[3]={0,0,0};

				if(nodes[idx].axis==0){
					for(int i = 0; i < nodes[idx].group_size; i++){
						if(point_size<nodes[idx].x_sort_ids[i]) printf("out of range nodes[idx].x_sort_ids[%d] = %d \n",i,nodes[idx].x_sort_ids[i]);
						if(point_size<nodes[idx].y_sort_ids[i]) printf("out of range nodes[idx].y_sort_ids[%d] = %d \n",i,nodes[idx].y_sort_ids[i]);
						if(point_size<nodes[idx].z_sort_ids[i]) printf("out of range nodes[idx].z_sort_ids[%d] = %d \n",i,nodes[idx].z_sort_ids[i]);
						if(i<nodes[idx].middle){
							left_x_sort_ids[left_axis_count[0]] = nodes[idx].x_sort_ids[i];
							left_axis_count[0]++;
							next_group[nodes[idx].x_sort_ids[i]] = -1;
						}
						else if(i>nodes[idx].middle){
							right_x_sort_ids[right_axis_count[0]] = nodes[idx].x_sort_ids[i];
							right_axis_count[0]++;
							next_group[nodes[idx].x_sort_ids[i]] = 1;
						}
						else{
							next_group[nodes[idx].x_sort_ids[i]] = 0;
						}
					}
				}
				else if(nodes[idx].axis==1){
					for(int i = 0; i < nodes[idx].group_size; i++){
						if(point_size<nodes[idx].x_sort_ids[i]) printf("out of range nodes[idx].x_sort_ids[%d] = %d \n",i,nodes[idx].x_sort_ids[i]);
						if(point_size<nodes[idx].y_sort_ids[i]) printf("out of range nodes[idx].y_sort_ids[%d] = %d \n",i,nodes[idx].y_sort_ids[i]);
						if(point_size<nodes[idx].z_sort_ids[i]) printf("out of range nodes[idx].z_sort_ids[%d] = %d \n",i,nodes[idx].z_sort_ids[i]);
						if(i<nodes[idx].middle){
							left_y_sort_ids[left_axis_count[1]] = nodes[idx].y_sort_ids[i];
							left_axis_count[1]++;
							next_group[nodes[idx].y_sort_ids[i]] = -1;
						}
						else if(i>nodes[idx].middle){
							right_y_sort_ids[right_axis_count[1]] = nodes[idx].y_sort_ids[i];
							right_axis_count[1]++;
							next_group[nodes[idx].y_sort_ids[i]] = 1;
						}
						else{
							next_group[nodes[idx].y_sort_ids[i]] = 0;
						}
					}

				}
				else if(nodes[idx].axis==2){
					for(int i = 0; i < nodes[idx].group_size; i++){
						if(point_size<nodes[idx].x_sort_ids[i]) printf("out of range nodes[idx].x_sort_ids[%d] = %d \n",i,nodes[idx].x_sort_ids[i]);
						if(point_size<nodes[idx].y_sort_ids[i]) printf("out of range nodes[idx].y_sort_ids[%d] = %d \n",i,nodes[idx].y_sort_ids[i]);
						if(point_size<nodes[idx].z_sort_ids[i]) printf("out of range nodes[idx].z_sort_ids[%d] = %d \n",i,nodes[idx].z_sort_ids[i]);
						if(i<nodes[idx].middle){
							left_z_sort_ids[left_axis_count[2]] = nodes[idx].z_sort_ids[i];
							left_axis_count[2]++;
							next_group[nodes[idx].z_sort_ids[i]] = -1;
						}
						else if(i>nodes[idx].middle){
							right_z_sort_ids[right_axis_count[2]] = nodes[idx].z_sort_ids[i];
							right_axis_count[2]++;
							next_group[nodes[idx].z_sort_ids[i]] = 1;
						}
						else{
							next_group[nodes[idx].z_sort_ids[i]] = 0;
						}
					}
				}
				hipDeviceSynchronize();
				// printf("depth = %d\n",nodes[idx].depth);
				// printf("median_id = %d \n",idx);
				hipDeviceSynchronize();
				// if(nodes[idx].depth==9) printf("section1 idx = %d\n",idx);
				// if(idx==3040) printf("section1 idx = %d\n",idx);
				// printf("1 ");
				left_group_size = left_axis_count[nodes[idx].axis];
				right_group_size = right_axis_count[nodes[idx].axis];

				for(int i = 0; i < nodes[idx].group_size; i++){
					for(int j = 0; j < 3; j++){
						if(j==nodes[idx].axis) continue;
						if(j==0){//x実装
							if(next_group[nodes[idx].x_sort_ids[i]] == -1){
								left_x_sort_ids[left_axis_count[j]] = nodes[idx].x_sort_ids[i];
								left_axis_count[j]++;
							}
							else if(next_group[nodes[idx].x_sort_ids[i]] == 1){
								right_x_sort_ids[right_axis_count[j]] = nodes[idx].x_sort_ids[i];
								right_axis_count[j]++;
							}
						}
						if(j==1){//y実装
							if(next_group[nodes[idx].y_sort_ids[i]] == -1){
								left_y_sort_ids[left_axis_count[j]] = nodes[idx].y_sort_ids[i];
								left_axis_count[j]++;
							}
							else if(next_group[nodes[idx].y_sort_ids[i]] == 1){
								right_y_sort_ids[right_axis_count[j]] = nodes[idx].y_sort_ids[i];
								right_axis_count[j]++;
							}
						}
						if(j==2){//z実装
							if(next_group[nodes[idx].z_sort_ids[i]] == -1){
								left_z_sort_ids[left_axis_count[j]] = nodes[idx].z_sort_ids[i];
								left_axis_count[j]++;
							}
							else if(next_group[nodes[idx].z_sort_ids[i]] == 1){
								right_z_sort_ids[right_axis_count[j]] = nodes[idx].z_sort_ids[i];
								right_axis_count[j]++;
							}
						}
					}
				}
				hipDeviceSynchronize();
				// if(nodes[idx].depth==9) printf("section2 idx = %d\n",idx);
				// if(idx==3040) printf("section2 idx = %d\n",idx);
				// printf("2 ");
				free(next_group);
				int next_axis = (nodes[idx].depth + 1) % 3;
				if(left_group_size > 0){
					size_t left_middle = ((left_group_size - 1) / 2);
					int left_median_id;
					if(next_axis == 0) left_median_id = left_x_sort_ids[left_middle];
					if(next_axis == 1) left_median_id = left_y_sort_ids[left_middle];
					if(next_axis == 2) left_median_id = left_z_sort_ids[left_middle];

					nodes[idx].left_id = left_median_id;

					nodes[left_median_id].ready = true;
					nodes[left_median_id].node_is_right = false;
					nodes[left_median_id].parent_id = idx;
					nodes[left_median_id].depth = nodes[idx].depth + 1;
					nodes[left_median_id].axis = next_axis;
					nodes[left_median_id].middle = left_middle;
					nodes[left_median_id].group_size = left_group_size;
					nodes[left_median_id].x_sort_ids = left_x_sort_ids;
					nodes[left_median_id].y_sort_ids = left_y_sort_ids;
					nodes[left_median_id].z_sort_ids = left_z_sort_ids;
				}
				if(right_group_size > 0){
					size_t right_middle = ((right_group_size - 1) / 2);
					int right_median_id;
					if(next_axis == 0) right_median_id = right_x_sort_ids[right_middle];
					if(next_axis == 1) right_median_id = right_y_sort_ids[right_middle];
					if(next_axis == 2) right_median_id = right_z_sort_ids[right_middle];

					nodes[idx].right_id = right_median_id;

					nodes[right_median_id].ready = true;
					nodes[right_median_id].node_is_right = true;
					nodes[right_median_id].parent_id = idx;
					nodes[right_median_id].depth = nodes[idx].depth + 1;
					nodes[right_median_id].axis = next_axis;
					nodes[right_median_id].middle = right_middle;
					nodes[right_median_id].group_size = right_group_size;
					nodes[right_median_id].x_sort_ids = right_x_sort_ids;
					nodes[right_median_id].y_sort_ids = right_y_sort_ids;
					nodes[right_median_id].z_sort_ids = right_z_sort_ids;
				}
				hipDeviceSynchronize();
				// if(nodes[idx].depth==9) printf("section3 idx = %d\n",idx);
				// if(idx==3040) printf("section3 idx = %d\n",idx);
				// printf("3 ");
			}
			if(nodes[idx].parent_id >= 0){//親あり
				free(nodes[idx].x_sort_ids);
				free(nodes[idx].y_sort_ids);
				free(nodes[idx].z_sort_ids);
			}
			hipDeviceSynchronize();
			// if(nodes[idx].depth==9) printf("section4 idx = %d\n",idx);
			// if(idx==3040) printf("section4 idx = %d\n",idx);
			// printf("4 ");
			end_list[idx] = 1;
			hipDeviceSynchronize();
			// printf("5 ");
			// if(idx==3040) printf("section5 idx = %d\n",idx);
			// if(nodes[idx].depth==9) printf("end idx = %d\n",idx);
		}
	}
}

__device__ void d_PointRangeCheckAndAdd(int *range_indices_size,int *range_indices,int head_id,float* points,float* search_point,float range_sq)
{
	float dist_sq = powf(points[head_id*3+0]-search_point[0],2)+powf(points[head_id*3+1]-search_point[1],2)+powf(points[head_id*3+2]-search_point[2],2);
	if(dist_sq<range_sq){
		// printf("device inside");
		range_indices[*range_indices_size] = head_id;
		*range_indices_size+=1;
	} 
}

__device__ int d_SearchSubTree(int *range_indices_size,int *range_indices,int root_id,node* nodes,float* points,float* search_point,float range_sq)
{
	int head_id = root_id;
	bool cross,next_is_right;
	//潜り
	while(1){
		if(search_point[nodes[head_id].axis]>points[head_id*3+nodes[head_id].axis]) next_is_right = true;
		else next_is_right = false;

		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
			if(next_is_right){//right
				head_id = nodes[head_id].right_id;
			}
			else{//left
				head_id = nodes[head_id].left_id;
			}
		}
		else break;
	}
	//rootが底
	if(head_id==root_id) {
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		return 1;
	}
	//昇り
	int last_id;
	while(1){
		cross = false;
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		//昇る
		last_id = head_id;
		head_id = nodes[head_id].parent_id;
		//交差判定　1軸のみ低効率
		float axis_diff_sq = powf(points[head_id*3+nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
		if(axis_diff_sq < range_sq) cross = true;
		int sub_tree=0;
		if(cross){
			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
		}
		if(head_id==root_id){
			d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
			break;
		}
	}

	return 1;
}

__device__ int d_SearchSubTree2(int *range_indices_size,int *range_indices,int root_id,detailed_node* nodes,float* points,float* search_point,float range_sq)
{
	int head_id = root_id;
	bool cross,next_is_right;
	//潜り
	while(1){
		if(search_point[nodes[head_id].axis]>points[head_id*3+nodes[head_id].axis]) next_is_right = true;
		else next_is_right = false;

		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
			if(next_is_right){//right
				head_id = nodes[head_id].right_id;
			}
			else{//left
				head_id = nodes[head_id].left_id;
			}
		}
		else break;
	}
	//rootが底
	if(head_id==root_id) {
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		return 1;
	}
	//昇り
	int last_id;
	while(1){
		cross = false;
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		//昇る
		last_id = head_id;
		head_id = nodes[head_id].parent_id;
		//交差判定　1軸のみ低効率
		float axis_diff_sq = powf(points[head_id*3+nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
		if(axis_diff_sq < range_sq) cross = true;
		int sub_tree=0;
		if(cross){
			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = d_SearchSubTree2(range_indices_size,range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = d_SearchSubTree2(range_indices_size,range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
		}
		if(head_id==root_id){
			d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
			break;
		}
	}

	return 1;
}

__device__ int EigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 

__global__ void NormalsGPU(/*detailed_node* detailed_nodes,*/long long int* neighbor_time,int *point_neighbor_size,int* point_neighbor,int* d_parent_ids,int* d_left_ids,int* d_right_ids,int* d_axes,int root_id,float* points,int point_size,int* neighbor_points_indices,int* neighbor_start_indices,int neighbor_points_count,float* normals,float* curvatures,long long int* covariance_time,long long int* eigen_time)
{
    // printf("normalsGPU");
    //インデックス取得
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
    unsigned int output_id=50;
    // printf("idx = %d ,", idx);
	if(idx==output_id){
		long long int neighbor_start, neighbor_stop;
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_start));
		////////////////////////////////////////////////////////////////
		node *nodes = (node*)malloc(sizeof(node) * point_size);

		for(int i=0;i<point_size;i++){
			nodes[i].parent_id=d_parent_ids[i];
			nodes[i].left_id=d_left_ids[i];
			nodes[i].right_id=d_right_ids[i];
			nodes[i].axis=d_axes[i];
		}
		float search_point[3];
		search_point[0]=points[idx*3+0];
		search_point[1]=points[idx*3+1];
		search_point[2]=points[idx*3+2];
		
		int *range_indices = (int*)malloc(sizeof(int) * point_size);
		int range_indices_size = 0;

		float range_sq = 0.15*0.15;

		//探索関数の実行
		int range_search = d_SearchSubTree(&range_indices_size,range_indices,root_id,nodes,points,search_point,range_sq);
		// int range_search = d_SearchSubTree2(&range_indices_size,range_indices,root_id,detailed_nodes,points,search_point,range_sq);
		// std::cout<<"range_indices.size()"<<range_indices.size()<<std::endl;
		if(range_search==1) {
			// printf("device range_indices size is =%d",range_indices_size);
			// printf("device range_indices is [");
			for(int i=0;i<range_indices_size;i++){
				// printf("%d,",range_indices[i]);
				point_neighbor[i]=range_indices[i];
			}
			// printf("]\n");
			point_neighbor_size[0]=range_indices_size;
			// printf("device size %d\n",range_indices_size);
		}
		
		free (nodes);
		free (range_indices);
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_stop));
		neighbor_time[idx]=neighbor_stop - neighbor_start;
		////////////////////////////////////////////////////////////////////
	}


    if(idx<point_size){//対象スレッド内のみ計算
        //デバッグ用
        // if(idx==0||idx==10||idx==20) printf("points(%d) = %f,%f,%f\n",idx,points[idx*3+0],points[idx*3+1],points[idx*3+2]);
        
        // printf("idx<point_size");
        //近傍点終点インデックスの定義
        int end_indices;
        if(idx==(point_size-1)) end_indices = neighbor_points_count;
        else end_indices = neighbor_start_indices[idx+1]-1;

        int neighbor_size=(end_indices-neighbor_start_indices[idx]+1)/3;
        // if(idx==0||idx==10||idx==20) printf("neighbor(%d) start = %d, end = %d,size = %d\n",idx,neighbor_start_indices[idx],end_indices,neighbor_size);
        // printf("neighbor_size = %d\n", neighbor_size);
        if(neighbor_size>=3){//近傍点数3以上
            long long int covariance_start, covariance_stop;
            long long int eigen_start,eigen_stop;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_start));
            // printf("neighbor_size>=3");
            //平均計算
            float x_average=0,y_average=0,z_average=0;
            // if(idx==output_id) printf("neighbor_size = %d\n",neighbor_size);

            // if(idx==output_id) printf("neighbor_points(%d) = {\n",idx);
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                // デバッグ用
                // if((idx==0||idx==10||idx==20)&&(i==(neighbor_start_indices[idx]))) printf("neighbor_points_indices(%d) = %d\n",idx,neighbor_points_indices[i]);
                // if((idx==0||idx==10||idx==20)&&i==neighbor_start_indices[idx]) printf("neighbor_points(%d) = %f,%f,%f\n",idx,points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                // if(idx==output_id) printf("{%f, %f, %f},\n",points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                x_average+=points[neighbor_points_indices[i]*3+0];
                y_average+=points[neighbor_points_indices[i]*3+1];
                z_average+=points[neighbor_points_indices[i]*3+2];
            }
            // if(idx==output_id) printf("};\n");
            x_average/=neighbor_size;
            y_average/=neighbor_size;
            z_average/=neighbor_size;

            // //要素計算
            float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                sxx+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+0]-x_average);
                syy+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                szz+=(points[neighbor_points_indices[i]*3+2]-z_average)*(points[neighbor_points_indices[i]*3+2]-z_average);

                sxy+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                sxz+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
                syz+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
            }

            sxx/=neighbor_size;
            syy/=neighbor_size;
            szz/=neighbor_size;
            sxy/=neighbor_size;
            sxz/=neighbor_size;
            syz/=neighbor_size;

            //共分散行列
            float a[3*3]={
                sxx,sxy,sxz,
                sxy,syy,syz,
                sxz,syz,szz,
            };

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_stop));
            covariance_time[idx]=covariance_stop - covariance_start;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_start));
            
            // __syncthreads();
            // if(idx==output_id){
            //     printf("                          %f ,%f ,%f \ncovariance matrix(%d)=   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,idx,sxy,syy,syz,sxz,syz,szz);
            // }

            //固有値計算
            float eigen_vector[3 * 3];
            EigenJacobiMethod(a, eigen_vector, 3);

            // __syncthreads();
            // if(neighbor_size<3){
            //     printf("               %f ,%f ,%f \neigen_value=   %f ,%f ,%f \n               %f ,%f ,%f \n\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7],a[8]);
            // }

            float eigen_value[3];
            eigen_value[0]=a[0];
            eigen_value[1]=a[4];
            eigen_value[2]=a[8];

            int min_eigen_axis=0;
            float min_eigen_value=eigen_value[0];
            float eigen_sum=0;
            for(int i=1;i<3;i++){//x,y,z
                if(eigen_value[i]<min_eigen_value){
                    min_eigen_value=eigen_value[i];
                    min_eigen_axis=i;
                }
                //正規化用にnorm計算しておく
                eigen_sum += eigen_value[i];
            }

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_stop));
            eigen_time[idx]=eigen_stop - eigen_start;

            normals[idx*3+0]=eigen_vector[min_eigen_axis+0];
            normals[idx*3+1]=eigen_vector[min_eigen_axis+3];
            normals[idx*3+2]=eigen_vector[min_eigen_axis+6];

            curvatures[idx]=min_eigen_value/eigen_sum;

            // if(idx==output_id){
            //     printf("normals(%d) = %f, %f, %f\n\n\n\n",idx,normals[idx*3+0],normals[idx*3+1],normals[idx*3+2]);
            //     printf("curvature(%d) = %f\n",idx,curvatures[idx]);
            // }

            //デバッグ用
            // normals[idx*3+0]=idx*10+0;
            // normals[idx*3+1]=idx*10+1;
            // normals[idx*3+2]=idx*10+2;
            // printf("normal_x = %f ,normal_y = %f ,normal_z = %f \n", normals[idx*3+1],normals[idx*3+2],normals[idx*3+3]);
        }
        else{
            normals[idx*3+0]=0;
            normals[idx*3+1]=0;
            normals[idx*3+2]=0;
            curvatures[idx]=0;
            covariance_time[idx]=0;
            eigen_time[idx]=0;
        }
    }
    
}

// __global__ void ChildKernel(void* data)
// {

// 	printf("child : %d, %d\n", blockIdx.x, threadIdx.x);

// }

// __global__ void ParentKernel(void* data)
// {

// 	printf("parent: %d, %d\n", blockIdx.x, threadIdx.x);

// 	ChildKernel<<<1, 2>>>(data);
// 	hipDeviceSynchronize();

// }

// __global__ void KernelFunctionArgumentTypeCheck(node* nodes)
// {
// 	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
//     unsigned int idx = ix;
// 	printf("KernelFunctionArgumentTypeCheck: %d, %d\n", blockIdx.x, threadIdx.x);
// 	for(int i=0;i<3;i++){
// 		nodes[i].parent_id = i; nodes[i].left_id = i; nodes[i].right_id = i; nodes[i].axis = i;
// 	}
// }

// __global__ void MyKernel(float* devPtr, size_t pitch, int width, int height)
// {
//   for (int r = 0; r < height; ++r) {
//     float* row = (float*)((char*)devPtr + r * pitch);
//     for (int c = 0; c < width; ++c) {
//       float element = row[c];
//     }
//   }
// }

// __global__ void d_ParallelRecursionTest(int data_size,int_with_ready* data)
// {
// 	unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
//     unsigned int idx = ix;
// 	if(idx<data_size){
// 		if(data[idx].ready){
// 			if(idx!=0) data[idx].i = data[idx-1].i + 1;
// 			data[idx+1].ready = true;
// 		}
// 	}
// }

extern void ComputeNormals(std::vector<long long int>& neighbor_time,std::vector<int>& point_neighbor,std::vector<std::vector<float>> points_array,std::vector<std::vector<int>> neighbor_points_indices,std::vector<int> neighbor_start_indices,int neighbor_points_count,std::vector<std::vector<float>>& normals_array,std::vector<float>& curvatures_array,std::vector<long long int>& covariance_compute_time,std::vector<long long int>& eigen_compute_time)
{


	// points_array.clear();
	// points_array.resize(8);
	// points_array = {{6, 0, 0}, 
	// 				{5, 3, 0},
	// 				{3, 4, 0},
	// 				{4, 6, 0},
	// 				{2, 5, 0},
	// 				{1, 2, 0},
	// 				{0, 1, 0},
	// 				{-3.21161e+38,4.57384e-41,-3.21161e+38}};
	// // points_array.resize(7);

	// int test_size = 684;
	// if(points_array.size()>test_size) points_array.resize(test_size);

	clock_t build_start,build_end;
	// build_start = clock();
	// if(frames==43) std::cout<<"dead point is ("<<points_array[77][0]<<","<<points_array[77][0]<<","<<points_array[77][0]<<")"<<std::endl;

	
	/////////////////////////////////////////////////////////////////////////////////////////
	int root_id=-1;
	std::vector <node> nodes;
	nodes.resize(points_array.size());
	std::vector<std::vector<int>> axis_sort_ids(3,std::vector<int>(points_array.size()));
	point_with_id point_with_ids[points_array.size()];
	for(int i=0;i<points_array.size();i++){
		point_with_ids[i].id = i;
		point_with_ids[i].pos[0] = points_array[i][0];
		point_with_ids[i].pos[1] = points_array[i][1];
		point_with_ids[i].pos[2] = points_array[i][2];
	}
	for(sort_axis=0; sort_axis<3; sort_axis++){
		qsort(point_with_ids, points_array.size(), sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < points_array.size() ; i++){
			axis_sort_ids[sort_axis][i]=point_with_ids[i].id;
		}
	}
	int create_end = CreateNode(&root_id,points_array.size(),nodes,axis_sort_ids,0,-1,false);
	/////////////////////////////////////////////////////////////////////////////////////////


	
	/////////////////////////////////////////////////////////////////////////////////////////施工

	// if(frames==0){
	// 	std::cout<<"size = "<<points_array.size()<<std::endl;
	// 	std::cout<<"points_array = {";
	// 	for(int i=0;i<points_array.size();i++){
	// 		std::cout<<"{";
	// 		for(int j=0;j<3;j++){
	// 			std::cout<<points_array[i][j];
	// 			if(j!=(3-1)) std::cout<<",";
	// 		}
	// 		std::cout<<"}";
	// 		if(i!=(points_array.size()-1)) std::cout<<",";
	// 		std::cout<<std::endl;
	// 	}
	// 	std::cout<<"};"<<std::endl;
	// }


	if(first){
		// int test_size = 4928;
		// std::vector<std::vector<float>> test_points(test_size);
		
		std::vector<std::vector<float>> test_points;
		std::copy(points_array.begin(), points_array.end(), std::back_inserter(test_points));
		//CPU
		// int cpu_root_id=-1;
		// std::vector <node> cpu_nodes;
		// cpu_nodes.resize(test_points.size());
		// std::vector<std::vector<int>> cpu_axis_sort_ids(3,std::vector<int>(test_points.size()));
		// point_with_id cpu_point_with_ids[test_points.size()];
		// for(int i=0;i<test_points.size();i++){
		// 	cpu_point_with_ids[i].id = i;
		// 	cpu_point_with_ids[i].pos[0] = test_points[i][0];
		// 	cpu_point_with_ids[i].pos[1] = test_points[i][1];
		// 	cpu_point_with_ids[i].pos[2] = test_points[i][2];
		// }
		// for(sort_axis=0; sort_axis<3; sort_axis++){
		// 	qsort(cpu_point_with_ids, test_points.size(), sizeof(point_with_id), AxisSort);
		// 	for (int i=0 ; i < test_points.size() ; i++){
		// 		cpu_axis_sort_ids[sort_axis][i]=cpu_point_with_ids[i].id;
		// 	}
		// }
		// int cpu_create_end = CreateNode2(&cpu_root_id,test_points.size(),cpu_nodes,cpu_axis_sort_ids,0,-1,false);

		//GPU
		std::vector <detailed_node> detailed_nodes;
		detailed_nodes.resize(test_points.size());
		std::vector<int> x_sort_ids(test_points.size());
		std::vector<int> y_sort_ids(test_points.size());
		std::vector<int> z_sort_ids(test_points.size());
		std::vector<int> end_list(test_points.size());

		// point_with_id point_with_ids[test_points.size()];
		for(int i=0;i<test_points.size();i++){
			point_with_ids[i].id = i;
			point_with_ids[i].pos[0] = test_points[i][0];
			point_with_ids[i].pos[1] = test_points[i][1];
			point_with_ids[i].pos[2] = test_points[i][2];
			detailed_nodes[i].ready = false;
			end_list[i] = -1;
		}
		for(sort_axis=0; sort_axis<3; sort_axis++){
			qsort(point_with_ids, test_points.size(), sizeof(point_with_id), AxisSort);
			for (int i=0 ; i < test_points.size(); i++){
				if(sort_axis==0){
					x_sort_ids[i]=point_with_ids[i].id;
				}
				if(sort_axis==1){
					y_sort_ids[i]=point_with_ids[i].id;
				}
				if(sort_axis==2){
					z_sort_ids[i]=point_with_ids[i].id;
				}
			}
		}
		//最初のmedian特定
		size_t root_middle = ((test_points.size() - 1) / 2);
		int root_median_id = x_sort_ids[root_middle];//最初はx

		detailed_nodes[root_median_id].ready = true;
		detailed_nodes[root_median_id].node_is_right = false;
		detailed_nodes[root_median_id].parent_id = -1;
		detailed_nodes[root_median_id].depth = 0;
		detailed_nodes[root_median_id].axis = 0;
		detailed_nodes[root_median_id].middle = root_middle;
		detailed_nodes[root_median_id].group_size = test_points.size();

		int *d_end_list;
		detailed_node *d_detailed_nodes;
		hipMalloc((void **)&detailed_nodes[root_median_id].x_sort_ids, test_points.size() * sizeof(int));
		hipMalloc((void **)&detailed_nodes[root_median_id].y_sort_ids, test_points.size() * sizeof(int));
		hipMalloc((void **)&detailed_nodes[root_median_id].z_sort_ids, test_points.size() * sizeof(int));
		hipMalloc((void **)&d_end_list, test_points.size() * sizeof(int));
		hipMalloc((void **)&d_detailed_nodes, test_points.size() * sizeof(detailed_node));

		hipMemcpy(detailed_nodes[root_median_id].x_sort_ids, &x_sort_ids[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(detailed_nodes[root_median_id].y_sort_ids, &y_sort_ids[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(detailed_nodes[root_median_id].z_sort_ids, &z_sort_ids[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_end_list, &end_list[0], test_points.size() * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_detailed_nodes, &detailed_nodes[0], test_points.size() * sizeof(detailed_node), hipMemcpyHostToDevice);
		hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*64);//5000pointで16で死ぬので64が最適
		// hipDeviceSetLimit(hipLimitStackSize, 1024);//並列ならいらん
		int dimx_create_node = 32;//32
		dim3 block_create_node(dimx_create_node, 1);
		dim3 grid_create_node((test_points.size() + block_create_node.x - 1) / block_create_node.x, 1);
		float estimate_depth = log2((test_points.size()+1.0f)/2.0f);
		int depth_count = 0;

		// std::cout << "frames" << frames <<"------------------------------------------------------------------------------------------------------------"<< std::endl;
		// if(frames==114) std::cout<<"dead point is ("<<test_points[684][0]<<","<<test_points[684][1]<<","<<test_points[684][2]<<")"<<std::endl;
		// if(frames==114) std::cout<<"around point is ("<<test_points[683][0]<<","<<test_points[683][1]<<","<<test_points[683][2]<<")"<<std::endl;
		// if(frames==114) std::cout<<"around point is ("<<test_points[682][0]<<","<<test_points[682][1]<<","<<test_points[682][2]<<")"<<std::endl;

		// if(frames==114){
		// 	std::cout<<"test_points = {";
		// 	for(int i=0;i<test_points.size();i++){
		// 		std::cout<<"{";
		// 		for(int j=0;j<3;j++){
		// 			std::cout<<test_points[i][j];
		// 			if(j!=(3-1)) std::cout<<",";
		// 		}
		// 		std::cout<<"}";
		// 		if(i!=(test_points.size()-1)) std::cout<<",";
		// 		std::cout<<std::endl;
		// 	}
		// 	std::cout<<"};"<<std::endl;
		// }
		// std::cout<<"estimate_depth = "<<estimate_depth<<std::endl;
		
		while(1){
			// if(depth_count==test_points.size()) break;
			// std::cout<<"call depth = "<< depth_count <<std::endl;
			// std::cout<<"create kernel start"<<std::endl;
			d_DepthCreateNode<<<grid_create_node,block_create_node>>>(test_points.size(),d_detailed_nodes,d_end_list);
			// std::cout<<"create kernel end"<<std::endl;
			if(depth_count >= (estimate_depth*2)){//*2で基本的に終わる(カバー領域は指数関数的に増えるため安心してok)
				// std::cout<<"limit termination"<<std::endl;
				break;
			} 

			depth_count++;
			
			// if(all_end) break;
		}
		
		
		//これ以降に遅くしてるやつがいる
		// 犯人はmemcpy
		build_start = clock();
		hipFree(detailed_nodes[root_median_id].x_sort_ids);
		hipFree(detailed_nodes[root_median_id].y_sort_ids);
		hipFree(detailed_nodes[root_median_id].z_sort_ids);


		hipMemcpy(&end_list[0], d_end_list, test_points.size() * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&detailed_nodes[0], d_detailed_nodes, test_points.size() * sizeof(detailed_node), hipMemcpyDeviceToHost);
		build_end = clock();

		

		

		hipFree(d_end_list);
		hipFree(d_detailed_nodes);
		
		// bool all_end = std::all_of(end_list.begin(), end_list.end(), [](int end) { return 0 < end; });
		// if(all_end) std::cout<<"successful termination"<<std::endl;
		// if(!all_end) std::cout<<"not all end"<<std::endl;
		
		// TreeOutCsv(detailed_nodes,0,root_median_id);
		
		// first=false;
	}
	// root_id=root_median_id;
	//表示用スクリプト

	/////////////////////////////////////////////////////////////////////////////////////////施工
	
	
	printf("create tree time is %.5fs\n",(double)(build_end-build_start)/CLOCKS_PER_SEC);
	//root_id表示
	// std::cout << "root_id = " << root_id << std::endl;
	//nodes表示

	// std::cout<<"search kernel start"<<std::endl;
	// std::cout<<"3.01"<<std::endl;
    //ホスト1次配列宣言
	//kd
	std::vector<int> h_parent_ids(points_array.size());
	std::vector<int> h_left_ids(points_array.size());
	std::vector<int> h_right_ids(points_array.size());
	std::vector<int> h_axes(points_array.size());

	std::vector<int> h_point_neighbor(points_array.size());
	std::vector<int> h_point_neighbor_size(1);
	std::vector<long long int> h_neighbor_time(points_array.size());

	//normal
    std::vector<float> h_points(points_array.size() * 3);
    std::vector<int> h_neighbor_points_indices(neighbor_points_count);
    std::vector<float> h_normals(points_array.size() * 3);
    std::vector<float> h_curvatures(points_array.size());
    std::vector<long long int> h_covariance_compute_time(points_array.size());
    std::vector<long long int> h_eigen_compute_time(points_array.size());
    // std::cout<<"3.02"<<std::endl;
    //デバイス1次配列宣言
	//kd
	int *d_parent_ids,*d_left_ids,*d_right_ids,*d_axes;
	int *d_point_neighbor,*d_point_neighbor_size;
	long long int *d_neighbor_time;

	//normal
    float *d_points,*d_normals,*d_curvatures;
    int *d_neighbor_points_indices,*d_neighbor_start_indices;
    long long int *d_covariance_compute_time,*d_eigen_compute_time;


    // std::cout<<"3.03"<<std::endl;
    //メモリ確保
	//kd
	hipMalloc((void **)&d_parent_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_left_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_right_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_axes, points_array.size() * sizeof(int));

	hipMalloc((void **)&d_point_neighbor, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_point_neighbor_size, sizeof(int));
	hipMalloc((void **)&d_neighbor_time, points_array.size() * sizeof(long long int));
	//normal
    hipMalloc((void **)&d_points, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_neighbor_points_indices, neighbor_points_count * sizeof(int));
    hipMalloc((void **)&d_neighbor_start_indices, points_array.size() * sizeof(int));
    hipMalloc((void **)&d_normals, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_curvatures, points_array.size() * sizeof(float));
    hipMalloc((void **)&d_covariance_compute_time, points_array.size() * sizeof(long long int));
    hipMalloc((void **)&d_eigen_compute_time, points_array.size() * sizeof(long long int));
    // std::cout<<"3.04"<<std::endl;
    //1次配列化
    int k=0,l=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            h_points[k]=points_array[i][j];
            k++;
        }
        for(int j=0;j<neighbor_points_indices[i].size();j++){//近傍
            h_neighbor_points_indices[l]=neighbor_points_indices[i][j];
            l++;
        }
		h_parent_ids[i]=nodes[i].parent_id;
		h_left_ids[i]=nodes[i].left_id;
		h_right_ids[i]=nodes[i].right_id;
		h_axes[i]=nodes[i].axis;
    }



    // std::cout<<"3.05"<<std::endl;
    //コピー
	//kd
	hipMemcpy(d_parent_ids, &h_parent_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_left_ids, &h_left_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_right_ids, &h_right_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_axes, &h_axes[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	//normal
    hipMemcpy(d_points, &h_points[0], points_array.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_points_indices, &h_neighbor_points_indices[0], neighbor_points_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_start_indices, &neighbor_start_indices[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
    // std::cout<<"3.06"<<std::endl;
    //並列スレッド設定
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((points_array.size() + block.x - 1) / block.x, 1);
    // std::cout<<"3.07"<<std::endl;
    // std::cout<<"normalsGPUstart"<<std::endl;
    // hipDeviceSetLimit(hipLimitStackSize, 1024*8);
    //実行
    NormalsGPU<<<grid,block>>>(/*d_detailed_nodes,*/d_neighbor_time,d_point_neighbor_size,d_point_neighbor,d_parent_ids,d_left_ids,d_right_ids,d_axes,root_id,d_points,points_array.size(),d_neighbor_points_indices,d_neighbor_start_indices,neighbor_points_count,d_normals,d_curvatures,d_covariance_compute_time,d_eigen_compute_time);
    // std::cout<<"normalsGPUend"<<std::endl;
    // std::cout<<"3.08"<<std::endl;
    //コピー
	//kd
	hipMemcpy(&h_point_neighbor[0], d_point_neighbor, points_array.size() * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_point_neighbor_size[0], d_point_neighbor_size, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_neighbor_time[0], d_neighbor_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
	//normal
    hipMemcpy(&h_normals[0], d_normals, points_array.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_curvatures[0], d_curvatures, points_array.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_covariance_compute_time[0], d_covariance_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_compute_time[0], d_eigen_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    // std::cout<<"3.09"<<std::endl;
    //2次配列化
	for(int i=0;i<h_point_neighbor_size[0];i++){
		point_neighbor[i]=h_point_neighbor[i];
	}
	
	point_neighbor.resize(h_point_neighbor_size[0]);
	// std::cout<<"host cu size "<<h_point_neighbor_size[0]<<std::endl;

    k=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            normals_array[i][j]=h_normals[k];
            k++;
        }
        curvatures_array[i]=h_curvatures[i];
        covariance_compute_time[i]=h_covariance_compute_time[i];
        eigen_compute_time[i]=h_eigen_compute_time[i];
		neighbor_time[i]=h_neighbor_time[i];
    }
	// std::cout<<"search kernel end"<<std::endl;
    // std::cout<<"cu_normals : "<<normals_array[0][0]<<","<<normals_array[0][1]<<","<<normals_array[0][2]<<std::endl;
    // std::cout<<"3.10"<<std::endl;
    //メモリ解放
	//kd2

	//kd
	hipFree(d_parent_ids);
	hipFree(d_left_ids);
	hipFree(d_right_ids);
	hipFree(d_axes);

	hipFree(d_point_neighbor);
	hipFree(d_point_neighbor_size);
	hipFree(d_neighbor_time);

	//normal
    hipFree(d_points);
    hipFree(d_neighbor_points_indices);
    hipFree(d_neighbor_start_indices);
    hipFree(d_normals);
    hipFree(d_curvatures);
    hipFree(d_covariance_compute_time);
    hipFree(d_eigen_compute_time);


	frames++;
}