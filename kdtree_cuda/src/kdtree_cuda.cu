#include "hip/hip_runtime.h"


#include <string.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "kdtree_cuda/kdtree_cuda.hpp"
#include <vector>
#include <iterator>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int sort_axis=0;

typedef struct
{
	int	id;
	float pos[3];
} point_with_id;

typedef struct
{
	int	parent_id;
	int left_id;
	int right_id;
	int axis;
} node;

bool first=true;

//	年齢(昇順)
__host__ int AxisSort(const void * n1, const void * n2)
{
	if (((point_with_id *)n1)->pos[sort_axis] > ((point_with_id *)n2)->pos[sort_axis])
	{
		return 1;
	}
	else if (((point_with_id *)n1)->pos[sort_axis] < ((point_with_id *)n2)->pos[sort_axis])
	{
		return -1;
	}
	else
	{
		return 0;
	}
}

__host__ int CreateTree(int* root_id,std::vector <node>& nodes, std::vector<std::vector<float>> points,std::vector<int> group_indices,int parent_id,bool node_is_right)
{
	//入力データ初期化
	int group_size = group_indices.size();
	// std::cout<<"group_size"<<group_size<<std::endl;
	point_with_id point_with_ids[group_size];
	std::vector<std::vector<int>> axis_sort_ids(3, std::vector<int>(group_size));
	// std::cout<<"oppai 1 "<<std::endl;
	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////3*points
		point_with_ids[i].id=group_indices[i];
		point_with_ids[i].pos[0]=points[group_indices[i]][0];
		point_with_ids[i].pos[1]=points[group_indices[i]][1];
		point_with_ids[i].pos[2]=points[group_indices[i]][2];
	}
	// std::cout<<"oppai 2 "<<std::endl;

	//ソート
	float max[3],min[3],median[3],length[3];
	int axis_median_id[3];
	int median_id;
	for(sort_axis=0; sort_axis<3; sort_axis++){//x,y,zそれぞれにソート
		// std::cout<<"sort_axis = "<<sort_axis<<std::endl;
		qsort(point_with_ids, group_size, sizeof(point_with_id), AxisSort);
		for (int i=0 ; i < group_size ; i++){///////////////////////////////////////////////////////////////////////////////////////////3*points
			axis_sort_ids[sort_axis][i]=point_with_ids[i].id;
			// printf("%d, %f, %f, %f \n", point_with_ids[i].id, point_with_ids[i].pos[0], point_with_ids[i].pos[1], point_with_ids[i].pos[2]);
		}
		// std::cout<<std::endl;
		//max,min,median,axis_median_id取得
		max[sort_axis]=point_with_ids[group_size-1].pos[sort_axis];//minとmaxいらんかも
		min[sort_axis]=point_with_ids[0].pos[sort_axis];
		length[sort_axis]=max[sort_axis]-min[sort_axis];
		median[sort_axis]=point_with_ids[(group_size-1)/2].pos[sort_axis];//偶数なら小さい方(-1)消せば大きい方
		axis_median_id[sort_axis]=point_with_ids[(group_size-1)/2].id;
	}
	// std::cout<<"oppai 3 "<<std::endl;
	// std::cout<<"x_length = "<< length[0] <<", x_median["<<axis_median_id[0]<<"] = "<<median[0]<<std::endl;
	// std::cout<<"y_length = "<< length[1] <<", y_median["<<axis_median_id[1]<<"] = "<<median[1]<<std::endl;
	// std::cout<<"z_length = "<< length[2] <<", z_median["<<axis_median_id[2]<<"] = "<<median[2]<<std::endl;
	// std::cout<<std::endl;
	// for(int i=0;i<group_size;i++){
	// 	std::cout<<"point_id["<<i<<"] = "<<point_with_ids[i].id<<std::endl;
	// }
	

	//中央値id設定、長軸設定
	if(length[0]>=length[1]&&length[0]>=length[2]){
		median_id=axis_median_id[0];
		nodes[median_id].axis=0;
	}
	if(length[1]>=length[0]&&length[1]>=length[2]){
		median_id=axis_median_id[1];
		nodes[median_id].axis=1;
	}
	if(length[2]>=length[0]&&length[2]>=length[1]){
		median_id=axis_median_id[2];
		nodes[median_id].axis=2;
	}
	// std::cout<<"oppai 4 "<<std::endl;

	for(int i=0;i<group_size;i++){/////////////////////////////////////////////////////////////////////////////////////////////////points
		group_indices[i]=axis_sort_ids[nodes[median_id].axis][i];
	}
	// std::cout<<"oppai 5 "<<std::endl;
	// memcpy(&group_indices[0], axis_sort_ids[nodes[median_id].axis], group_size*sizeof(int));
	// std::vector<int> group_indices2;
	// group_indices.resize(0);
	// copy(axis_sort_ids[nodes[median_id].axis].begin(), axis_sort_ids[nodes[median_id].axis].end(), back_inserter(group_indices) );



	//node初期化
	nodes[median_id].left_id=-1;
	nodes[median_id].right_id=-1;

	//親設定、親の左右設定
	nodes[median_id].parent_id=parent_id;
	if(parent_id>=0){//親あり
		if(!node_is_right) nodes[parent_id].left_id=median_id;
		if(node_is_right) nodes[parent_id].right_id=median_id;
	}
	else{//親なし
		*root_id=median_id;
	}
	// std::cout<<"oppai 6 "<<std::endl;

	// std::vector<int> right_group(group_size);
	// std::vector<int> left_group(group_size);
	// int right_count=0;
	// int left_count=0;

	// for(int i=0;i<=((group_size-1)/2)-1;i++){////////////////////////////////////////////////////////////////////////////////////points
	// 	left_group[left_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	left_count++;
	// }
	// left_group.resize(left_count);///////////////////////////////////////////////////////////////////////////////////////////////points
	// for(int i=((group_size-1)/2)+1;i<group_size;i++){
	// 	right_group[right_count] = axis_sort_ids[nodes[median_id].axis][i];
	// 	right_count++;
	// }
	// right_group.resize(right_count);

	// std::vector<int> v0{1,2,3,4};
	// std::vector<int> v9(v0.begin(),v0.end());  // [1,2,3,4]
	// std::vector<int> v10(v0.begin(),v0.end());  

	// group_indices.resize(8);
	// for(int i=0;i<8;i++){
	// 	group_indices[i]=i;
	// }
	// size_t middle = ((8-1)/2);

	size_t middle = ((group_size-1)/2);
	std::vector<int>::iterator middleIter(group_indices.begin());
	std::advance(middleIter, middle);
	// std::cout<<"advance end"<<std::endl;

	std::vector<int> left_group(group_indices.begin(), middleIter);
	++middleIter;
	std::vector<int> right_group(middleIter, group_indices.end());
	// std::cout<<"oppai 7 "<<std::endl;
	// std::cout<<"group end"<<std::endl;
	// std::cout<<"left group is [";
	// for(int i=0;i<left_group.size();i++){
	// 	if(i<10) std::cout<<left_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	// std::cout<<"right group is [";
	// for(int i=0;i<right_group.size();i++){
	// 	if(i<10) std::cout<<right_group[i]<<",";
	// }
	// std::cout<<"]"<<std::endl;

	//


	// std::cout<<"median_id"<<median_id<<std::endl;
	// std::cout<<"parent_id"<<parent_id<<std::endl;
	// std::cout<<"left_id"<<nodes[median_id].left_id<<std::endl;
	// std::cout<<"right_id"<<nodes[median_id].right_id<<std::endl;
	// std::cout<<"axis"<<nodes[median_id].axis<<std::endl;

	//right,left group表示
	// std::cout<<"left_group is (";
	// for(int i=0;i<left_group.size();i++){
	// 	std::cout<<left_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<"right_group is (";
	// for(int i=0;i<right_group.size();i++){
	// 	std::cout<<right_group[i]<<",";
	// }
	// std::cout<<")"<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<std::endl;
	// std::cout<<"--------------------------------------------------------------------------------"<<std::endl;
	bool left=false;
	bool right=false;
	if(group_size>1){//子がいる
		if(left_group.size()>0){//左に子がいる
			left= CreateTree(root_id,nodes,points,left_group,median_id,false);
		}
		else left=true;
		if(right_group.size()>0){//右に子がいる
			right= CreateTree(root_id,nodes,points,right_group,median_id,true);
		}
		else right=true;
		// std::cout<<"oppai 8 "<<std::endl;
		if(right&&left) return 1;
	}
	else return 1;//子がいない
}

__device__ void d_PointRangeCheckAndAdd(int *range_indices_size,int *range_indices,int head_id,float* points,float* search_point,float range_sq)//vecotrなしpowなしpushbackなしで作り直す
{
	float dist_sq = powf(points[head_id*3+0]-search_point[0],2)+powf(points[head_id*3+1]-search_point[1],2)+powf(points[head_id*3+2]-search_point[2],2);
	if(dist_sq<range_sq){
		// printf("device inside");
		range_indices[*range_indices_size] = head_id;
		*range_indices_size+=1;
	} 
}

__device__ int d_SearchSubTree(int *range_indices_size,int *range_indices,int root_id,node* nodes,float* points,float* search_point,float range_sq)//vecotrなしpowなしで作り直す
{
	int head_id = root_id;
	bool cross,next_is_right;
	//潜り
	while(1){
		if(search_point[nodes[head_id].axis]>points[head_id*3+nodes[head_id].axis]) next_is_right = true;
		else next_is_right = false;

		if(nodes[head_id].right_id>=0&&nodes[head_id].left_id<0) head_id = nodes[head_id].right_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id<0) head_id = nodes[head_id].left_id;//一人っ子
		else if(nodes[head_id].left_id>=0&&nodes[head_id].right_id>=0){//双子
			if(next_is_right){//right
				head_id = nodes[head_id].right_id;
			}
			else{//left
				head_id = nodes[head_id].left_id;
			}
		}
		else break;
	}
	//rootが底
	if(head_id==root_id) {
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		return 1;
	}
	//昇り
	int last_id;
	while(1){
		cross = false;
		d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
		//昇る
		last_id = head_id;
		head_id = nodes[head_id].parent_id;
		//交差判定　1軸のみ低効率
		float axis_diff_sq = powf(points[head_id*3+nodes[head_id].axis] - search_point[nodes[head_id].axis],2);
		if(axis_diff_sq < range_sq) cross = true;
		int sub_tree=0;
		if(cross){
			if(last_id==nodes[head_id].right_id&&nodes[head_id].left_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].left_id,nodes,points,search_point,range_sq);//右から上がってきた
			if(last_id==nodes[head_id].left_id&&nodes[head_id].right_id>0) sub_tree = d_SearchSubTree(range_indices_size,range_indices,nodes[head_id].right_id,nodes,points,search_point,range_sq);//左から上がってきた
		}
		if(head_id==root_id){
			d_PointRangeCheckAndAdd(range_indices_size,range_indices,head_id,points,search_point,range_sq);
			break;
		}
	}

	return 1;
}

__device__ int EigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 

__global__ void NormalsGPU(long long int* neighbor_time,int *point_neighbor_size,int* point_neighbor,int* d_parent_ids,int* d_left_ids,int* d_right_ids,int* d_axes,int root_id,float* points,int point_size,int* neighbor_points_indices,int* neighbor_start_indices,int neighbor_points_count,float* normals,float* curvatures,long long int* covariance_time,long long int* eigen_time)
{
    // printf("normalsGPU");
    //インデックス取得
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = ix;
    unsigned int output_id=50;
    // printf("idx = %d ,", idx);
	if(idx==output_id){
		long long int neighbor_start, neighbor_stop;
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_start));
		////////////////////////////////////////////////////////////////
		node *nodes = (node*)malloc(sizeof(node) * point_size);

		for(int i=0;i<point_size;i++){
			nodes[i].parent_id=d_parent_ids[i];
			nodes[i].left_id=d_left_ids[i];
			nodes[i].right_id=d_right_ids[i];
			nodes[i].axis=d_axes[i];
		}
		float search_point[3];
		search_point[0]=points[idx*3+0];
		search_point[1]=points[idx*3+1];
		search_point[2]=points[idx*3+2];
		
		int *range_indices = (int*)malloc(sizeof(int) * point_size);
		int range_indices_size = 0;

		float range_sq = 0.15*0.15;

		//探索関数の実行
		int range_search = d_SearchSubTree(&range_indices_size,range_indices,root_id,nodes,points,search_point,range_sq);
		// std::cout<<"range_indices.size()"<<range_indices.size()<<std::endl;
		if(range_search==1) {
			// printf("device range_indices size is =%d",range_indices_size);
			// printf("device range_indices is [");
			for(int i=0;i<range_indices_size;i++){
				// printf("%d,",range_indices[i]);
				point_neighbor[i]=range_indices[i];
			}
			// printf("]\n");
			point_neighbor_size[0]=range_indices_size;
			// printf("device size %d\n",range_indices_size);
		}
		
		free (nodes);
		free (range_indices);
		asm volatile("mov.u64  %0, %globaltimer;" : "=l"(neighbor_stop));
		neighbor_time[idx]=neighbor_stop - neighbor_start;
		////////////////////////////////////////////////////////////////////
	}


    if(idx<point_size-1){//対象スレッド内のみ計算
        //デバッグ用
        // if(idx==0||idx==10||idx==20) printf("points(%d) = %f,%f,%f\n",idx,points[idx*3+0],points[idx*3+1],points[idx*3+2]);
        
        // printf("idx<point_size");
        //近傍点終点インデックスの定義
        int end_indices;
        if(idx==(point_size-1)) end_indices = neighbor_points_count;
        else end_indices = neighbor_start_indices[idx+1]-1;

        int neighbor_size=(end_indices-neighbor_start_indices[idx]+1)/3;
        // if(idx==0||idx==10||idx==20) printf("neighbor(%d) start = %d, end = %d,size = %d\n",idx,neighbor_start_indices[idx],end_indices,neighbor_size);
        // printf("neighbor_size = %d\n", neighbor_size);
        if(neighbor_size>=3){//近傍点数3以上
            long long int covariance_start, covariance_stop;
            long long int eigen_start,eigen_stop;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_start));
            // printf("neighbor_size>=3");
            //平均計算
            float x_average=0,y_average=0,z_average=0;
            // if(idx==output_id) printf("neighbor_size = %d\n",neighbor_size);

            // if(idx==output_id) printf("neighbor_points(%d) = {\n",idx);
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                // デバッグ用
                // if((idx==0||idx==10||idx==20)&&(i==(neighbor_start_indices[idx]))) printf("neighbor_points_indices(%d) = %d\n",idx,neighbor_points_indices[i]);
                // if((idx==0||idx==10||idx==20)&&i==neighbor_start_indices[idx]) printf("neighbor_points(%d) = %f,%f,%f\n",idx,points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                // if(idx==output_id) printf("{%f, %f, %f},\n",points[neighbor_points_indices[i]*3+0],points[neighbor_points_indices[i]*3+1],points[neighbor_points_indices[i]*3+2]);
                x_average+=points[neighbor_points_indices[i]*3+0];
                y_average+=points[neighbor_points_indices[i]*3+1];
                z_average+=points[neighbor_points_indices[i]*3+2];
            }
            // if(idx==output_id) printf("};\n");
            x_average/=neighbor_size;
            y_average/=neighbor_size;
            z_average/=neighbor_size;

            // //要素計算
            float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
            for(int i=neighbor_start_indices[idx];i<=end_indices;i+=3){//近傍点
                sxx+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+0]-x_average);
                syy+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                szz+=(points[neighbor_points_indices[i]*3+2]-z_average)*(points[neighbor_points_indices[i]*3+2]-z_average);

                sxy+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+1]-y_average);
                sxz+=(points[neighbor_points_indices[i]*3+0]-x_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
                syz+=(points[neighbor_points_indices[i]*3+1]-y_average)*(points[neighbor_points_indices[i]*3+2]-z_average);
            }

            sxx/=neighbor_size;
            syy/=neighbor_size;
            szz/=neighbor_size;
            sxy/=neighbor_size;
            sxz/=neighbor_size;
            syz/=neighbor_size;

            //共分散行列
            float a[3*3]={
                sxx,sxy,sxz,
                sxy,syy,syz,
                sxz,syz,szz,
            };

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(covariance_stop));
            covariance_time[idx]=covariance_stop - covariance_start;

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_start));
            
            // __syncthreads();
            // if(idx==output_id){
            //     printf("                          %f ,%f ,%f \ncovariance matrix(%d)=   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,idx,sxy,syy,syz,sxz,syz,szz);
            // }

            //固有値計算
            float eigen_vector[3 * 3];
            EigenJacobiMethod(a, eigen_vector, 3);

            // __syncthreads();
            // if(neighbor_size<3){
            //     printf("               %f ,%f ,%f \neigen_value=   %f ,%f ,%f \n               %f ,%f ,%f \n\n",a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7],a[8]);
            // }

            float eigen_value[3];
            eigen_value[0]=a[0];
            eigen_value[1]=a[4];
            eigen_value[2]=a[8];

            int min_eigen_axis=0;
            float min_eigen_value=eigen_value[0];
            float eigen_sum=0;
            for(int i=1;i<3;i++){//x,y,z
                if(eigen_value[i]<min_eigen_value){
                    min_eigen_value=eigen_value[i];
                    min_eigen_axis=i;
                }
                //正規化用にnorm計算しておく
                eigen_sum += eigen_value[i];
            }

            asm volatile("mov.u64  %0, %globaltimer;" : "=l"(eigen_stop));
            eigen_time[idx]=eigen_stop - eigen_start;

            normals[idx*3+0]=eigen_vector[min_eigen_axis+0];
            normals[idx*3+1]=eigen_vector[min_eigen_axis+3];
            normals[idx*3+2]=eigen_vector[min_eigen_axis+6];

            curvatures[idx]=min_eigen_value/eigen_sum;

            // if(idx==output_id){
            //     printf("normals(%d) = %f, %f, %f\n\n\n\n",idx,normals[idx*3+0],normals[idx*3+1],normals[idx*3+2]);
            //     printf("curvature(%d) = %f\n",idx,curvatures[idx]);
            // }

            //デバッグ用
            // normals[idx*3+0]=idx*10+0;
            // normals[idx*3+1]=idx*10+1;
            // normals[idx*3+2]=idx*10+2;
            // printf("normal_x = %f ,normal_y = %f ,normal_z = %f \n", normals[idx*3+1],normals[idx*3+2],normals[idx*3+3]);
        }
        else{
            normals[idx*3+0]=0;
            normals[idx*3+1]=0;
            normals[idx*3+2]=0;
            curvatures[idx]=0;
            covariance_time[idx]=0;
            eigen_time[idx]=0;
        }
    }
    
}

extern void ComputeNormals(std::vector<long long int>& neighbor_time,std::vector<int>& point_neighbor,std::vector<std::vector<float>> points_array,std::vector<std::vector<int>> neighbor_points_indices,std::vector<int> neighbor_start_indices,int neighbor_points_count,std::vector<std::vector<float>>& normals_array,std::vector<float>& curvatures_array,std::vector<long long int>& covariance_compute_time,std::vector<long long int>& eigen_compute_time)
{
	std::vector<int> root_indices(points_array.size());
	for(int i=0;i<points_array.size();i++){
		root_indices[i]=i;
	}

	//nodesとroot_id初期化しなくていい？
	std::vector <node> nodes;
	nodes.resize(points_array.size());
	// std::cout<<"sizeof(root_indices) = "<<sizeof(root_indices)<<std::endl;
	int root_id=-1;
	clock_t build_start,build_end;
	build_start = clock();
	int create_end = CreateTree(&root_id,nodes,points_array,root_indices,-1,false);
	build_end = clock();
	printf("create tree time is %.2fs\n",(double)(build_end-build_start)/CLOCKS_PER_SEC);

	// if(first){
	// 	if(create_end==1){
	// 		for(int i=0;i<points_array.size();i++){
	// 			std::cout<<"node["<<i<<"] axis = "<<nodes[i].axis<<", parent_id = "<<nodes[i].parent_id<<", left_id = "<<nodes[i].left_id<<", right_id = "<<nodes[i].right_id<<std::endl;
	// 		}
	// 	}
	// } 

	// std::vector<float> search_point={8,1,0};
	// // std::vector<float> search_point={11,5,0};
	// std::vector<int> range_indices;
	// float range_sq = 8.5*8.5;
	// //探索関数の実行
	// int range_search = SearchSubTree(range_indices,root_id,nodes,points,search_point,range_sq);
	// // std::cout<<"range_indices.size()"<<range_indices.size()<<std::endl;
	// if(first){
	// 	if(range_search==1) {
	// 		std::cout<<"host range_indices is [";
	// 		for(int i=0;i<range_indices.size();i++){
	// 			std::cout<<range_indices[i]<<",";
	// 		}
	// 		std::cout<<"]"<<std::endl;
	// 	}
	// }


	// std::cout<<"3.01"<<std::endl;
    //ホスト1次配列宣言
	//kd
	std::vector<int> h_parent_ids(points_array.size());
	std::vector<int> h_left_ids(points_array.size());
	std::vector<int> h_right_ids(points_array.size());
	std::vector<int> h_axes(points_array.size());

	std::vector<int> h_point_neighbor(points_array.size());
	std::vector<int> h_point_neighbor_size(1);
	std::vector<long long int> h_neighbor_time(points_array.size());

	//normal
    std::vector<float> h_points(points_array.size() * 3);
    std::vector<int> h_neighbor_points_indices(neighbor_points_count);
    std::vector<float> h_normals(points_array.size() * 3);
    std::vector<float> h_curvatures(points_array.size());
    std::vector<long long int> h_covariance_compute_time(points_array.size());
    std::vector<long long int> h_eigen_compute_time(points_array.size());
    // std::cout<<"3.02"<<std::endl;
    //デバイス1次配列宣言
	//kd
	int *d_parent_ids,*d_left_ids,*d_right_ids,*d_axes;
	int *d_point_neighbor,*d_point_neighbor_size;
	long long int *d_neighbor_time;

	//normal
    float *d_points,*d_normals,*d_curvatures;
    int *d_neighbor_points_indices,*d_neighbor_start_indices;
    long long int *d_covariance_compute_time,*d_eigen_compute_time;


    // std::cout<<"3.03"<<std::endl;
    //メモリ確保
	//kd
	hipMalloc((void **)&d_parent_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_left_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_right_ids, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_axes, points_array.size() * sizeof(int));

	hipMalloc((void **)&d_point_neighbor, points_array.size() * sizeof(int));
	hipMalloc((void **)&d_point_neighbor_size, sizeof(int));
	hipMalloc((void **)&d_neighbor_time, points_array.size() * sizeof(long long int));
	//normal
    hipMalloc((void **)&d_points, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_neighbor_points_indices, neighbor_points_count * sizeof(int));
    hipMalloc((void **)&d_neighbor_start_indices, points_array.size() * sizeof(int));
    hipMalloc((void **)&d_normals, points_array.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_curvatures, points_array.size() * sizeof(float));
    hipMalloc((void **)&d_covariance_compute_time, points_array.size() * sizeof(long long int));
    hipMalloc((void **)&d_eigen_compute_time, points_array.size() * sizeof(long long int));
    // std::cout<<"3.04"<<std::endl;
    //1次配列化
    int k=0,l=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            h_points[k]=points_array[i][j];
            k++;
        }
        for(int j=0;j<neighbor_points_indices[i].size();j++){//近傍
            h_neighbor_points_indices[l]=neighbor_points_indices[i][j];
            l++;
        }
		h_parent_ids[i]=nodes[i].parent_id;
		h_left_ids[i]=nodes[i].left_id;
		h_right_ids[i]=nodes[i].right_id;
		h_axes[i]=nodes[i].axis;
    }



    // std::cout<<"3.05"<<std::endl;
    //コピー
	//kd
	hipMemcpy(d_parent_ids, &h_parent_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_left_ids, &h_left_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_right_ids, &h_right_ids[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_axes, &h_axes[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
	//normal
    hipMemcpy(d_points, &h_points[0], points_array.size() * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_points_indices, &h_neighbor_points_indices[0], neighbor_points_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_start_indices, &neighbor_start_indices[0], points_array.size() * sizeof(int), hipMemcpyHostToDevice);
    // std::cout<<"3.06"<<std::endl;
    //並列スレッド設定
    int dimx = 32;
    dim3 block(dimx, 1);
    dim3 grid((points_array.size() + block.x - 1) / block.x, 1);
    // std::cout<<"3.07"<<std::endl;
    // std::cout<<"normalsGPUstart"<<std::endl;
    hipDeviceSetLimit(hipLimitStackSize, 1024*8);
    //実行
    NormalsGPU<<<grid,block>>>(d_neighbor_time,d_point_neighbor_size,d_point_neighbor,d_parent_ids,d_left_ids,d_right_ids,d_axes,root_id,d_points,points_array.size(),d_neighbor_points_indices,d_neighbor_start_indices,neighbor_points_count,d_normals,d_curvatures,d_covariance_compute_time,d_eigen_compute_time);
    // std::cout<<"normalsGPUend"<<std::endl;
    // std::cout<<"3.08"<<std::endl;
    //コピー
	//kd
	hipMemcpy(&h_point_neighbor[0], d_point_neighbor, points_array.size() * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_point_neighbor_size[0], d_point_neighbor_size, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_neighbor_time[0], d_neighbor_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
	//normal
    hipMemcpy(&h_normals[0], d_normals, points_array.size() * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_curvatures[0], d_curvatures, points_array.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_covariance_compute_time[0], d_covariance_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_compute_time[0], d_eigen_compute_time, points_array.size() * sizeof(long long int), hipMemcpyDeviceToHost);
    // std::cout<<"3.09"<<std::endl;
    //2次配列化
	for(int i=0;i<h_point_neighbor_size[0];i++){
		point_neighbor[i]=h_point_neighbor[i];
	}
	
	point_neighbor.resize(h_point_neighbor_size[0]);
	// std::cout<<"host cu size "<<h_point_neighbor_size[0]<<std::endl;

    k=0;
    for(int i=0;i<points_array.size();i++){//点群
        for(int j=0;j<3;j++){//x,y,z
            normals_array[i][j]=h_normals[k];
            k++;
        }
        curvatures_array[i]=h_curvatures[i];
        covariance_compute_time[i]=h_covariance_compute_time[i];
        eigen_compute_time[i]=h_eigen_compute_time[i];
		neighbor_time[i]=h_neighbor_time[i];
    }
    // std::cout<<"cu_normals : "<<normals_array[0][0]<<","<<normals_array[0][1]<<","<<normals_array[0][2]<<std::endl;
    // std::cout<<"3.10"<<std::endl;
    //メモリ解放
	//kd
	hipFree(d_parent_ids);
	hipFree(d_left_ids);
	hipFree(d_right_ids);
	hipFree(d_axes);

	hipFree(d_point_neighbor);
	hipFree(d_point_neighbor_size);
	hipFree(d_neighbor_time);

	//normal
    hipFree(d_points);
    hipFree(d_neighbor_points_indices);
    hipFree(d_neighbor_start_indices);
    hipFree(d_normals);
    hipFree(d_curvatures);
    hipFree(d_covariance_compute_time);
    hipFree(d_eigen_compute_time);
}