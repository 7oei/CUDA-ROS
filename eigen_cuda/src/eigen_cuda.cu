#include "hip/hip_runtime.h"
#include <stdio.h>
#include "eigen_cuda/eigen_cuda.hpp"
#include <vector>

__global__ void covariance(float* neighbor_points,float* matrix,int point_size) {
    //平均計算
    float x_average=0,y_average=0,z_average=0;
    for(int i=0;i<point_size*3;i+=3){
        x_average+=neighbor_points[i];
        y_average+=neighbor_points[i+1];
        z_average+=neighbor_points[i+2];
    }
    x_average/=point_size;
    y_average/=point_size;
    z_average/=point_size;

    //要素計算
    float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
    for(int i=0;i<point_size*3;i+=3){
        sxx+=(neighbor_points[i]-x_average)*(neighbor_points[i]-x_average);
        syy+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+1]-y_average);
        szz+=(neighbor_points[i+2]-z_average)*(neighbor_points[i+2]-z_average);

        sxy+=(neighbor_points[i]-x_average)*(neighbor_points[i+1]-y_average);
        sxz+=(neighbor_points[i]-x_average)*(neighbor_points[i+2]-z_average);
        syz+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+2]-z_average);
    }
    sxx/=point_size;
    syy/=point_size;
    szz/=point_size;
    sxy/=point_size;
    sxz/=point_size;
    syz/=point_size;
    //出力
    matrix[0]=sxx;matrix[1]=sxy;matrix[2]=sxz;
    matrix[3]=sxy;matrix[4]=syy;matrix[5]=syz;
    matrix[6]=sxz;matrix[7]=syz;matrix[8]=szz;
    
}

extern void covarianceMatrix(std::vector<std::vector<float>> neighbor_points,float Matrix[3][3]){
    //変数宣言
    std::vector<float> h_neighbor_points(neighbor_points.size() * 3);
    std::vector<float> h_matrix(3 * 3);
    float *d_neighbor_points, *d_matrix;

    //メモリ確保
    hipMalloc((void **)&d_neighbor_points, neighbor_points.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_matrix, 3 * 3 * sizeof(float));

    //配列化
    int k=0;
    for(int i=0;i<neighbor_points.size();i++){
        for(int j=0;j<3;j++){
            h_neighbor_points[k]=neighbor_points[i][j];
            k++;
        }
    }
    
    //コピー
    hipMemcpy(d_neighbor_points, &h_neighbor_points[0], neighbor_points.size() * 3 * sizeof(float), hipMemcpyHostToDevice);

    covariance<<<1, 1>>>(d_neighbor_points,d_matrix,neighbor_points.size());

    //配列にコピー
    hipMemcpy(&h_matrix[0], d_matrix, 3 * 3 * sizeof(float), hipMemcpyDeviceToHost);

    //行列化
    k=0;
    for(int i=0;i<3;i++){
        for(int j=0;j<3;j++){
            Matrix[i][j]=h_matrix[k];
            k++;
        }
    }

    //メモリバラシ
    hipFree(d_neighbor_points);
    hipFree(d_matrix);
    
}