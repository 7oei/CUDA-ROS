#include "hip/hip_runtime.h"
#include <stdio.h>
#include "min_cuda_node/hello.hpp"

__global__ void hello(void) {
    printf("hello cuda !! thredIdx=%d\n", threadIdx.x);
}

void callHello(void) {
    hello<<<1, 4>>>();
    hipDeviceSynchronize();
}