#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cv_cuda/cv_cuda.hpp"

__global__ void cvhello(void) {
    printf("hello cuda !! thredIdx=%d\n", threadIdx.x);
}

void cvcallHello(void) {
    cvhello<<<1, 4>>>();
    hipDeviceSynchronize();
}