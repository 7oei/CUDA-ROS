#include "hip/hip_runtime.h"
#include <stdio.h>
#include "normal_cuda/normal_cuda.hpp"
#include <vector>
#include <iostream>

__device__ int eigenJacobiMethod(float *a, float *v, int n, float eps = 1e-8, int iter_max = 100)
{
    float *bim, *bjm;
    float bii, bij, bjj, bji;
 
    bim = new float[n];
    bjm = new float[n];
 
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            v[i*n+j] = (i == j) ? 1.0 : 0.0;
        }
    }
 
    int cnt = 0;
    for(;;){
        int i, j;
 
        float x = 0.0;
        for(int ia = 0; ia < n; ++ia){
            for(int ja = 0; ja < n; ++ja){
                int idx = ia*n+ja;
                if(ia != ja && fabs(a[idx]) > x){
                    i = ia;
                    j = ja;
                    x = fabs(a[idx]);
                }
            }
        }
 
        float aii = a[i*n+i];
        float ajj = a[j*n+j];
        float aij = a[i*n+j];
 
        float alpha, beta;
        alpha = (aii-ajj)/2.0;
        beta  = sqrt(alpha*alpha+aij*aij);
 
        float st, ct;
        ct = sqrt((1.0+fabs(alpha)/beta)/2.0);    // sinθ
        st = (((aii-ajj) >= 0.0) ? 1.0 : -1.0)*aij/(2.0*beta*ct);    // cosθ
 
        // A = PAPの計算
        for(int m = 0; m < n; ++m){
            if(m == i || m == j) continue;
 
            float aim = a[i*n+m];
            float ajm = a[j*n+m];
 
            bim[m] =  aim*ct+ajm*st;
            bjm[m] = -aim*st+ajm*ct;
        }
 
        bii = aii*ct*ct+2.0*aij*ct*st+ajj*st*st;
        bij = 0.0;
 
        bjj = aii*st*st-2.0*aij*ct*st+ajj*ct*ct;
        bji = 0.0;
 
        for(int m = 0; m < n; ++m){
            a[i*n+m] = a[m*n+i] = bim[m];
            a[j*n+m] = a[m*n+j] = bjm[m];
        }
        a[i*n+i] = bii;
        a[i*n+j] = bij;
        a[j*n+j] = bjj;
        a[j*n+i] = bji;
 
        // V = PVの計算
        for(int m = 0; m < n; ++m){
            float vmi = v[m*n+i];
            float vmj = v[m*n+j];
 
            bim[m] =  vmi*ct+vmj*st;
            bjm[m] = -vmi*st+vmj*ct;
        }
        for(int m = 0; m < n; ++m){
            v[m*n+i] = bim[m];
            v[m*n+j] = bjm[m];
        }
 
        float e = 0.0;
        for(int ja = 0; ja < n; ++ja){
            for(int ia = 0; ia < n; ++ia){
                if(ia != ja){
                    e += fabs(a[ja*n+ia]);
                }
            }
        }
        if(e < eps) break;
 
        cnt++;
        if(cnt > iter_max) break;
    }
 
    delete [] bim;
    delete [] bjm;
 
    return cnt;
} 


__global__ void covarianceGPU(float* neighbor_points,float* matrix,int point_size) {
    //平均計算
    float x_average=0,y_average=0,z_average=0;
    for(int i=0;i<point_size*3;i+=3){
        x_average+=neighbor_points[i];
        y_average+=neighbor_points[i+1];
        z_average+=neighbor_points[i+2];
    }
    x_average/=point_size;
    y_average/=point_size;
    z_average/=point_size;

    //要素計算
    float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
    for(int i=0;i<point_size*3;i+=3){
        sxx+=(neighbor_points[i]-x_average)*(neighbor_points[i]-x_average);
        syy+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+1]-y_average);
        szz+=(neighbor_points[i+2]-z_average)*(neighbor_points[i+2]-z_average);

        sxy+=(neighbor_points[i]-x_average)*(neighbor_points[i+1]-y_average);
        sxz+=(neighbor_points[i]-x_average)*(neighbor_points[i+2]-z_average);
        syz+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+2]-z_average);
    }
    sxx/=point_size;
    syy/=point_size;
    szz/=point_size;
    sxy/=point_size;
    sxz/=point_size;
    syz/=point_size;
    //出力
    matrix[0]=sxx;matrix[1]=sxy;matrix[2]=sxz;
    matrix[3]=sxy;matrix[4]=syy;matrix[5]=syz;
    matrix[6]=sxz;matrix[7]=syz;matrix[8]=szz;
    
}

__global__ void eigenGPU(float* neighbor_points,float* eigen_vector,float* eigen_value,int point_size) {
    //平均計算
    float x_average=0,y_average=0,z_average=0;
    for(int i=0;i<point_size*3;i+=3){
        x_average+=neighbor_points[i];
        y_average+=neighbor_points[i+1];
        z_average+=neighbor_points[i+2];
    }
    x_average/=point_size;
    y_average/=point_size;
    z_average/=point_size;

    //要素計算
    float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
    for(int i=0;i<point_size*3;i+=3){
        sxx+=(neighbor_points[i]-x_average)*(neighbor_points[i]-x_average);
        syy+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+1]-y_average);
        szz+=(neighbor_points[i+2]-z_average)*(neighbor_points[i+2]-z_average);

        sxy+=(neighbor_points[i]-x_average)*(neighbor_points[i+1]-y_average);
        sxz+=(neighbor_points[i]-x_average)*(neighbor_points[i+2]-z_average);
        syz+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+2]-z_average);
    }
    sxx/=point_size;
    syy/=point_size;
    szz/=point_size;
    sxy/=point_size;
    sxz/=point_size;
    syz/=point_size;
    //共分散行列
    float a[3*3]={
        sxx,sxy,sxz,
        sxy,syy,syz,
        sxz,syz,szz,
    };
    //固有値計算
    eigenJacobiMethod(a, eigen_vector, 3);
    eigen_value[0]=a[0];
    eigen_value[1]=a[4];
    eigen_value[2]=a[8];
    
}

__global__ void normalGPU(float* neighbor_points,float* normal_vecotr,int point_size) {
    //平均計算
    float x_average=0,y_average=0,z_average=0;
    for(int i=0;i<point_size*3;i+=3){
        x_average+=neighbor_points[i];
        y_average+=neighbor_points[i+1];
        z_average+=neighbor_points[i+2];
    }
    x_average/=point_size;
    y_average/=point_size;
    z_average/=point_size;

    //要素計算
    float sxx=0,sxy=0,sxz=0,syy=0,syz=0,szz=0;
    for(int i=0;i<point_size*3;i+=3){
        sxx+=(neighbor_points[i]-x_average)*(neighbor_points[i]-x_average);
        syy+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+1]-y_average);
        szz+=(neighbor_points[i+2]-z_average)*(neighbor_points[i+2]-z_average);

        sxy+=(neighbor_points[i]-x_average)*(neighbor_points[i+1]-y_average);
        sxz+=(neighbor_points[i]-x_average)*(neighbor_points[i+2]-z_average);
        syz+=(neighbor_points[i+1]-y_average)*(neighbor_points[i+2]-z_average);
    }
    sxx/=point_size;
    syy/=point_size;
    szz/=point_size;
    sxy/=point_size;
    sxz/=point_size;
    syz/=point_size;
    //共分散行列
    float a[3*3]={
        sxx,sxy,sxz,
        sxy,syy,syz,
        sxz,syz,szz,
    };
    // printf("                          %f ,%f ,%f \ncovariance matrix     =   %f ,%f ,%f \n                          %f ,%f ,%f \n\n",sxx,sxy,sxz,sxy,syy,syz,sxz,syz,szz);
    //固有値計算
    float eigen_vector[3 * 3];
    eigenJacobiMethod(a, eigen_vector, 3);
    float eigen_value[3];
    eigen_value[0]=a[0];
    eigen_value[1]=a[4];
    eigen_value[2]=a[8];

    int min_eigen_axis=0;
    float min_eigen_value=eigen_value[0];
    for(int i=1;i<3;i++){
        if(eigen_value[i]<min_eigen_value){
            min_eigen_value=eigen_value[i];
            min_eigen_axis=i;
        }
    }
    normal_vecotr[0]=eigen_vector[min_eigen_axis*3+0];
    normal_vecotr[1]=eigen_vector[min_eigen_axis*3+1];
    normal_vecotr[2]=eigen_vector[min_eigen_axis*3+2];
    
    // printf("normals = %f, %f, %f\n\n\n\n",normal_vecotr[0],normal_vecotr[1],normal_vecotr[2]);
    
}


extern void covariance(std::vector<std::vector<float>> neighbor_points,float matrix[3][3]){
    //変数宣言
    std::vector<float> h_neighbor_points(neighbor_points.size() * 3);
    std::vector<float> h_matrix(3 * 3);
    float *d_neighbor_points, *d_matrix;

    //メモリ確保
    hipMalloc((void **)&d_neighbor_points, neighbor_points.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_matrix, 3 * 3 * sizeof(float));

    //配列化
    int k=0;
    for(int i=0;i<neighbor_points.size();i++){
        for(int j=0;j<3;j++){
            h_neighbor_points[k]=neighbor_points[i][j];
            k++;
        }
    }
    
    //コピー
    hipMemcpy(d_neighbor_points, &h_neighbor_points[0], neighbor_points.size() * 3 * sizeof(float), hipMemcpyHostToDevice);

    covarianceGPU<<<1, 1>>>(d_neighbor_points,d_matrix,neighbor_points.size());

    //配列にコピー
    hipMemcpy(&h_matrix[0], d_matrix, 3 * 3 * sizeof(float), hipMemcpyDeviceToHost);

    //行列化
    k=0;
    for(int i=0;i<3;i++){
        for(int j=0;j<3;j++){
            matrix[i][j]=h_matrix[k];
            k++;
        }
    }

    //メモリバラシ
    hipFree(d_neighbor_points);
    hipFree(d_matrix);
    
}

extern void eigen(std::vector<std::vector<float>> neighbor_points,float eigen_vector[3][3],float eigen_value[3]){
    //変数宣言
    std::vector<float> h_neighbor_points(neighbor_points.size() * 3);
    std::vector<float> h_eigen_vector(3 * 3);
    std::vector<float> h_eigen_value(3);
    float *d_neighbor_points, *d_eigen_vector, *d_eigen_value;

    //メモリ確保
    hipMalloc((void **)&d_neighbor_points, neighbor_points.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_eigen_vector, 3 * 3 * sizeof(float));
    hipMalloc((void **)&d_eigen_value, 3 * sizeof(float));

    //配列化
    int k=0;
    for(int i=0;i<neighbor_points.size();i++){
        for(int j=0;j<3;j++){
            h_neighbor_points[k]=neighbor_points[i][j];
            k++;
        }
    }
    
    //コピー
    hipMemcpy(d_neighbor_points, &h_neighbor_points[0], neighbor_points.size() * 3 * sizeof(float), hipMemcpyHostToDevice);

    eigenGPU<<<1, 1>>>(d_neighbor_points,d_eigen_vector,d_eigen_value,neighbor_points.size());

    //配列にコピー
    hipMemcpy(&h_eigen_vector[0], d_eigen_vector, 3 * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_eigen_value[0], d_eigen_value, 3 * sizeof(float), hipMemcpyDeviceToHost);

    //行列化
    k=0;
    for(int i=0;i<3;i++){
        for(int j=0;j<3;j++){
            eigen_vector[i][j]=h_eigen_vector[k];
            eigen_value[i]=h_eigen_value[i];
            k++;
        }
    }

    //メモリバラシ
    hipFree(d_neighbor_points);
    hipFree(d_eigen_vector);
    hipFree(d_eigen_value);
    
}


extern void normal(std::vector<std::vector<float>> neighbor_points,float normal_vecotr[3]){
    //変数宣言
    std::vector<float> h_neighbor_points(neighbor_points.size() * 3);
    std::vector<float> h_normal_vector(3);
    float *d_neighbor_points, *d_normal_vecotr;

    //メモリ確保
    hipMalloc((void **)&d_neighbor_points, neighbor_points.size() * 3 * sizeof(float));
    hipMalloc((void **)&d_normal_vecotr, 3 * sizeof(float));

    //配列化
    int k=0;
    for(int i=0;i<neighbor_points.size();i++){
        for(int j=0;j<3;j++){
            h_neighbor_points[k]=neighbor_points[i][j];
            k++;
        }
    }
    
    //コピー
    hipMemcpy(d_neighbor_points, &h_neighbor_points[0], neighbor_points.size() * 3 * sizeof(float), hipMemcpyHostToDevice);

    normalGPU<<<1, 1>>>(d_neighbor_points,d_normal_vecotr,neighbor_points.size());

    //配列にコピー
    hipMemcpy(&h_normal_vector[0], d_normal_vecotr, 3 * sizeof(float), hipMemcpyDeviceToHost);

    //行列化
    for(int i=0;i<3;i++){
        normal_vecotr[i]=h_normal_vector[i];
    }

    //メモリバラシ
    hipFree(d_neighbor_points);
    hipFree(d_normal_vecotr);
    
}